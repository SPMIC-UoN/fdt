#include "hip/hip_runtime.h"
/*  PVM_single.cu

    Tim Behrens, Saad Jbabdi, Stam Sotiropoulos, Moises Hernandez  - FMRIB Image Analysis Group

    Copyright (C) 2005 University of Oxford  */

/*  CCOPYRIGHT  */

#include "diffmodels_utils.h"
#include "levenberg_marquardt.cu"
#include "options.h"

//#include <fstream>

/////////////////////////////////////
/////////////////////////////////////
/// 	    PVM_single		  /// 
/////////////////////////////////////
/////////////////////////////////////

__device__ 
inline double isoterm_PVM_single(const int pt,const double* _d,const double *bvals){
  	return exp(-bvals[pt]**_d);
}

__device__ 
inline double isoterm_d_PVM_single(const int pt,const double* _d,const double *bvals){
  	return (-bvals[pt]*exp(-bvals[pt]**_d));
}

__device__ 
inline double anisoterm_PVM_single(const int pt,const double* _d,const double3 x, const double *bvecs, const double *bvals, const int ndirections){
	double dp = bvecs[pt]*x.x+bvecs[ndirections+pt]*x.y+bvecs[(2*ndirections)+pt]*x.z;
	return exp(-bvals[pt]**_d*dp*dp);
}

__device__ 
inline double anisoterm_d_PVM_single(const int pt,const double* _d,const double3 x,const double *bvecs, const double *bvals, const int ndirections){
	double dp = bvecs[pt]*x.x+bvecs[ndirections+pt]*x.y+bvecs[(2*ndirections)+pt]*x.z;
  	return(-bvals[pt]*dp*dp*exp(-bvals[pt]**_d*dp*dp));
}

__device__ 
inline double anisoterm_th_PVM_single(const int pt,const double* _d,const double3 x, const double _th,const double _ph,const double *bvecs, const double *bvals, const int ndirections){
	double sinth,costh,sinph,cosph;
	sincos(_th,&sinth,&costh);
	sincos(_ph,&sinph,&cosph);
	double dp = bvecs[pt]*x.x+bvecs[ndirections+pt]*x.y+bvecs[(2*ndirections)+pt]*x.z;
	double dp1 = (costh*(bvecs[pt]*cosph+bvecs[ndirections+pt]*sinph)-bvecs[(2*ndirections)+pt]*sinth);
  	return(-2*bvals[pt]**_d*dp*dp1*exp(-bvals[pt]**_d*dp*dp));
}

__device__ 
inline double anisoterm_ph_PVM_single(const int pt,const double* _d,const double3 x, const double _th,const double _ph,const double *bvecs, const double *bvals, const int ndirections){
	double sinth,sinph,cosph;
	sinth=sin(_th);
	sincos(_ph,&sinph,&cosph);
  	double dp = bvecs[pt]*x.x+bvecs[ndirections+pt]*x.y+bvecs[(2*ndirections)+pt]*x.z;
	double dp1 = sinth*(-bvecs[pt]*sinph+bvecs[ndirections+pt]*cosph);
  	return(-2*bvals[pt]**_d*dp*dp1*exp(-bvals[pt]**_d*dp*dp));
}

//in diffmodel.cc
__device__ void fix_fsum_PVM_single(	//INPUT 
					bool m_include_f0, 
					int nfib,
					int nparams,
					//INPUT - OUTPUT){
					double *params)
{
  	double sum=0;
  	if (m_include_f0) 
    		sum=params[nparams-1];
  	for(int i=0;i<nfib;i++){
    		sum += params[2+(i*3)];
    		if(sum>=1){
			for(int j=i;j<nfib;j++)
				params[2+(j*3)]=FSMALL_gpu; 
			break;
		}
  	}
}

//in diffmodel.cc
__device__  void sort_PVM_single(int nfib,double* params)
{
	double temp_f, temp_th, temp_ph;
	// Order vector descending using f parameters as index
  	for(int i=1; i<(nfib); i++){ 
    		for(int j=0; j<(nfib-i); j++){ 
      			if (params[2+j*3] < params[2+(j+1)*3]){ 
        			temp_f = params[2+j*3];
				temp_th = params[2+j*3+1];
				temp_ph = params[2+j*3+2];
        			params[2+j*3] = params[2+(j+1)*3]; 
				params[2+j*3+1] = params[2+(j+1)*3+1]; 
				params[2+j*3+2] = params[2+(j+1)*3+2]; 
        			params[2+(j+1)*3] = temp_f; 
				params[2+(j+1)*3+1] = temp_th; 
				params[2+(j+1)*3+2] = temp_ph; 
      			} 
    		} 
  	} 
}

//cost function PVM_single
__device__ void cf_PVM_single(	//INPUT
				const double*		params,
				const double*		mdata,
				const double*		bvecs, 
				const double*		bvals,
				const int		ndirections,
				const int		nfib,
				const int 		nparams,
				const bool 		m_include_f0,
				const int		idSubVOX,
				double*			reduction,	//shared memory
				double* 		fs,		//shared memory
				double*			x,		//shared memory	
				double* 		_d,		//shared memory
				double* 		sumf,		//shared memory
				//OUTPUT
				double*			cfv)
{
	if(idSubVOX<nfib){
		int kk = 2+3*(idSubVOX);
		double sinth,costh,sinph,cosph;
		sincos(params[kk+1],&sinth,&costh);
		sincos(params[kk+2],&sinph,&cosph);
    		fs[idSubVOX] = x2f_gpu(params[kk]);
		x[idSubVOX*3] = sinth*cosph;
    		x[idSubVOX*3+1] = sinth*sinph;
    		x[idSubVOX*3+2] = costh;
  	}

	__syncthreads(); 	

	if(idSubVOX==0){
		*_d = abs(params[1]);
		*cfv = 0.0;
		*sumf=0;
		for(int k=0;k<nfib;k++) *sumf+= fs[k];
	}
	
	int ndir = ndirections/THREADS_BLOCK_FIT;
	if(idSubVOX<(ndirections%THREADS_BLOCK_FIT)) ndir++;
	
	double err;
	double3 x2;
	int dir_iter=idSubVOX;

	__syncthreads();

	reduction[idSubVOX]=0;
	for(int dir=0;dir<ndir;dir++){
		err = 0.0;
    		for(int k=0;k<nfib;k++){
			x2.x=x[k*3];
			x2.y=x[k*3+1];
			x2.z=x[k*3+2];	
			err += fs[k]*anisoterm_PVM_single(dir_iter,_d,x2,bvecs,bvals,ndirections); 
    		}
		if(m_include_f0){
			double temp_f0=x2f_gpu(params[nparams-1]);
			err= (params[0]*((temp_f0+(1-*sumf-temp_f0)*isoterm_PVM_single(dir_iter,_d,bvals))+err))-mdata[dir_iter];
		}else{
			err =  (params[0]*((1-*sumf)*isoterm_PVM_single(dir_iter,_d,bvals)+err))-mdata[dir_iter];
		}
		reduction[idSubVOX]+= err*err;  
		dir_iter+=THREADS_BLOCK_FIT;
  	}  
	__syncthreads();

	if(idSubVOX==0){
		for(int i=0;i<THREADS_BLOCK_FIT;i++){
			*cfv+=reduction[i];
		}
	}	
}

//gradient function PVM_single
__device__ void grad_PVM_single(	//INPUT
					const double*		params,
					const double*		mdata,
					const double*		bvecs, 
					const double*		bvals,
					const int 		ndirections,
					const int		nfib,
					const int 		nparams,
					const bool 		m_include_f0,
					const int		idSubVOX,		
					double*			reduction,	//shared memory
					double* 		fs,		//shared memory
					double*			x,		//shared memory
					double* 		_d,		//shared memory
					double* 		sumf,		//shared memory
					//OUTPUT
					double*			grad)
{
	if(idSubVOX<nfib){
		int kk = 2+3*(idSubVOX);
		double sinth,costh,sinph,cosph;
		sincos(params[kk+1],&sinth,&costh);
		sincos(params[kk+2],&sinph,&cosph);
    		fs[idSubVOX] = x2f_gpu(params[kk]);
    		x[idSubVOX*3] = sinth*cosph;
    		x[idSubVOX*3+1] = sinth*sinph;
    		x[idSubVOX*3+2] = costh;
  	}

	__syncthreads(); 

	if(idSubVOX==0){
		*sumf=0;
		for(int i=0;i<nfib;i++) *sumf+=fs[i];
		*_d = abs(params[1]);
		for (int p=0;p<nparams;p++) grad[p]=0;
	}

	int ndir = ndirections/THREADS_BLOCK_FIT;
	if(idSubVOX<(ndirections%THREADS_BLOCK_FIT)) ndir++;
	int max_dir = ndirections/THREADS_BLOCK_FIT;
	if(ndirections%THREADS_BLOCK_FIT) max_dir++;

	double J[MAXNPARAMS];
	double diff;
  	double sig;
	double3 xx;
	int dir_iter=idSubVOX;

	__syncthreads();

  	for(int dir=0;dir<max_dir;dir++){
		for (int p=0; p<nparams; p++) J[p]=0;
		if(dir<ndir){
    			sig = 0;
    			for(int k=0;k<nfib;k++){
      				int kk = 2+3*(k);
      				xx.x=x[k*3];
      				xx.y=x[k*3+1];
      				xx.z=x[k*3+2];			
				sig +=  fs[k]*anisoterm_PVM_single(dir_iter,_d,xx,bvecs,bvals,ndirections);
				J[1] +=  (params[1]>0?1.0:-1.0)*params[0]*fs[k]*anisoterm_d_PVM_single(dir_iter,_d,xx,bvecs,bvals,ndirections);
      				J[kk] = params[0]*(anisoterm_PVM_single(dir_iter,_d,xx,bvecs,bvals,ndirections)-isoterm_PVM_single(dir_iter,_d,bvals)) * two_pi_gpu*sign_gpu(params[kk])*1/(1+params[kk]*params[kk]);
      				J[kk+1] = params[0]*fs[k]*anisoterm_th_PVM_single(dir_iter,_d,xx,params[kk+1],params[kk+2],bvecs,bvals,ndirections);
      				J[kk+2] = params[0]*fs[k]*anisoterm_ph_PVM_single(dir_iter,_d,xx,params[kk+1],params[kk+2],bvecs,bvals,ndirections);
    			}

    			if(m_include_f0){
				double temp_f0=x2f_gpu(params[nparams-1]);
				J[nparams-1]= params[0]*(1-isoterm_PVM_single(dir_iter,_d,bvals))* two_pi_gpu*sign_gpu(params[nparams-1])*1/(1+params[nparams-1]*params[nparams-1]);
				sig= params[0]*((temp_f0+(1-*sumf-temp_f0)*isoterm_PVM_single(dir_iter,_d,bvals))+sig);
    				J[1] += (params[1]>0?1.0:-1.0)*params[0]*(1-*sumf-temp_f0)*isoterm_d_PVM_single(dir_iter,_d,bvals);
    			}else{
				sig = params[0]*((1-*sumf)*isoterm_PVM_single(dir_iter,_d,bvals)+sig);
				J[1] += (params[1]>0?1.0:-1.0)*params[0]*(1-*sumf)*isoterm_d_PVM_single(dir_iter,_d,bvals);
    			}
    			diff = sig - mdata[dir_iter];
    			J[0] = sig/params[0];
		}

		for (int p=0;p<nparams;p++){ 
			reduction[idSubVOX]=2*J[p]*diff;

			__syncthreads();
			if(idSubVOX==0){
				for(int i=0;i<THREADS_BLOCK_FIT;i++){
					grad[p] += reduction[i];
				}
			}
			__syncthreads(); 
		} 
		dir_iter+=THREADS_BLOCK_FIT;
  	}
}

//hessian function PVM_single
__device__ void hess_PVM_single(	//INPUT
					const double*		params,
					const double*		bvecs, 
					const double*		bvals,
					const int		ndirections,
					const int 		nfib,
					const int 		nparams,
					const bool 		m_include_f0,
					const int		idSubVOX,
					double*			reduction,	//shared memory					
					double* 		fs,		//shared memory
					double*			x,		//shared memory
					double* 		_d,		//shared memory
					double* 		sumf,		//shared memory
					//OUTPUT
					double*			hess)
{
	if(idSubVOX<nfib){
		int kk = 2+3*(idSubVOX);
		double sinth,costh,sinph,cosph;
		sincos(params[kk+1],&sinth,&costh);
		sincos(params[kk+2],&sinph,&cosph);
    		fs[idSubVOX] = x2f_gpu(params[kk]);
    		x[idSubVOX*3] = sinth*cosph;
    		x[idSubVOX*3+1] = sinth*sinph;
    		x[idSubVOX*3+2] = costh;
  	}

	__syncthreads(); 

	if(idSubVOX==0){
		*sumf=0;
		for(int i=0;i<nfib;i++) *sumf+=fs[i];
		*_d = abs(params[1]);
		for (int p=0;p<nparams;p++){
			for (int p2=0;p2<nparams;p2++){ 
				hess[p*nparams+p2] = 0;
			}
		}
	}

	int ndir = ndirections/THREADS_BLOCK_FIT;
	if(idSubVOX<(ndirections%THREADS_BLOCK_FIT)) ndir++;
	int max_dir = ndirections/THREADS_BLOCK_FIT;
	if(ndirections%THREADS_BLOCK_FIT) max_dir++;

	double J[MAXNPARAMS];
  	double sig;
	double3 xx;
	int dir_iter=idSubVOX; 

	__syncthreads(); 
	
  	for(int dir=0;dir<max_dir;dir++){
		for (int p=0; p<nparams; p++) J[p]=0;
		if(dir<ndir){
    			sig = 0;
    			for(int k=0;k<nfib;k++){
      				int kk = 2+3*(k);
      				xx.x=x[k*3];
      				xx.y=x[k*3+1];
      				xx.z=x[k*3+2];		
				sig += fs[k]*anisoterm_PVM_single(dir_iter,_d,xx,bvecs,bvals,ndirections);
      				J[1] += (params[1]>0?1.0:-1.0)*params[0]*fs[k]*anisoterm_d_PVM_single(dir_iter,_d,xx,bvecs,bvals,ndirections);
      				J[kk] = params[0]*(anisoterm_PVM_single(dir_iter,_d,xx,bvecs,bvals,ndirections)-isoterm_PVM_single(dir_iter,_d,bvals)) * two_pi_gpu*sign_gpu(params[kk])*1/(1+params[kk]*params[kk]);
		      		J[kk+1] = params[0]*fs[k]*anisoterm_th_PVM_single(dir_iter,_d,xx,params[kk+1],params[kk+2],bvecs,bvals,ndirections);
		      		J[kk+2] = params[0]*fs[k]*anisoterm_ph_PVM_single(dir_iter,_d,xx,params[kk+1],params[kk+2],bvecs,bvals,ndirections);
    			}	

    			if(m_include_f0){
				double temp_f0=x2f_gpu(params[nparams-1]);
				J[nparams-1]= params[0]*(1-isoterm_PVM_single(dir_iter,_d,bvals))* two_pi_gpu*sign_gpu(params[nparams-1])*1/(1+params[nparams-1]*params[nparams-1]);
				sig=params[0]*((temp_f0+(1-*sumf-temp_f0)*isoterm_PVM_single(dir_iter,_d,bvals))+sig);
    				J[1] += (params[1]>0?1.0:-1.0)*params[0]*(1-*sumf-temp_f0)*isoterm_d_PVM_single(dir_iter,_d,bvals);	
    			}else{
				sig = params[0]*((1-*sumf)*isoterm_PVM_single(dir_iter,_d,bvals)+sig);
	    			J[1] +=  (params[1]>0?1.0:-1.0)*params[0]*(1-*sumf)*isoterm_d_PVM_single(dir_iter,_d,bvals);
    			}   
    			J[0] = sig/params[0];
		}

		for (int p=0;p<nparams;p++){
			for (int p2=p;p2<nparams;p2++){ 

				reduction[idSubVOX]=2*(J[p]*J[p2]);
				__syncthreads();
				if(idSubVOX==0){
					for(int i=0;i<THREADS_BLOCK_FIT;i++){
						hess[p*nparams+p2] += reduction[i];
					}
				}
				__syncthreads(); 
			}
		}
		dir_iter+=THREADS_BLOCK_FIT;
  	}

	if(idSubVOX==0){
	  	for (int j=0; j<nparams; j++) {
	    		for (int i=j+1; i<nparams; i++) {
	     			hess[i*nparams+j]=hess[j*nparams+i];	
	    		}
	  	}
	}
}

//in diffmodel.cc
extern "C" __global__ void fit_PVM_single_kernel(	//INPUT
							const double* 		data, 
							const double* 		bvecs,
							const double* 		bvals, 
							const int 		nvox, 
							const int		ndirections,
							const int 		nfib, 
							const int		nparams,
							const bool 		m_include_f0, 
							//INPUT-OUTPUT
							double* 		params)
{
	int idSubVOX = threadIdx.x;
	int idVOX = blockIdx.x;
	int threadsBlock = blockDim.x;

	////////// DYNAMIC SHARED MEMORY ///////////
	extern __shared__ double shared[];
	double* reduction = (double*)shared;				//threadsBlock
	double* myparams = (double*) &reduction[threadsBlock];		//nparams
	double* grad = (double*) &myparams[nparams];			//nparams      
   	double* hess = (double*) &grad[nparams];			//nparams*nparams   
	double* step = (double*) &hess[nparams*nparams];		//nparams      
 	double* inverse = (double*) &step[nparams];			//nparams   
	double* pcf = (double*) &inverse[nparams];			//1   
	double* ncf = (double*) &pcf[1];				//1   
	double* lambda = (double*) &ncf[1];				//1  
	double* cftol = (double*) &lambda[1];				//1  
	double* ltol = (double*) &cftol[1];				//1  
	double* olambda = (double*) &ltol[1];				//1  

	double* fs = (double*) &olambda[1];				//nfib
  	double* x = (double*) &fs[nfib];				//nfib*3
	double* _d = (double*) &x[nfib*3];				//1
  	double* sumf = (double*) &_d[1];				//1

	double* C = (double*)&sumf[1];					//nparams*nparams;
	double* el =  (double*)&C[nparams*nparams];			//nparams

	int* indx = (int*)&el[nparams];					//nparams
	int* success = (int*) &indx[nparams];				//1
	int* end = (int*) &success[1];					//1    
	////////// DYNAMIC SHARED MEMORY ///////////

	if(idSubVOX<nparams){
		myparams[idSubVOX]=params[(idVOX*nparams)+idSubVOX];
	}

	__syncthreads();

	// do the fit
	levenberg_marquardt_PVM_single_gpu(&data[idVOX*ndirections],&bvecs[idVOX*3*ndirections],&bvals[idVOX*ndirections],ndirections,nfib,nparams,m_include_f0,idSubVOX,step,grad,hess,inverse, pcf,ncf,lambda,cftol,ltol,olambda,success,end,reduction,fs,x,_d,sumf,C,el,indx,myparams);

	__syncthreads();
	
  	// finalise parameters
	//m_s0 in myparams[0] 	m_d in myparams[1] 	m_f-m_th-m_ph in myparams[2,3,4,5, etc..]   	m_f0 in myparams[nparams-1]

	if(idSubVOX==0){  	
  		myparams[1] = abs(myparams[1]); 
  		for(int k=1;k<=nfib;k++){
    			int kk = 2 + 3*(k-1);
    			myparams[kk] = x2f_gpu(myparams[kk]);
  		}
  		if(m_include_f0)
    			myparams[nparams-1]=x2f_gpu(myparams[nparams-1]);

  		sort_PVM_single(nfib,myparams);
  		fix_fsum_PVM_single(m_include_f0,nfib,nparams,myparams);
	}
	__syncthreads();

	if(idSubVOX<nparams){
		params[idVOX*nparams+idSubVOX]=myparams[idSubVOX];	
	}
}

//in diffmodel.cc
extern "C" __global__ void get_residuals_PVM_single_kernel(	//INPUT
								const double* 		data, 
								const double* 		params,
								const double* 		bvecs, 
								const double* 		bvals, 
								const int 		nvox, 
								const int		ndirections,
								const int 		nfib, 
								const int		nparams,
								const bool 		m_include_f0,
								const bool* 		includes_f0,
								//OUTPUT
								double*			residuals)
{
	int idSubVOX = threadIdx.x;
	int idVOX = blockIdx.x;
	int threadsBlock = blockDim.x;

	////////// DYNAMIC SHARED MEMORY ///////////
	extern __shared__ double shared[];
	double* myparams = (double*) shared;			//nparams
	double* fs = (double*) &myparams[nparams];		//nfib
  	double* x = (double*) &fs[nfib];			//nfib*3
	double* _d = (double*) &x[nfib*3];			//1
  	double* sumf = (double*) &_d[1];			//1
	int* my_include_f0 = (int*) &sumf[1];			//1	
	////////// DYNAMIC SHARED MEMORY ///////////

	double val;
	double predicted_signal;
	double mydata;

	if(idSubVOX==0){
		*my_include_f0 = includes_f0[idVOX];

		//m_s0-myparams[0]  m_d-myparams[1]  m_f-m_th-m_ph-myparams[2,3,4,5 etc..]  m_f0-myparams[nparams-1]

		myparams[0]=params[(idVOX*nparams)+0];
		myparams[1]=params[(idVOX*nparams)+1];

  		if (*my_include_f0)
    			myparams[nparams-1]=f2x_gpu(params[(idVOX*nparams)+nparams-1]);
	}

	if(idSubVOX<nfib){
		int kk = 2+3*idSubVOX;
		double sinth,costh,sinph,cosph;

		myparams[kk]   = f2x_gpu(params[(idVOX*nparams)+kk]);
    		myparams[kk+1] = params[(idVOX*nparams)+kk+1];
    		myparams[kk+2] = params[(idVOX*nparams)+kk+2];

		sincos(myparams[kk+1],&sinth,&costh);
		sincos(myparams[kk+2],&sinph,&cosph);

    		fs[idSubVOX] = x2f_gpu(myparams[kk]);
    		x[idSubVOX*3] = sinth*cosph;
    		x[idSubVOX*3+1] = sinth*sinph;
    		x[idSubVOX*3+2] = costh;
  	}

	__syncthreads(); 

	if(idSubVOX==0){
		*sumf=0;
		for(int i=0;i<nfib;i++) *sumf+=fs[i];
		*_d = abs(myparams[1]);
	}
  	
  	int ndir = ndirections/threadsBlock;
	if(idSubVOX<(ndirections%threadsBlock)) ndir++;
	
	double3 x2;
	int dir_iter=idSubVOX; 

	__syncthreads();

	for(int dir=0;dir<ndir;dir++){
		mydata = data[(idVOX*ndirections)+dir_iter];
  		predicted_signal=0;	//pred = 0;
    		val = 0.0;
    		for(int k=0;k<nfib;k++){
			x2.x=x[k*3];
			x2.y=x[k*3+1];
			x2.z=x[k*3+2];	 
      			val += fs[k]*anisoterm_PVM_single(dir_iter,_d,x2,&bvecs[idVOX*3*ndirections],&bvals[idVOX*ndirections],ndirections);
    		}	
    		if (*my_include_f0){
      			double temp_f0=x2f_gpu(myparams[nparams-1]);
      			predicted_signal = myparams[0]*(temp_f0+(1-*sumf-temp_f0)*isoterm_PVM_single(dir_iter,_d,&bvals[idVOX*ndirections])+val);
    		}else{
      			predicted_signal = myparams[0]*((1-*sumf)*isoterm_PVM_single(dir_iter,_d,&bvals[idVOX*ndirections])+val); 
		}
	
		//residuals=m_data-predicted_signal;
		residuals[idVOX*ndirections+dir_iter]= mydata - predicted_signal;

		dir_iter+=threadsBlock;
  	}
}

