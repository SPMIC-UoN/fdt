#include "hip/hip_runtime.h"
#include "diffmodels_utils.h"
#include "levenberg_marquardt.cu"
#include "options.h"

//#include <fstream>

/////////////////////////////////////
/////////////////////////////////////
/// 	    PVM_single		  /// 
/////////////////////////////////////
/////////////////////////////////////

__device__ 
inline double isoterm_PVM_single(const int pt,const double _d,const double *bvals){
  	return exp(double(-bvals[pt]*_d));
}

__device__ 
inline double isoterm_d_PVM_single(const int pt,const double _d,const double *bvals){
  	return (-bvals[pt]*exp(double(-bvals[pt]*_d)));
}

__device__ 
inline double anisoterm_PVM_single(const int pt,const double _d,const double3 x, const double *bvecs, const double *bvals){
	double dp = bvecs[pt]*x.x+bvecs[NDIRECTIONS+pt]*x.y+bvecs[(2*NDIRECTIONS)+pt]*x.z;
	return exp(double(-bvals[pt]*_d*dp*dp));
}

__device__ 
inline double anisoterm_d_PVM_single(const int pt,const double _d,const double3 x,const double *bvecs, const double *bvals){
	double dp = bvecs[pt]*x.x+bvecs[NDIRECTIONS+pt]*x.y+bvecs[(2*NDIRECTIONS)+pt]*x.z;
  	return(-bvals[pt]*dp*dp*exp(double(-bvals[pt]*_d*dp*dp)));
}

__device__ 
inline double anisoterm_th_PVM_single(const int pt,const double _d,const double3 x, const double _th,const double _ph,const double *bvecs, const double *bvals){

	double dp = bvecs[pt]*x.x+bvecs[NDIRECTIONS+pt]*x.y+bvecs[(2*NDIRECTIONS)+pt]*x.z;
	double dp1 = (cos(double(_th))*(bvecs[pt]*cos(double(_ph))+bvecs[NDIRECTIONS+pt]*sin(double(_ph)))-bvecs[(2*NDIRECTIONS)+pt]*sin(double(_th)));
  	return(-2*bvals[pt]*_d*dp*dp1*exp(double(-bvals[pt]*_d*dp*dp)));
}

__device__ 
inline double anisoterm_ph_PVM_single(const int pt,const double _d,const double3 x, const double _th,const double _ph,const double *bvecs, const double *bvals){
  	double dp = bvecs[pt]*x.x+bvecs[NDIRECTIONS+pt]*x.y+bvecs[(2*NDIRECTIONS)+pt]*x.z;
	double dp1 = sin(double(_th))*(-bvecs[pt]*sin(double(_ph))+bvecs[NDIRECTIONS+pt]*cos(double(_ph)));
  	return(-2*bvals[pt]*_d*dp*dp1*exp(double(-bvals[pt]*_d*dp*dp)));
}


//in diffmodel.cc
__device__ void fix_fsum_PVM_single(	//INPUT 
					bool m_include_f0, 
					int nfib,
					int nparams,
					//INPUT - OUTPUT){
					double *params)
{
  	double sum=0;
  	if (m_include_f0) 
    		sum=params[nparams-1];
  	for(int i=0;i<nfib;i++){
    		sum += params[2+(i*3)];
    		if(sum>=1){
			for(int j=i;j<nfib;j++)
				params[2+(j*3)]=FSMALL_gpu; 
			break;
		}
  	}
}



//in diffmodel.cc
__device__  void sort_PVM_single(int nfib,int nparams,double* params)
{
	double temp_f, temp_th, temp_ph;
	// Order vector descending using f parameters as index
  	for(int i=1; i<(nfib); i++){ 
    		for(int j=0; j<(nfib-i); j++){ 
      			if (params[2+j*3] < params[2+i*3]){ 
        			temp_f = params[2+j*3];
				temp_th = params[2+j*3+1];
				temp_ph = params[2+j*3+2];
        			params[2+j*3] = params[2+i*3]; 
				params[2+j*3+1] = params[2+i*3+1]; 
				params[2+j*3+2] = params[2+i*3+2]; 
        			params[2+i*3] = temp_f; 
				params[2+i*3+1] = temp_th; 
				params[2+i*3+2] = temp_ph; 
      			} 
    		} 
  	} 
}


//in diffmodels.cc -- for calculate residuals
__device__ void  forwardModel_PVM_single(	//INPUT
						const double* 		p,
						const double*		bvecs, 
						const double*		bvals,
						const int		nfib,
						const int 		nparams,
						const bool 		m_include_f0,
						//OUTPUT
						double*		 	predicted_signal)
{
  	for(int i=0;i<NDIRECTIONS;i++){
		predicted_signal[i]=0;		//pred = 0;
	}
  	double val;
  	double _d = abs(p[1]);
  	////////////////////////////////////
  	double fs[NFIBRES];
  	double x[NFIBRES*3];	
  	double sumf=0;
	double3 x2;
  	for(int k=0;k<nfib;k++){
    		int kk = 2+3*k;
	    	fs[k] = x2f_gpu(p[kk]);
	    	sumf += fs[k];
		x[k*3] = sin(p[kk+1])*cos(p[kk+2]);
    		x[k*3+1] = sin(p[kk+1])*sin(p[kk+2]);
    		x[k*3+2] = cos(p[kk+1]);
  	}
  	////////////////////////////////////
  	for(int i=0;i<NDIRECTIONS;i++){
    		val = 0.0;
    		for(int k=0;k<nfib;k++){
			x2.x=x[k*3];
			x2.y=x[k*3+1];
			x2.z=x[k*3+2];	 
      			val += fs[k]*anisoterm_PVM_single(i,_d,x2,bvecs,bvals);
    		}	
    		if (m_include_f0){
      			double temp_f0=x2f_gpu(p[nparams-1]);
      			predicted_signal[i] = p[0]*(temp_f0+(1-sumf-temp_f0)*isoterm_PVM_single(i,_d,bvals)+val);
    		} 
    		else
      			predicted_signal[i] = p[0]*((1-sumf)*isoterm_PVM_single(i,_d,bvals)+val); 
  	}
}


//in diffmodels.cc -- for calculate residuals
__device__ void get_prediction_PVM_single(	//INPUT
						const double*	params,
						const double*	bvecs, 
						const double*	bvals,
						const int 	nfib,
						const int 	nparams,
						const bool 	m_include_f0,
						//OUTPUT
						double* 	predicted_signal)
{
	//m_s0-myparams[0] 	m_d-myparams[1] 	m_d_std-myparams[2]		m_f-m_th-m_ph-myparams[3,4,5,6 etc..]   	m_f0-myparams[nparams-1]
  	double p[NPARAMS];
  	p[0] = params[0];
  	p[1] = params[1];		
  	for(int k=0;k<nfib;k++){
    		int kk = 2+3*k;
    		p[kk]   = f2x_gpu(params[kk]);
    		p[kk+1] = params[kk+1];
    		p[kk+2] = params[kk+2];
  	}
  	if (m_include_f0)
    		p[nparams-1]=f2x_gpu(params[nparams-1]);
  	forwardModel_PVM_single(p,bvecs,bvals,nfib,nparams,m_include_f0,predicted_signal);
}


//cost function PVM_single
__device__ double cf_PVM_single(	//INPUT
					const double*		params,
					const double*		mdata,
					const double*		bvecs, 
					const double*		bvals,
					const int 		nparams,
					const bool 		m_include_f0)
{
	double cfv = 0.0;
  	double err;
	double _d = abs(params[1]);
	double fs[NFIBRES];    
	double x[NFIBRES*3];	
	double sumf=0;
	double3 x2;

	for(int k=0;k<NFIBRES;k++){
    		int kk = 2+3*(k);
    		fs[k] = x2f_gpu(params[kk]);
    		sumf += fs[k];
    		
    		x[k*3] = sin(params[kk+1])*cos(params[kk+2]);
    		x[k*3+1] = sin(params[kk+1])*sin(params[kk+2]);
    		x[k*3+2] = cos(params[kk+1]);
  	}
	
	for(int i=0;i<NDIRECTIONS;i++){
		err = 0.0;
    		for(int k=0;k<NFIBRES;k++){
			x2.x=x[k*3];
			x2.y=x[k*3+1];
			x2.z=x[k*3+2];	
			err += fs[k]*anisoterm_PVM_single(i,_d,x2,bvecs,bvals); 
    		}
		if(m_include_f0){
			double temp_f0=x2f_gpu(params[nparams-1]);
			err= (params[0]*((temp_f0+(1-sumf-temp_f0)*isoterm_PVM_single(i,_d,bvals))+err))-mdata[i];
		}else{
			err =  (params[0]*((1-sumf)*isoterm_PVM_single(i,_d,bvals)+err))-mdata[i];
		}
		cfv += err*err;  
  	}  
	return(cfv);
}

//gradient function PVM_single
__device__ void grad_PVM_single(	//INPUT
					const double*		params,
					const double*		mdata,
					const double*		bvecs, 
					const double*		bvals,
					const int 		nparams,
					const bool 		m_include_f0,
					//OUTPUT
					double*			grad)
{
  	double _d = abs(params[1]);
  	double fs[NFIBRES];
  	double x[NFIBRES*3];	
  	double3 xx;		
  	double sumf=0;

  	for(int k=0;k<NFIBRES;k++){
    		int kk = 2+3*(k);
    		fs[k] = x2f_gpu(params[kk]);
    		sumf += fs[k];
    		x[k*3] = sin(params[kk+1])*cos(params[kk+2]);
    		x[k*3+1] = sin(params[kk+1])*sin(params[kk+2]);
    		x[k*3+2] = cos(params[kk+1]);
  	}
 
  	double J[NPARAMS];
  	double diff;
  	double sig;

	for (int p=0;p<nparams;p++) grad[p]=0;

  	for(int i=0;i<NDIRECTIONS;i++){
    		sig = 0;
    		for(int a=0;a<nparams;a++) J[a]=0;
    		for(int k=0;k<NFIBRES;k++){
      			int kk = 2+3*(k);
      			xx.x=x[k*3];
      			xx.y=x[k*3+1];
      			xx.z=x[k*3+2];			
			sig +=  fs[k]*anisoterm_PVM_single(i,_d,xx,bvecs,bvals);
			J[1] +=  (params[1]>0?1.0:-1.0)*params[0]*fs[k]*anisoterm_d_PVM_single(i,_d,xx,bvecs,bvals);
      			J[kk] = params[0]*(anisoterm_PVM_single(i,_d,xx,bvecs,bvals)-isoterm_PVM_single(i,_d,bvals)) * two_pi_gpu*sign_gpu(params[kk])*1/(1+params[kk]*params[kk]);
      			J[kk+1] = params[0]*fs[k]*anisoterm_th_PVM_single(i,_d,xx,params[kk+1],params[kk+2],bvecs,bvals);
      			J[kk+2] = params[0]*fs[k]*anisoterm_ph_PVM_single(i,_d,xx,params[kk+1],params[kk+2],bvecs,bvals);
    		}

    		if(m_include_f0){
			double temp_f0=x2f_gpu(params[nparams-1]);
			J[nparams-1]= params[0]*(1-isoterm_PVM_single(i,_d,bvals))* two_pi_gpu*sign_gpu(params[nparams-1])*1/(1+params[nparams-1]*params[nparams-1]);
			sig= params[0]*((temp_f0+(1-sumf-temp_f0)*isoterm_PVM_single(i,_d,bvals))+sig);
    			J[1] += (params[1]>0?1.0:-1.0)*params[0]*(1-sumf-temp_f0)*isoterm_d_PVM_single(i,_d,bvals);
    		}else{
			sig = params[0]*((1-sumf)*isoterm_PVM_single(i,_d,bvals)+sig);
			J[1] += (params[1]>0?1.0:-1.0)*params[0]*(1-sumf)*isoterm_d_PVM_single(i,_d,bvals);
    		}
    		diff = sig - mdata[i];
    		J[0] = sig/params[0];

		for (int p=0;p<nparams;p++) grad[p] += 2*J[p]*diff; 
  	}
}

//hessian function PVM_single
__device__ void hess_PVM_single(	//INPUT
					const double*		params,
					const double*		bvecs, 
					const double*		bvals,
					const int 		nparams,
					const bool 		m_include_f0,
					double*			hess)
{
  	double _d = abs(params[1]);
  	double fs[NFIBRES];
  	double x[NFIBRES*3];	
  	double3 xx;
  	double sumf=0;

  	for(int k=0;k<NFIBRES;k++){
    		int kk = 2+3*(k);
    		fs[k] = x2f_gpu(params[kk]);
    		sumf += fs[k];
    		x[k*3] = sin(params[kk+1])*cos(params[kk+2]);
    		x[k*3+1] = sin(params[kk+1])*sin(params[kk+2]);
    		x[k*3+2] = cos(params[kk+1]);
  	}
 
  	double J[NPARAMS];
  	double sig;

	for (int p=0;p<nparams;p++){
		for (int p2=0;p2<nparams;p2++){ 
			hess[p*nparams+p2] = 0;
		}
	}

  	for(int i=0;i<NDIRECTIONS;i++){
    		sig = 0;
    		for(int a=0;a<nparams;a++) J[a]=0;
    		for(int k=0;k<NFIBRES;k++){
      			int kk = 2+3*(k);
      			xx.x=x[k*3];
      			xx.y=x[k*3+1];
      			xx.z=x[k*3+2];		
			sig += fs[k]*anisoterm_PVM_single(i,_d,xx,bvecs,bvals);
      			J[1] += (params[1]>0?1.0:-1.0)*params[0]*fs[k]*anisoterm_d_PVM_single(i,_d,xx,bvecs,bvals);
      			J[kk] = params[0]*(anisoterm_PVM_single(i,_d,xx,bvecs,bvals)-isoterm_PVM_single(i,_d,bvals)) * two_pi_gpu*sign_gpu(params[kk])*1/(1+params[kk]*params[kk]);
		      	J[kk+1] = params[0]*fs[k]*anisoterm_th_PVM_single(i,_d,xx,params[kk+1],params[kk+2],bvecs,bvals);
		      	J[kk+2] = params[0]*fs[k]*anisoterm_ph_PVM_single(i,_d,xx,params[kk+1],params[kk+2],bvecs,bvals);
    		}

    		if(m_include_f0){
			double temp_f0=x2f_gpu(params[nparams-1]);
			J[nparams-1]= params[0]*(1-isoterm_PVM_single(i,_d,bvals))* two_pi_gpu*sign_gpu(params[nparams-1])*1/(1+params[nparams-1]*params[nparams-1]);
			sig=params[0]*((temp_f0+(1-sumf-temp_f0)*isoterm_PVM_single(i,_d,bvals))+sig);
    			J[1] += (params[1]>0?1.0:-1.0)*params[0]*(1-sumf-temp_f0)*isoterm_d_PVM_single(i,_d,bvals);	
    		}else{
			sig = params[0]*((1-sumf)*isoterm_PVM_single(i,_d,bvals)+sig);
	    		J[1] +=  (params[1]>0?1.0:-1.0)*params[0]*(1-sumf)*isoterm_d_PVM_single(i,_d,bvals);
    		}   
    		J[0] = sig/params[0];

		for (int p=0;p<nparams;p++){
			for (int p2=p;p2<nparams;p2++){ 
				hess[p*nparams+p2] += 2*(J[p]*J[p2]);
			}
		}
  	}

  	for (int j=0; j<nparams; j++) {
    		for (int i=j+1; i<nparams; i++) {
     			hess[i*nparams+j]=hess[j*nparams+i];	
    		}
  	}
}

//in diffmodel.cc
extern "C" __global__ void fit_PVM_single_kernel(	//INPUT
							const double* 		data, 
							const double* 		bvecs,
							const double* 		bvals, 
							const int 		nvox, 
							const int 		nfib, 
							const bool 		m_include_f0, 
							//INPUT-OUTPUT
							double* 		params)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;	
   	if (id >=nvox) { return; }	

	int nparams;
	if (m_include_f0)
      		nparams = nfib*3 + 3; 
    	else
      		nparams = nfib*3 + 2;

	double myparams[NPARAMS];
   	double mydata[NDIRECTIONS];

	for(int i=0;i<nparams;i++){
		myparams[i]=params[(id*nparams)+i];
   	}
	
   	for(int i=0;i<NDIRECTIONS;i++){
		mydata[i]=data[(id*NDIRECTIONS)+i];
   	}

	// do the fit
	levenberg_marquardt_PVM_single_gpu(mydata, &bvecs[id*3*NDIRECTIONS], &bvals[id*NDIRECTIONS], nparams, m_include_f0,  myparams);
	
  	// finalise parameters
	//m_s0 in myparams[0] 	m_d in myparams[1] 	m_f-m_th-m_ph in myparams[2,3,4,5, etc..]   	m_f0 in myparams[nparams-1]
  			
  	myparams[1] = abs(myparams[1]); 
  	for(int k=1;k<=nfib;k++){
    		int kk = 2 + 3*(k-1);
    		myparams[kk]  = x2f_gpu(myparams[kk]);
  	}
  	if (m_include_f0)
    		myparams[nparams-1]=x2f_gpu(myparams[nparams-1]);

  	sort_PVM_single(nfib,nparams,myparams);
  	fix_fsum_PVM_single(m_include_f0,nfib,nparams,myparams);

	for(int i=0;i<nparams;i++){
		params[id*nparams+i]=myparams[i];	
		//printf("PARAM[%i]: %.20f\n",i,myparams[i]);
	}
}

//in diffmodel.cc
extern "C" __global__ void get_residuals_PVM_single_kernel(	//INPUT
								const double* 		data, 
								const double* 		params,
								const double* 		bvecs, 
								const double* 		bvals, 
								const int 		nvox, 
								const int 		nfib, 
								const bool 		m_include_f0,
								const bool* 		includes_f0,
								//OUTPUT
								double*			residuals)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;	
   	if (id >=nvox) { return; }	

	int nparams;
	if (m_include_f0)
      		nparams = nfib*3 + 3; 
    	else
      		nparams = nfib*3 + 2;

	bool my_include_f0 = includes_f0[id];

	double myparams[NPARAMS];
   	double mydata[NDIRECTIONS];

	for(int i=0;i<nparams;i++){
		myparams[i]=params[(id*nparams)+i];
   	}
	
   	for(int i=0;i<NDIRECTIONS;i++){
		mydata[i]=data[(id*NDIRECTIONS)+i];
   	}

	double predicted_signal[NDIRECTIONS];

	get_prediction_PVM_single(myparams, &bvecs[id*3*NDIRECTIONS], &bvals[id*NDIRECTIONS], nfib, nparams, my_include_f0, predicted_signal);

	for(int i=0;i<NDIRECTIONS;i++){		//residuals=m_data-predicted_signal;
		residuals[id*NDIRECTIONS+i]= mydata[i] - predicted_signal[i];
	}
}

