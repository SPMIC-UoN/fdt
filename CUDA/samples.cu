#include "hip/hip_runtime.h"
/*  samples.cu

    Tim Behrens, Saad Jbabdi, Stam Sotiropoulos, Moises Hernandez  - FMRIB Image Analysis Group

    Copyright (C) 2005 University of Oxford  */

/*  CCOPYRIGHT  */

#include "newmat.h"
#include "newimage/newimageall.h"
#include "xfibresoptions.h"
#include "samples.h"

using namespace Xfibres;

////////////////////////////////////////////
//       MCMC SAMPLE STORAGE
////////////////////////////////////////////

Samples::Samples(int nvoxels,int nmeasures):
opts(xfibresOptions::getInstance()){

	/////////////// GPU version /////////////////////
    	m_sum_d=new float[nvoxels];
    	m_sum_S0=new float[nvoxels];
    	for(int i=0;i<nvoxels;i++){
    		m_sum_d[i]=0;
     		m_sum_S0[i]=0;
    	}
    	m_vec=new ColumnVector[nvoxels];
    	m_dyad=new vector<SymmetricMatrix>[nvoxels];
    	m_sum_f=new vector<float> [nvoxels];
    	m_sum_lam=new vector<float> [nvoxels];	
    	////////////////////////////////////////////////
    
    	//m_beenhere=m_vol2matrixkey*0;
    	int count=0;
    	int nsamples=0;
    
    	for(int i=0;i<opts.njumps.value();i++){
      		count++;
      		if(count==opts.sampleevery.value()){
			count=0;nsamples++;
      		}
    	}
 
    	m_dsamples.ReSize(nsamples,nvoxels);
    	m_dsamples=0;
    	m_S0samples.ReSize(nsamples,nvoxels);
    	m_S0samples=0;
    	m_lik_energy.ReSize(nsamples,nvoxels);
    
    	m_mean_dsamples.ReSize(nvoxels);
    	m_mean_dsamples=0;
    	m_mean_S0samples.ReSize(nvoxels);
    	m_mean_S0samples=0;
    	Matrix tmpvecs(3,nvoxels);
    	tmpvecs=0;
    	//m_sum_d=0;  changed GPU version
    	//m_sum_S0=0;  changed GPU version

    	if(opts.modelnum.value()==2){
      		m_d_stdsamples.ReSize(nsamples,nvoxels);
      		m_d_stdsamples=0;
      		m_mean_d_stdsamples.ReSize(nvoxels);
      		m_mean_d_stdsamples=0;
      		//m_sum_d_std=0;  changed GPU version

      		/////////////// GPU version /////////////////////
      		m_sum_d_std=new float[nvoxels];
      		for(int i=0;i<nvoxels;i++){
      			m_sum_d_std[i]=0;
      		}
      		////////////////////////////////////////////////
    	}

    	if (opts.f0.value()){
      		m_f0samples.ReSize(nsamples,nvoxels);
      		m_f0samples=0;
      		m_mean_f0samples.ReSize(nvoxels);
      		m_mean_f0samples=0;
      		//m_sum_f0=0;  changed GPU version

     	 	/////////////// GPU version /////////////////////
      		m_sum_f0=new float[nvoxels];
      		for(int i=0;i<nvoxels;i++)
      			m_sum_f0[i]=0;
      		////////////////////////////////////////////////
    	}

    	if (opts.rician.value()){
      		m_mean_tausamples.ReSize(nvoxels);
      		m_mean_tausamples=0;
      		//m_sum_tau=0;  changed GPU version

      		/////////////// GPU version /////////////////////
      		m_sum_tau=new float[nvoxels];
      		for(int i=0;i<nvoxels;i++)
      			m_sum_tau[i]=0;
      		////////////////////////////////////////////////
    	}

    	SymmetricMatrix tmpdyad(3);
    	tmpdyad=0;
    	m_nsamps=nsamples;
    	//m_vec.ReSize(3);  changed GPU version

    	/////////////// GPU version /////////////////////
    	for(int i=0;i<nvoxels;i++){ 
        	m_vec[i].ReSize(3);
		for(int f=0;f<opts.nfibres.value();f++){
			m_dyad[i].push_back(tmpdyad);
                	m_sum_f[i].push_back(0);
                	m_sum_lam[i].push_back(0);
        	}
    	}	 
    	////////////////////////////////////////////////

    	for(int f=0;f<opts.nfibres.value();f++){
      		m_thsamples.push_back(m_S0samples);
      		m_phsamples.push_back(m_S0samples);
      		m_fsamples.push_back(m_S0samples);
      		m_lamsamples.push_back(m_S0samples);

      		m_dyadic_vectors.push_back(tmpvecs);
      		m_mean_fsamples.push_back(m_mean_S0samples);
      		m_mean_lamsamples.push_back(m_mean_S0samples);

      		//m_sum_lam.push_back(0);  changed GPU version
      		//m_sum_f.push_back(0);  changed GPU version
      		//m_dyad.push_back(tmpdyad);  changed GPU version
    	}
}

	//new version for GPU
void Samples::record(float rd,float rf0,float rtau,float rdstd,float rs0,float rlikelihood_energy, float *rth,float *rph, float *rf, int vox, int samp){
    	m_dsamples(samp,vox)=rd;
    	m_sum_d[vox-1]+=rd;

    	if(opts.modelnum.value()==2){
		m_d_stdsamples(samp,vox)=rdstd;
      		m_sum_d_std[vox-1]+=rdstd;
    	}
    	if (opts.f0.value()){
     		m_f0samples(samp,vox)=rf0;
      		m_sum_f0[vox-1]+=rf0;
    	}
    	if (opts.rician.value()){
      		m_sum_tau[vox-1]+=rtau;
    	}

    	m_S0samples(samp,vox)=rs0;
    	m_sum_S0[vox-1]+=rs0;
    	m_lik_energy(samp,vox)=rlikelihood_energy;
    	for(int f=0;f<opts.nfibres.value();f++){
      		float th=rth[f];
      		float ph=rph[f];
      		m_thsamples[f](samp,vox)=th;
      		m_phsamples[f](samp,vox)=ph;
      		m_fsamples[f](samp,vox)=rf[f];
     	 	//for means
      		m_vec[vox-1] << sin(th)*cos(ph) << sin(th)*sin(ph)<<cos(th) ;

      		m_dyad[vox-1][f] << m_dyad[vox-1][f]+m_vec[vox-1]*m_vec[vox-1].t();
      		m_sum_f[vox-1][f]+=rf[f];
      		m_sum_lam[vox-1][f]+=0;
    	}
}  

//new version for GPU
 void Samples::finish_voxel(int vox){
    	m_mean_dsamples(vox)=m_sum_d[vox-1]/m_nsamps;

    	if(opts.modelnum.value()==2)
      		m_mean_d_stdsamples(vox)=m_sum_d_std[vox-1]/m_nsamps;
    	if(opts.f0.value())
      		m_mean_f0samples(vox)=m_sum_f0[vox-1]/m_nsamps;
    	if(opts.rician.value())
      		m_mean_tausamples(vox)=m_sum_tau[vox-1]/m_nsamps;

    	m_mean_S0samples(vox)=m_sum_S0[vox-1]/m_nsamps;

    	m_sum_d[vox-1]=0;
    	m_sum_S0[vox-1]=0;
   
    	if(opts.rician.value())
    		m_sum_tau[vox-1]=0;

    	if(opts.modelnum.value()==2)
      		m_sum_d_std[vox-1]=0;
    	if (opts.f0.value())
      		m_sum_f0[vox-1]=0;

    	DiagonalMatrix dyad_D; //eigenvalues
    	Matrix dyad_V; //eigenvectors
    	int nfibs=0;
    	for(int f=0;f<opts.nfibres.value();f++){
      		EigenValues(m_dyad[vox-1][f],dyad_D,dyad_V);
      		int maxeig;
      		if(dyad_D(1)>dyad_D(2)){
			if(dyad_D(1)>dyad_D(3)) maxeig=1;
			else maxeig=3;
      		}
      		else{
			if(dyad_D(2)>dyad_D(3)) maxeig=2;
			else maxeig=3;
      		}
      		m_dyadic_vectors[f](1,vox)=dyad_V(1,maxeig);
      		m_dyadic_vectors[f](2,vox)=dyad_V(2,maxeig);
      		m_dyadic_vectors[f](3,vox)=dyad_V(3,maxeig);
      
      		if((m_sum_f[vox-1][f]/m_nsamps)>0.05){
			nfibs++;
      		}
      		m_mean_fsamples[f](vox)=m_sum_f[vox-1][f]/m_nsamps;
      		m_mean_lamsamples[f](vox)=m_sum_lam[vox-1][f]/m_nsamps;
      
      		m_dyad[vox-1][f]=0;
      		m_sum_f[vox-1][f]=0;
      		m_sum_lam[vox-1][f]=0;
    	}
    	//m_beenhere(int(m_matrix2volkey(vox,1)),int(m_matrix2volkey(vox,2)),int(m_matrix2volkey(vox,3)))=nfibs;
}

void save_part(RowVector data, string name, int idpart){
	Log& logger = LogSingleton::getInstance();
    	xfibresOptions& opts = xfibresOptions::getInstance();

	int nvox = data.Ncols();

	string file_name;

	file_name = logger.appendDir(name+"_"+num2str(idpart));
	ofstream out;
	out.open(file_name.data(), ios::out | ios::binary);
	out.write((char*)&data(1),nvox*sizeof(Real));
	out.close();
}

void save_part(Matrix data, string name, int idpart){
	Log& logger = LogSingleton::getInstance();
    	xfibresOptions& opts = xfibresOptions::getInstance();

	int nvox = data.Ncols();
	int nsamples = data.Nrows();

	string file_name;

	file_name = logger.appendDir(name+"_"+num2str(idpart));
	ofstream out;
	out.open(file_name.data(), ios::out | ios::binary);
	out.write((char*)&data(1,1),nvox*nsamples*sizeof(Real));
	out.close();
}
  
void Samples::save(int idpart){

	vector<Matrix> thsamples_out=m_thsamples;
	vector<Matrix> phsamples_out=m_phsamples;
	vector<Matrix> fsamples_out=m_fsamples;
	vector<Matrix> lamsamples_out=m_lamsamples;
    
    	vector<Matrix> dyadic_vectors_out=m_dyadic_vectors;
    	vector<Matrix> mean_fsamples_out;
    	for(unsigned int f=0;f<m_mean_fsamples.size();f++)
      		mean_fsamples_out.push_back(m_mean_fsamples[f]);

    	Log& logger = LogSingleton::getInstance();
    	if(opts.modelnum.value()==1){
		save_part(m_mean_dsamples,"mean_dsamples",idpart);
    	}
    	else if(opts.modelnum.value()==2){
		save_part(m_mean_dsamples,"mean_dsamples",idpart);
		save_part(m_mean_d_stdsamples,"mean_d_stdsamples",idpart);
		//save_part(m_dsamples,"m_d_stdsamples",idpart);
		//save_part(m_d_stdsamples,"d_stdsamples",idpart);
    	}
    	if (opts.f0.value()){
		save_part(m_mean_f0samples,"mean_f0samples",idpart);
		//save_part(m_f0samples,"f0samples",idpart);
    	}
    	if (opts.rician.value()){
		save_part(m_mean_tausamples,"mean_tausamples",idpart);	
    	}

	save_part(m_mean_S0samples,"mean_S0samples",idpart);
	
    	//Sort the output based on mean_fsamples
    	// 
    	vector<Matrix> sumf;
    	for(int f=0;f<opts.nfibres.value();f++){
      		Matrix tmp=sum(m_fsamples[f],1);
      		sumf.push_back(tmp);
    	}  
    	for(int vox=1;vox<=m_dsamples.Ncols();vox++){
      		vector<pair<float,int> > sfs;
      		pair<float,int> ftmp;
      
      		for(int f=0;f<opts.nfibres.value();f++){
			ftmp.first=sumf[f](1,vox);
			ftmp.second=f;
			sfs.push_back(ftmp);
      		}
      		sort(sfs.begin(),sfs.end());
      
      		for(int samp=1;samp<=m_dsamples.Nrows();samp++){
			for(int f=0;f<opts.nfibres.value();f++){;
	  			thsamples_out[f](samp,vox)=m_thsamples[sfs[(sfs.size()-1)-f].second](samp,vox);
	  			phsamples_out[f](samp,vox)=m_phsamples[sfs[(sfs.size()-1)-f].second](samp,vox);
	  			fsamples_out[f](samp,vox)=m_fsamples[sfs[(sfs.size()-1)-f].second](samp,vox);
	  			lamsamples_out[f](samp,vox)=m_lamsamples[sfs[(sfs.size()-1)-f].second](samp,vox);
			}
      		}
      
      		for(int f=0;f<opts.nfibres.value();f++){
			mean_fsamples_out[f](1,vox)=m_mean_fsamples[sfs[(sfs.size()-1)-f].second](vox);
			dyadic_vectors_out[f](1,vox)=m_dyadic_vectors[sfs[(sfs.size()-1)-f].second](1,vox);
			dyadic_vectors_out[f](2,vox)=m_dyadic_vectors[sfs[(sfs.size()-1)-f].second](2,vox);
			dyadic_vectors_out[f](3,vox)=m_dyadic_vectors[sfs[(sfs.size()-1)-f].second](3,vox);
      		}
    	}
    	// save the sorted fibres
    	for(int f=0;f<opts.nfibres.value();f++){
      		//      element_mod_n(thsamples_out[f],M_PI);
      		//      element_mod_n(phsamples_out[f],2*M_PI);

		save_part(thsamples_out[f],"th"+num2str(f+1)+"samples",idpart);

		save_part(phsamples_out[f],"ph"+num2str(f+1)+"samples",idpart);

		save_part(fsamples_out[f],"f"+num2str(f+1)+"samples",idpart);

		//save_part(mean_fsamples_out[f],"mean_f"+num2str(f+1)+"samples",idpart);
		//save_part(dyadic_vectors_out[f],"dyads"+num2str(f+1),idpart);
      
      			
    	}
}
