#include "hip/hip_runtime.h"
/*  levenberg_marquardt.cu

    Tim Behrens, Saad Jbabdi, Stam Sotiropoulos, Moises Hernandez  - FMRIB Image Analysis Group

    Copyright (C) 2005 University of Oxford  */

/*  CCOPYRIGHT  */

#ifndef __LEVENBERG
#define __LEVENBERG

#include "solver_mult_inverse.cu"
#include "diffmodels.cuh"
#include "options.h"

//CPU version in nonlin.h
__device__ const double EPS_gpu = 2.0e-16;       	//Losely based on NRinC 20.1

//CPU version in nonlin.cpp
__device__ inline bool zero_cf_diff_conv(double* cfo,double* cfn,double* cftol){
  	return(2.0*fabs(*cfo-*cfn) <= *cftol*(fabs(*cfo)+fabs(*cfn)+EPS_gpu));
}

__device__ void levenberg_marquardt_PVM_single_gpu(	//INPUT
							const float*		mydata, 
							const float*		bvecs, 
							const float*		bvals, 
							const int		ndirections,
							const int		nfib,
							const int 		nparams,
							const bool 		m_include_f0,
							const int		idSubVOX,
							float* 			step,		//shared memory
							float*			grad,           //shared memory     	          
						   	float* 			hess,		//shared memory
							float* 			inverse,	//shared memory
							double* 		pcf,		//shared memory
							double* 		ncf,		//shared memory
							double* 		lambda,		//shared memory
							double* 		cftol,		//shared memory
							double* 		ltol,		//shared memory
							double* 		olambda,	//shared memory
							int* 			success,    	//shared memory
							int* 			end,    	//shared memory
							float*			J,		//shared memory
							float*			reduction,	//shared memory
							float* 			fs,		//shared memory
						  	float*			x,		//shared memory
							float* 			_d,		//shared memory
						  	float* 			sumf,		//shared memory
							float*			C,		//shared memory
							float*			el,		//shared memory
							int*			indx,		//shared memory
							//INPUT-OUTPUT
							float*			myparams)	//shared memory
{
	int niter=0; 
	int maxiter=200;

   	if(idSubVOX==0){
		*end=false;
   		*lambda=0.1;
   		*cftol=1.0e-8;
   		*ltol=1.0e20;                  
   		*success = true;               
   		*olambda = 0.0;              
   		*ncf=0;  
	}

   	cf_PVM_single(myparams,mydata,bvecs,bvals,ndirections,nfib,nparams,m_include_f0,idSubVOX,reduction,fs,x,_d,sumf,pcf);  
	__syncthreads();

   	while (!(*success&&niter++>=maxiter)){ 	//if success we don't increase niter (first condition is true)
						//function cost has been decreased, we have advanced.
   		if(*success){
    			grad_PVM_single(myparams,mydata,bvecs,bvals,ndirections,nfib,nparams,m_include_f0,idSubVOX,J,reduction,fs,x,_d,sumf,grad); 
			__syncthreads(); 
    			hess_PVM_single(myparams,bvecs,bvals,ndirections,nfib,nparams,m_include_f0,idSubVOX,J,reduction,fs,x,_d,sumf,hess);  
    		}

		if(idSubVOX==0){
    			for (int i=0; i<nparams; i++) {                         
				hess[(i*nparams)+i]+=*lambda-*olambda;	//Levenberg LM_L
    			}

    			solver(hess,grad,nparams,C,el,indx,inverse);

    			for (int i=0;i<nparams;i++){
				step[i]=-inverse[i];		
    			}

   			for(int i=0;i<nparams;i++){
				step[i]=myparams[i]+step[i];
   			}
		}
		
		__syncthreads();
   		cf_PVM_single(step,mydata,bvecs,bvals,ndirections,nfib,nparams,m_include_f0,idSubVOX,reduction,fs,x,_d,sumf,ncf); 

		if(idSubVOX==0){
   			if (*success = (*ncf < *pcf)){ 
				*olambda = 0.0;
        			for(int i=0;i<nparams;i++){
					myparams[i]=step[i];
   				}
        			*lambda=*lambda/10.0;

				if (zero_cf_diff_conv(pcf,ncf,cftol)){
					*end=true;
				}
				*pcf=*ncf;
    			}else{
				*olambda=*lambda;
				*lambda=*lambda*10.0;
				if(*lambda> *ltol){ 
					*end=true;
				}
			}
    		}	
		__syncthreads();
		if(*end) return;		
   	}
}

__device__ void levenberg_marquardt_PVM_single_c_gpu(	//INPUT
							const float*		mydata, 
							const float*		bvecs, 
							const float*		bvals,
							const int		ndirections, 
							const int		nfib,
							const int 		nparams,
							const bool 		m_include_f0,
							const int		idSubVOX,
							float* 			step,		//shared memory
							float*			grad,           //shared memory     	          
						   	float* 			hess,		//shared memory
							float* 			inverse,	//shared memory
							double* 		pcf,		//shared memory
							double* 		ncf,		//shared memory
							double* 		lambda,		//shared memory
							double* 		cftol,		//shared memory
							double* 		ltol,		//shared memory
							double* 		olambda,	//shared memory
							int* 			success,    	//shared memory
							int* 			end,    	//shared memory
							float*			J,		//shared memory
							float*			reduction,	//shared memory
							float* 			fs,		//shared memory
							float*			f_deriv,	//shared memory
						  	float*			x,		//shared memory
							float* 			_d,		//shared memory
						  	float* 			sumf,		//shared memory
							float*			C,		//shared memory
							float*			el,		//shared memory
							int*			indx,		//shared memory
							//INPUT-OUTPUT
							float*			myparams)	//shared memory
{
	int niter=0; 
	int maxiter=200;

   	if(idSubVOX==0){
		*end=false;
   		*lambda=0.1;
   		*cftol=1.0e-8;
   		*ltol=1.0e20;                  
   		*success = true;               
   		*olambda = 0.0;              
   		*ncf=0;  
	}
			
	cf_PVM_single_c(myparams,mydata,bvecs,bvals,ndirections,nfib,nparams,m_include_f0,idSubVOX,reduction,fs,x,_d,sumf,pcf);  
	__syncthreads();
	
   	while (!(*success&&niter++ >= maxiter)){ 	//if success we don't increase niter (first condition is true)
							//function cost has been decreased, we have advanced.
   		if(*success){
			grad_PVM_single_c(myparams,mydata,bvecs,bvals,ndirections,nfib,nparams,m_include_f0,idSubVOX,J,reduction,fs,f_deriv,x,_d,sumf,grad);  
			__syncthreads();
    			hess_PVM_single_c(myparams,bvecs,bvals,ndirections,nfib,nparams,m_include_f0,idSubVOX,J,reduction,fs,f_deriv,x,_d,sumf,hess);  
    		}

		if(idSubVOX==0){
    			for (int i=0; i<nparams; i++) {                         
				hess[(i*nparams)+i]+=*lambda-*olambda;	//Levenberg LM_L
    			}

    			solver(hess,grad,nparams,C,el,indx,inverse);

    			for (int i=0;i<nparams;i++){
				step[i]=-inverse[i];		
    			}

   			for(int i=0;i<nparams;i++){
				step[i]=myparams[i]+step[i];
   			}
		}

		__syncthreads();
   		cf_PVM_single_c(step,mydata,bvecs,bvals,ndirections,nfib,nparams,m_include_f0,idSubVOX,reduction,fs,x,_d,sumf,ncf); 

		if(idSubVOX==0){
   			if (*success = (*ncf < *pcf)) {
				*olambda = 0.0;
        			for(int i=0;i<nparams;i++){
					myparams[i]=step[i];
   				}
        			*lambda=*lambda/10.0;

				if (zero_cf_diff_conv(pcf,ncf,cftol)){
					*end=true;
				}
				*pcf=*ncf;
    			}else{
				*olambda=*lambda;
				*lambda=*lambda*10.0;
				if(*lambda> *ltol){ 
					*end=true;
				}
    			}
		}
		__syncthreads();
		if(*end) return;		
   	}
}


__device__ void levenberg_marquardt_PVM_multi_gpu(	//INPUT
							const float*		mydata, 
							const float*		bvecs, 
							const float*		bvals, 
							const float		R,
							const float		invR,
							const int		ndirections,
							const int		nfib,
							const int 		nparams,
							const bool 		m_include_f0,
							const int		idSubVOX,
							const int		Gamma_for_ball_only,
							float* 			step,		//shared memory
							float*			grad,           //shared memory     	          
						   	float* 			hess,		//shared memory
							float* 			inverse,	//shared memory
							double* 		pcf,		//shared memory
							double* 		ncf,		//shared memory
							double* 		lambda,		//shared memory
							double* 		cftol,		//shared memory
							double* 		ltol,		//shared memory
							double* 		olambda,	//shared memory
							int* 			success,    	//shared memory
							int* 			end,    	//shared memory
							float*			J,		//shared memory
							float*			reduction,	//shared memory
							float* 			fs,		//shared memory
						  	float*			x,		//shared memory
							float* 			_a,		//shared memory
							float* 			_b,		//shared memory
						  	float* 			sumf,		//shared memory
							float*			C,		//shared memory
							float*			el,		//shared memory
							int*			indx,		//shared memory
							//INPUT-OUTPUT
							float*			myparams)	//shared memory
{
	int niter=0; 
	int maxiter=200;

   	if(idSubVOX==0){
		*end=false;
   		*lambda=0.1;
   		*cftol=1.0e-8;
   		*ltol=1.0e20;                  
   		*success = true;               
   		*olambda = 0.0;              
   		*ncf=0;  
	}

	cf_PVM_multi(myparams,mydata,bvecs,bvals,R,invR,ndirections,nfib,nparams,m_include_f0,idSubVOX,Gamma_for_ball_only,reduction,fs,x,_a,_b,sumf,pcf);  
	__syncthreads();
	
   	while (!(*success&&niter++ >= maxiter)){ 	//if success we don't increase niter (first condition is true)
							//function cost has been decreased, we have advanced.
   		if(*success){
			grad_PVM_multi(myparams,mydata,bvecs,bvals,R,invR,ndirections,nfib,nparams,m_include_f0,
			idSubVOX,Gamma_for_ball_only,J,reduction,fs,x,_a,_b,sumf,grad);  

			__syncthreads(); 
    			hess_PVM_multi(myparams,bvecs,bvals,R,invR,ndirections,nfib,nparams,m_include_f0,idSubVOX,Gamma_for_ball_only,J,reduction,fs,x,_a,_b,sumf,hess);  
    		}

		if(idSubVOX==0){
    			for (int i=0; i<nparams; i++) {                         
				hess[(i*nparams)+i]+=*lambda-*olambda;	//Levenberg LM_L
    			}

    			solver(hess,grad,nparams,C,el,indx,inverse);

    			for (int i=0;i<nparams;i++){
				step[i]=-inverse[i];		
    			}

   			for(int i=0;i<nparams;i++){
				step[i]=myparams[i]+step[i];
   			}
		}

		__syncthreads();
   		cf_PVM_multi(step,mydata,bvecs,bvals,R,invR,ndirections,nfib,nparams,m_include_f0,idSubVOX,Gamma_for_ball_only,reduction,fs,x,_a,_b,sumf,ncf); 

		if(idSubVOX==0){
   			if (*success = (*ncf < *pcf)) {
				*olambda = 0.0;
        			for(int i=0;i<nparams;i++){
					myparams[i]=step[i];
   				}
        			*lambda=*lambda/10.0;

				if (zero_cf_diff_conv(pcf,ncf,cftol)){
					*end=true;
				}
				*pcf=*ncf;
    			}else{
				*olambda=*lambda;
				*lambda=*lambda*10.0;
				if(*lambda> *ltol){ 
					*end=true;
				}
    			}
		}
		__syncthreads();
		if(*end) return;				
   	}
}
#endif
