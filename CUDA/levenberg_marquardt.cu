#include "hip/hip_runtime.h"
/*  levenberg_marquardt.cu

    Tim Behrens, Saad Jbabdi, Stam Sotiropoulos, Moises Hernandez  - FMRIB Image Analysis Group

    Copyright (C) 2005 University of Oxford  */

/*  CCOPYRIGHT  */

#ifndef __LEVENBERG
#define __LEVENBERG

#include "solver_mult_inverse.cu"
#include "diffmodels.cuh"
#include "dim_blocks.h"
#include "options.h"

//CPU version in nonlin.h
__device__ const double EPS_gpu = 2.0e-16;       	//Losely based on NRinC 20.1

//CPU version in nonlin.cpp
__device__ inline bool zero_cf_diff_conv(double cfo,double cfn,double cftol){
  	return(2.0*abs(cfo-cfn) <= cftol*(abs(cfo)+abs(cfn)+EPS_gpu));
}

__device__ void levenberg_marquardt_PVM_single_gpu(	//INPUT
							const double*		mydata, 
							const double*		bvecs, 
							const double*		bvals, 
							const int 		nparams,
							const bool 		m_include_f0,
							const int		idB,
							double* 		step,		//shared memory
							double*			grad,           //shared memory     	          
						   	double* 		hess,		//shared memory
							double* 		inverse,	//shared memory
							double 			&pcf,		//shared memory
							double 			&ncf,		//shared memory
							double 			&lambda,	//shared memory
							double 			&cftol,		//shared memory
							double 			&ltol,		//shared memory
							double 			&olambda,	//shared memory
							bool 			&success,    	//shared memory
							bool 			&end,    	//shared memory
							double*			shared,		//shared memory
							double* 		fs,		//shared memory
						  	double*			x,		//shared memory
							double 			&_d,		//shared memory
						  	double 			&sumf,		//shared memory
							//INPUT-OUTPUT
							double*			myparams)	//shared memory
{
	int niter=0; 
	int maxiter=200;

   	if(idB==0){
		end=false;
   		lambda=0.1;
   		cftol=1.0e-8;
   		ltol=1.0e20;                  
   		success = true;               
   		olambda = 0.0;              
   		ncf=0;  
	}

   	cf_PVM_single(myparams,mydata,bvecs,bvals,nparams,m_include_f0,idB,shared,fs,x,_d,sumf,pcf);  
	__syncthreads();

   	while (!(success&&niter++>=maxiter)){ 	//if success we not increase niter (first condition is true)
						//function cost has decreise, we have advanced.
   		if(success){
    			grad_PVM_single(myparams,mydata,bvecs,bvals,nparams,m_include_f0,idB,shared,fs,x,_d,sumf,grad); 
			__syncthreads(); 
    			hess_PVM_single(myparams,bvecs,bvals,nparams,m_include_f0,idB,shared,fs,x,_d,sumf,hess);  
    		}

		if(idB==0){
    			for (int i=0; i<nparams; i++) {                         
				hess[(i*nparams)+i]+=lambda-olambda;	//Levenberg LM_L
    			}

    			solver(hess,grad,nparams,inverse);

    			for (int i=0;i<nparams;i++){
				step[i]=-inverse[i];		
    			}

   			for(int i=0;i<nparams;i++){
				step[i]=myparams[i]+step[i];
   			}
		}
		
		__syncthreads();
   		cf_PVM_single(step,mydata,bvecs,bvals,nparams,m_include_f0,idB,shared,fs,x,_d,sumf,ncf); 

		if(idB==0){
   			if (success = (ncf < pcf)){ 
				olambda = 0.0;
        			for(int i=0;i<nparams;i++){
					myparams[i]=step[i];
   				}
        			lambda=lambda/10.0;

				if (zero_cf_diff_conv(pcf,ncf,cftol)){
					end=true;
				}
				pcf=ncf;
    			}else{
				olambda=lambda;
				lambda=lambda*10.0;
				if(lambda> ltol){ 
					end=true;
				}
			}
    		}	
		__syncthreads();
		if(end) return;		
   	}
}

__device__ void levenberg_marquardt_PVM_single_c_gpu(	//INPUT
							const double*		mydata, 
							const double*		bvecs, 
							const double*		bvals, 
							const int 		nparams,
							const bool 		m_include_f0,
							const int		idB,
							double* 		step,		//shared memory
							double*			grad,           //shared memory     	          
						   	double* 		hess,		//shared memory
							double* 		inverse,	//shared memory
							double 			&pcf,		//shared memory
							double 			&ncf,		//shared memory
							double 			&lambda,	//shared memory
							double 			&cftol,		//shared memory
							double 			&ltol,		//shared memory
							double 			&olambda,	//shared memory
							bool 			&success,    	//shared memory
							bool 			&end,    	//shared memory
							double*			shared,		//shared memory
							double* 		fs,		//shared memory
							double*			f_deriv,	//shared memory
						  	double*			x,		//shared memory
							double 			&_d,		//shared memory
						  	double 			&sumf,		//shared memory
							//INPUT-OUTPUT
							double*			myparams)	//shared memory
{
	int niter=0; 
	int maxiter=200;

   	if(idB==0){
		end=false;
   		lambda=0.1;
   		cftol=1.0e-8;
   		ltol=1.0e20;                  
   		success = true;               
   		olambda = 0.0;              
   		ncf=0;  
	}
			
	cf_PVM_single_c(myparams,mydata,bvecs,bvals,nparams,m_include_f0,idB,shared,fs,x,_d,sumf,pcf);  
	__syncthreads();
	
   	while (!(success&&niter++ >= maxiter)){ 	//if success we not increase niter (first condition is true)
							//function cost has decreise, we have advanced.
   		if(success){
			grad_PVM_single_c(myparams,mydata,bvecs,bvals,nparams,m_include_f0,idB,shared,fs,f_deriv,x,_d,sumf,grad);  
			__syncthreads();
    			hess_PVM_single_c(myparams,bvecs,bvals,nparams,m_include_f0,idB,shared,fs,f_deriv,x,_d,sumf,hess);  
    		}

		if(idB==0){
    			for (int i=0; i<nparams; i++) {                         
				hess[(i*nparams)+i]+=lambda-olambda;	//Levenberg LM_L
    			}

    			solver(hess,grad,nparams,inverse);

    			for (int i=0;i<nparams;i++){
				step[i]=-inverse[i];		
    			}

   			for(int i=0;i<nparams;i++){
				step[i]=myparams[i]+step[i];
   			}
		}

		__syncthreads();
   		cf_PVM_single_c(step,mydata,bvecs,bvals,nparams,m_include_f0,idB,shared,fs,x,_d,sumf,ncf); 

		if(idB==0){
   			if (success = (ncf < pcf)) {
				olambda = 0.0;
        			for(int i=0;i<nparams;i++){
					myparams[i]=step[i];
   				}
        			lambda=lambda/10.0;

				if (zero_cf_diff_conv(pcf,ncf,cftol)){
					end=true;
				}
				pcf=ncf;
    			}else{
				olambda=lambda;
				lambda=lambda*10.0;
				if(lambda> ltol){ 
					end=true;
				}
    			}
		}
		__syncthreads();
		if(end) return;		
   	}
}


__device__ void levenberg_marquardt_PVM_multi_gpu(	//INPUT
							const double*		mydata, 
							const double*		bvecs, 
							const double*		bvals, 
							const int 		nparams,
							const bool 		m_include_f0,
							const int		idB,
							double* 		step,		//shared memory
							double*			grad,           //shared memory     	          
						   	double* 		hess,		//shared memory
							double* 		inverse,	//shared memory
							double 			&pcf,		//shared memory
							double 			&ncf,		//shared memory
							double 			&lambda,	//shared memory
							double 			&cftol,		//shared memory
							double 			&ltol,		//shared memory
							double 			&olambda,	//shared memory
							bool 			&success,    	//shared memory
							bool 			&end,    	//shared memory
							double*			shared,		//shared memory
							double* 		fs,		//shared memory
						  	double*			x,		//shared memory
							double 			&_a,		//shared memory
							double 			&_b,		//shared memory
						  	double 			&sumf,		//shared memory
							//INPUT-OUTPUT
							double*			myparams)	//shared memory
{
	int niter=0; 
	int maxiter=200;

   	if(idB==0){
		end=false;
   		lambda=0.1;
   		cftol=1.0e-8;
   		ltol=1.0e20;                  
   		success = true;               
   		olambda = 0.0;              
   		ncf=0;  
	}

	cf_PVM_multi(myparams,mydata,bvecs,bvals,nparams,m_include_f0,idB,shared,fs,x,_a,_b,sumf,pcf);  
	__syncthreads();
	
   	while (!(success&&niter++ >= maxiter)){ 	//if success we not increase niter (first condition is true)
							//function cost has decreise, we have advanced.
   		if(success){
			grad_PVM_multi(myparams,mydata,bvecs,bvals,nparams,m_include_f0,idB,shared,fs,x,_a,_b,sumf,grad);  
			__syncthreads(); 
    			hess_PVM_multi(myparams,bvecs,bvals,nparams,m_include_f0,idB,shared,fs,x,_a,_b,sumf,hess);  
    		}

		if(idB==0){
    			for (int i=0; i<nparams; i++) {                         
				hess[(i*nparams)+i]+=lambda-olambda;	//Levenberg LM_L
    			}

    			solver(hess,grad,nparams,inverse);

    			for (int i=0;i<nparams;i++){
				step[i]=-inverse[i];		
    			}

   			for(int i=0;i<nparams;i++){
				step[i]=myparams[i]+step[i];
   			}
		}

		__syncthreads();
   		cf_PVM_multi(step,mydata,bvecs,bvals,nparams,m_include_f0,idB,shared,fs,x,_a,_b,sumf,ncf); 

		if(idB==0){
   			if (success = (ncf < pcf)) {
				olambda = 0.0;
        			for(int i=0;i<nparams;i++){
					myparams[i]=step[i];
   				}
        			lambda=lambda/10.0;

				if (zero_cf_diff_conv(pcf,ncf,cftol)){
					end=true;
				}
				pcf=ncf;
    			}else{
				olambda=lambda;
				lambda=lambda*10.0;
				if(lambda> ltol){ 
					end=true;
				}
    			}
		}
		__syncthreads();
		if(end) return;				
   	}
}
#endif
