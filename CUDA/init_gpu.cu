/*  init_gpu.cu

    Tim Behrens, Saad Jbabdi, Stam Sotiropoulos, Moises Hernandez  - FMRIB Image Analysis Group

    Copyright (C) 2005 University of Oxford  */

/*  CCOPYRIGHT  */

#include "sync_check.h"
#include <fstream>

void init_gpu(){
	
	int *q;
	hipMalloc((void **)&q, sizeof(int));
	hipFree(q);
	sync_check("init_gpu");

	int device;
  	hipGetDevice(&device);
  	printf ("\n...................In the GPU launcher on the device %d...................\n", device); 
	hipDeviceSetCacheConfig(hipFuncCachePreferShared);
	
	sync_check("init_gpu");
} 

double timeval_diff(struct timeval *a, struct timeval *b){
	return (double)(a->tv_sec +(double)a->tv_usec/1000000) - (double)(b->tv_sec +(double)b->tv_usec/1000000);
}
