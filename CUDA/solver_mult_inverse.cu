#include "hip/hip_runtime.h"
/*  solver_mult_inverse.cu

    Tim Behrens, Saad Jbabdi, Stam Sotiropoulos, Moises Hernandez  - FMRIB Image Analysis Group

    Copyright (C) 2005 University of Oxford  */

/*  CCOPYRIGHT  */

#include "options.h"

//X = A.i() * B . Used in Levenberg-Marquardt
//MATRIX INVERSE AS NEWMAT LU SOLVER
//implemented in NEWMAT:newmat7.cpp GeneralSolvI.
__device__ void solver(	//INPUT
			double *A, 
			double *P,
			int length,
			//TO USE
			double *C,
			double *el,
			int *indx,	
			//OUTPUT
			double *B)  
{  
	//double C[NPARAMS*NPARAMS];

	for(int i=0;i<length;i++){
		for(int j=0;j<length;j++){
			C[i*length+j]=A[i*length+j];
		}
	}
	
 	bool d=true; 
  	//int indx[NPARAMS];

   	double* akk = C;   
	double big = fabs(*akk); 
	int mu = 0; 
	double* ai = akk; 
	int k;

	for (k = 1; k<length; k++){
      		ai += length; 
		const double trybig = fabs(*ai);
      		if (big < trybig){ 
			big = trybig; 
			mu = k; 
		}
   	}

   	if(length) for (k = 0;;){

		indx[k] = mu;
		if (mu != k){
         		double* a1 = C + length*k; 
			double* a2 = C + length*mu; 
			d = !d;
         		int j = length;
         		while (j--){ 
				const double temp = *a1; 
				*a1++ = *a2; 
				*a2++ = temp; 
			}
      		}

      		double diag = *akk; 
		big = 0; 
		mu = k + 1;
      		if (diag != 0){
         		ai = akk; 
			int i = length - k - 1;
         		while (i--){
            			ai += length; 
				double* al = ai; 
				double mult = *al / diag; 
				*al = mult;
            			int l = length - k - 1; 
				double* aj = akk;
				if (l-- != 0){
				
					double aux=al[1]-(mult* *(++aj));
					*(++al) = aux;
					//*(++al) = __dadd_rn (*al,-mult* *(++aj)); //FAIL in cuda 4.2 compiler
					
               				const double trybig = fabs(*al);
               				if (big < trybig){ 
						big = trybig; 
						mu = length - i - 1; 
					}
               				while (l--){ 
						double aux= al[1]-(mult* *(++aj));
						*(++al) = aux;
						//*(++al) = __dadd_rn (*al,-mult* *(++aj)); //FAIL in cuda 4.2 compiler
					}
           			 }
         		}
      		}
      		if (++k == length) break;      
      		akk += length + 1;
   	}


//////////////////////////////

	//double el[NPARAMS];

	for(int e=0;e<length;e++){
		el[e]=P[e];		
    	}
		
   	int j;
	int ii = length; 
	int ip;    
	double temp;
	int i;
     
	for (i=0; i<length; i++){
 		ip = indx[i]; 
		temp = el[ip]; 
		el[ip] = el[i];
		el[i] = temp;
      		if (temp != 0.0) { ii = i; break; }
   	}
	
  	double* bi; 
	double* ai2;
   	i = ii + 1;

  	if (i < length){
      		bi = el + ii; 
		ai2 = C + ii + i * length;
      		for (;;){
         		int ip = indx[i]; 
			double sum = el[ip]; 
			el[ip] = el[i];
         		double* aij = ai2; 
			double* bj = bi; 
			j = i - ii;
         		while (j--){ 
				sum -=  *aij++* *bj++; 
			}
         		el[i] = sum;
         		if (++i == length) break;
         		ai2 += length;
      		}
   	}

   	ai2 = C + length*length;

   	for (i = length - 1; i >= 0; i--){
      		double* bj = el+i; 
		ai2 -= length; 
		double* ajx = ai2+i;
      		double sum = *bj; 
		double diag = *ajx;
      		j = length - i; 
		while(--j){ 
			sum -= *(++ajx)* *(++bj);  
		}
      		el[i] = sum / diag;
			
   	}
	for(int e=0;e<length;e++){
		B[e]=el[e];
    	}
}

