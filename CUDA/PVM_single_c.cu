#include "hip/hip_runtime.h"
/*  PVM_single_c.cu

    Tim Behrens, Saad Jbabdi, Stam Sotiropoulos, Moises Hernandez  - FMRIB Image Analysis Group

    Copyright (C) 2005 University of Oxford  */

/*  CCOPYRIGHT  */

#include "diffmodels_utils.h"
#include "levenberg_marquardt.cu"
#include "options.h"

#include <fstream>

/////////////////////////////////////
/////////////////////////////////////
/// 	    PVM_single_c	  /// 
/////////////////////////////////////
/////////////////////////////////////

__device__ 
inline double isoterm_PVM_single_c(const int pt,const double _d,const double *bvals){
  	return exp(double(-bvals[pt]*_d));
}

__device__ 
inline double isoterm_lambda_PVM_single_c(const int pt,const double lambda,const double *bvals){
  	return(-2*bvals[pt]*lambda*exp(double(-bvals[pt]*lambda*lambda)));
}

__device__ 
inline double anisoterm_PVM_single_c(const int pt,const double _d,const double3 x, const double *bvecs, const double *bvals){
	double dp = bvecs[pt]*x.x+bvecs[NDIRECTIONS+pt]*x.y+bvecs[(2*NDIRECTIONS)+pt]*x.z;
	return exp(double(-bvals[pt]*_d*dp*dp));
}

__device__ 
inline double anisoterm_lambda_PVM_single_c(const int pt,const double lambda,const double3 x, const double *bvecs, const double *bvals){
	double dp = bvecs[pt]*x.x+bvecs[NDIRECTIONS+pt]*x.y+bvecs[(2*NDIRECTIONS)+pt]*x.z;
	return(-2*bvals[pt]*lambda*dp*dp*exp(double(-bvals[pt]*lambda*lambda*dp*dp)));
}

__device__ 
inline double anisoterm_th_PVM_single_c(const int pt,const double _d,const double3 x, const double _th,const double _ph,const double *bvecs, const double *bvals){

	double dp = bvecs[pt]*x.x+bvecs[NDIRECTIONS+pt]*x.y+bvecs[(2*NDIRECTIONS)+pt]*x.z;
	double dp1 = cos(double(_th))*(bvecs[pt]*cos(double(_ph))+bvecs[NDIRECTIONS+pt]*sin(double(_ph)))-bvecs[(2*NDIRECTIONS)+pt]*sin(double(_th));
  	return(-2*bvals[pt]*_d*dp*dp1*exp(double(-bvals[pt]*_d*dp*dp)));
}

__device__ 
inline double anisoterm_ph_PVM_single_c(const int pt,const double _d,const double3 x, const double _th,const double _ph,const double *bvecs, const double *bvals){
  	double dp = bvecs[pt]*x.x+bvecs[NDIRECTIONS+pt]*x.y+bvecs[(2*NDIRECTIONS)+pt]*x.z;
	double dp1 = sin(double(_th))*(-bvecs[pt]*sin(double(_ph))+bvecs[NDIRECTIONS+pt]*cos(double(_ph)));
  	return(-2*bvals[pt]*_d*dp*dp1*exp(double(-bvals[pt]*_d*dp*dp)));
}



//If the sum of the fractions is >1, then zero as many fractions
//as necessary, so that the sum becomes smaller than 1.
//in diffmodel.cc
__device__ void fix_fsum_PVM_single_c(		//INPUT 
						int nfib,
						//INPUT - OUTPUT){
						double *fs)
{
  	double sumf=0.0;
  	for(int i=0;i<nfib;i++){
    		sumf+=fs[i];
    		if(sumf>=1){
      			for(int j=i;j<nfib;j++) 
				fs[j]=FSMALL_gpu;  //make the fraction almost zero
      			break;
    		}
  	}
}

//Returns 1-Sum(f_j), 1<=j<=ii. (ii<=nfib)
//Used for transforming beta to f and vice versa
//in diffmodel.cc
__device__ double partial_fsum_PVM_single_c(double* fs, int ii){
  	double fsum=1.0;
  	for(int j=0;j<ii;j++){
   		fsum-=fs[j];
	}
  	return fsum;
}



//in diffmodel.cc
__device__ void sort_PVM_single_c(int nfib,int nparams,double* params)
{
	double temp_f, temp_th, temp_ph;
	// Order vector descending using f parameters as index
  	for(int i=1; i<(nfib); i++){ 
    		for(int j=0; j<(nfib-i); j++){ 
      			if (params[2+j*3] < params[2+i*3]){ 
        			temp_f = params[2+j*3];
				temp_th = params[2+j*3+1];
				temp_ph = params[2+j*3+2];
        			params[2+j*3] = params[2+i*3]; 
				params[2+j*3+1] = params[2+i*3+1]; 
				params[2+j*3+2] = params[2+i*3+2]; 
        			params[2+i*3] = temp_f; 
				params[2+i*3+1] = temp_th; 
				params[2+i*3+2] = temp_ph; 
      			} 
    		} 
  	} 

	//if (m_return_fanning){
     	 //	fantmp=m_fanning_angles;
      	//	Hess_vec_tmp=m_invprHes_e1;
      	//	Hess=m_Hessian;
  	//}

	//if (m_return_fanning){
     	 	//m_fanning_angles(i)=fantmp(fvals[ii].second);
      		//m_invprHes_e1[i-1]=Hess_vec_tmp[fvals[ii].second-1];
      		//m_Hessian[i-1]=Hess[fvals[ii].second-1];
    	//}
}

//in diffmodels.cc -- for calculate residuals
__device__ void  forwardModel_PVM_single_c(	//INPUT
						const double* 		p,
						const double*		bvecs, 
						const double*		bvals,
						const int		nfib,
						const int 		nparams,
						const bool 		m_include_f0,
						//OUTPUT
						double*		 	predicted_signal)
{
  	for(int i=0;i<NDIRECTIONS;i++){
		predicted_signal[i]=0;		//pred = 0;
	}
  	double val;
  	double _d = lambda2d_gpu(p[1]);
  	////////////////////////////////////
  	double fs[NFIBRES];
  	double x[NFIBRES*3];	
  	double sumf=0;
	double3 x2;
	double partial_fsum;
  	for(int k=0;k<nfib;k++){
    		int kk = 2+3*k;
		////// partial_fsum //////
		partial_fsum=1.0;
		for(int j=0;j<k;j++)
			partial_fsum-=fs[j];
    		//////////////////////////
	    	fs[k] = beta2f_gpu(p[kk])*partial_fsum;
	    	sumf += fs[k];
		x[k*3] = sin(p[kk+1])*cos(p[kk+2]);
    		x[k*3+1] = sin(p[kk+1])*sin(p[kk+2]);
    		x[k*3+2] = cos(p[kk+1]);
  	}
  	////////////////////////////////////
  	for(int i=0;i<NDIRECTIONS;i++){
    		val = 0.0;
    		for(int k=0;k<nfib;k++){
			x2.x=x[k*3];
			x2.y=x[k*3+1];
			x2.z=x[k*3+2];	 
      			val += fs[k]*anisoterm_PVM_single_c(i,_d,x2,bvecs,bvals);
    		}	
    		if (m_include_f0){
			//partial_fsum ///////////
			partial_fsum=1.0;
			for(int j=0;j<NFIBRES;j++)
				partial_fsum-=fs[j];
	     		//////////////////////////
      			double temp_f0= beta2f_gpu(p[nparams-1])*partial_fsum;
      			predicted_signal[i] = p[0]*(temp_f0+(1-sumf-temp_f0)*isoterm_PVM_single_c(i,_d,bvals)+val);
    		} 
    		else
      			predicted_signal[i] = p[0]*((1-sumf)*isoterm_PVM_single_c(i,_d,bvals)+val); 
  	}
}

//in diffmodels.cc -- for calculate residuals
__device__ void get_prediction_PVM_single_c(	//INPUT
						const double*	params,
						const double*	bvecs, 
						const double*	bvals,
						const int 	nfib,
						const int 	nparams,
						const bool 	m_include_f0,
						//OUTPUT
						double* 	predicted_signal)
{
	//m_s0-myparams[0] 	m_d-myparams[1] 	m_d_std-myparams[2]		m_f-m_th-m_ph-myparams[3,4,5,6 etc..]   	m_f0-myparams[nparams-1]
  	double p[NPARAMS];
  	p[0] = params[0];
	if(params[1]<0)  p[1] = 0;	//This can be due to numerical errors..sqrt
  	else p[1] = d2lambda_gpu(params[1]);	
	double partial_fsum;	
	double fs[NFIBRES];
  	for(int k=0;k<nfib;k++){
    		int kk = 2+3*k;
		//partial_fsum ///////////
		partial_fsum=1.0;
		for(int j=0;j<k;j++)
			partial_fsum-=fs[j];
	     	//////////////////////////
		fs[k] = params[kk];
		double tmpr=params[kk]/partial_fsum;
    		if (tmpr>1.0) tmpr=1; //This can be due to numerical errors
		if (tmpr<0.0) tmpr=0; //This can be due to numerical errors..sqrt
    		p[kk]   = f2beta_gpu(tmpr);
    		p[kk+1] = params[kk+1];
    		p[kk+2] = params[kk+2];
  	}
  	if (m_include_f0){
		//partial_fsum ///////////
		partial_fsum=1.0;
		for(int j=0;j<NFIBRES;j++)
			partial_fsum-=fs[j];
	     	//////////////////////////	
		double tmpr=params[nparams-1]/partial_fsum;
    		if (tmpr>1.0) tmpr=1; //This can be due to numerical errors..asin
		if (tmpr<0.0) tmpr=0; //This can be due to numerical errors..sqrt
    		p[nparams-1]= f2beta_gpu(tmpr);	
	}
  	forwardModel_PVM_single_c(p,bvecs,bvals,nfib,nparams,m_include_f0,predicted_signal);
}

//cost function PVM_single_c
__device__ double cf_PVM_single_c(	//INPUT
					const double*		params,
					const double*		mdata,
					const double*		bvecs, 
					const double*		bvals,
					const int 		nparams,
					const bool 		m_include_f0)
{
	double cfv = 0.0;
  	double err;
	double _d = lambda2d_gpu(params[1]);
	double fs[NFIBRES];    
	double x[NFIBRES*3];	
	double sumf=0;
	double3 x2;
        double partial_fsum;

	for(int k=0;k<NFIBRES;k++){
    		int kk = 2+3*(k);
    		//partial_fsum ///////////
		partial_fsum=1.0;
		for(int j=0;j<k;j++)
			partial_fsum-=fs[j];
    		//////////////////////////
		fs[k] = beta2f_gpu(params[kk])*partial_fsum;
    		sumf += fs[k];
    		
    		x[k*3] = sin(params[kk+1])*cos(params[kk+2]);
    		x[k*3+1] = sin(params[kk+1])*sin(params[kk+2]);
    		x[k*3+2] = cos(params[kk+1]);
  	}
	
	for(int i=0;i<NDIRECTIONS;i++){
    		err = 0.0;
    		for(int k=0;k<NFIBRES;k++){
			x2.x=x[k*3];
			x2.y=x[k*3+1];
			x2.z=x[k*3+2];	
			err += fs[k]*anisoterm_PVM_single_c(i,_d,x2,bvecs,bvals); 
    		}
		if(m_include_f0){
			//partial_fsum ///////////
			partial_fsum=1.0;
			for(int j=0;j<NFIBRES;j++)
				partial_fsum-=fs[j];
	     		//////////////////////////
	      		double temp_f0=beta2f_gpu(params[nparams-1])*partial_fsum;
			err=(params[0]*((temp_f0+(1-sumf-temp_f0)*isoterm_PVM_single_c(i,_d,bvals))+err))-mdata[i];
		}else{

			err = params[0]*((1-sumf)*isoterm_PVM_single_c(i,_d,bvals)+err)-mdata[i];
		
		}
		cfv += err*err;  	
  	}  

	return(cfv);
}


//gradient function PVM_single_c
__device__ void grad_PVM_single_c(	//INPUT
					const double*		params,
					const double*		mdata,
					const double*		bvecs, 
					const double*		bvals,
					const int 		nparams,
					const bool 		m_include_f0,
					//OUTPUT
					double*			grad)
{
  	double _d = lambda2d_gpu(params[1]);
  	double fs[NFIBRES];
  	double bs[NFIBRES];
  	double x[NFIBRES*3];	
  	double3 xx;		
  	double sumf=0;
  	double partial_fsum;

  	for(int k=0;k<NFIBRES;k++){
    		int kk = 2+3*(k);
    		bs[k]=params[kk];
    		//partial_fsum ///////////
		partial_fsum=1.0;
		for(int j=0;j<k;j++){
			partial_fsum-=fs[j];
		}
   		 //////////////////////////

    		fs[k] = beta2f_gpu(params[kk])*partial_fsum;
    		sumf += fs[k];
    		x[k*3] = sin(params[kk+1])*cos(params[kk+2]);
    		x[k*3+1] = sin(params[kk+1])*sin(params[kk+2]);
    		x[k*3+2] = cos(params[kk+1]);
  	}

  	////////// fraction deriv //////////////
  	// f_deriv=fractions_deriv(nfib, fs, bs);  
  	//////////////////////////////////////
  	double f_deriv[NFIBRES*NFIBRES];
  	double fsum;
  	for (int j=0; j<NFIBRES; j++){
   		 for (int k=0; k<NFIBRES; k++){
			f_deriv[j*NFIBRES+k]=0;
    		}
  	}  

  	for (int j=0; j<NFIBRES; j++){
    		for (int k=0; k<NFIBRES; k++){
      			if (j==k){
				fsum=1; 
				for (int n=0; n<=(j-1); n++)
	  				fsum-=fs[n];
	  			f_deriv[j*NFIBRES+k] = sin(double(2*bs[k]))*fsum;
      			}else if (j>k){
				fsum=0;
				for (int n=0; n<=(j-1); n++)
	  				fsum += f_deriv[n*NFIBRES+k];
				f_deriv[j*NFIBRES+k] += -sin(bs[j])*sin(bs[j])*fsum;

      			}
    		} 	   
  	}
  	///////////////////////////////
  	double J[NPARAMS];
  	double diff;
  	double sig,Iso_term;
  	double Aniso_terms[NFIBRES];

	for (int p=0;p<nparams;p++) grad[p]=0;
  
  	for(int i=0;i<NDIRECTIONS;i++){
    		Iso_term=isoterm_PVM_single_c(i,_d,bvals);  //Precompute some terms for this datapoint
    		for(int k=0;k<NFIBRES;k++){
      			xx.x=x[k*3];
      			xx.y=x[k*3+1];
     			xx.z=x[k*3+2];	
      			Aniso_terms[k]=anisoterm_PVM_single_c(i,_d,xx,bvecs,bvals);
    		}
    		sig = 0;
    		for(int a=0;a<nparams;a++) J[a]=0;
    		for(int k=0;k<NFIBRES;k++){
     			int kk = 2+3*(k);
      			xx.x=x[k*3];
      			xx.y=x[k*3+1];
      			xx.z=x[k*3+2];		
      			sig += fs[k]*Aniso_terms[k];
     			J[1] += params[0]*fs[k]*anisoterm_lambda_PVM_single_c(i,params[1],xx,bvecs,bvals);
     			J[kk] = 0;
      			for (int j=0;j<NFIBRES;j++){
				if(f_deriv[j*NFIBRES+k]!=0){
	  				J[kk] += params[0]*(Aniso_terms[j]-Iso_term)*f_deriv[j*NFIBRES+k]; 
				}
      			}
      			J[kk+1] = params[0]*fs[k]*anisoterm_th_PVM_single_c(i,_d,xx,params[kk+1],params[kk+2],bvecs,bvals);
      			J[kk+2] = params[0]*fs[k]*anisoterm_ph_PVM_single_c(i,_d,xx,params[kk+1],params[kk+2],bvecs,bvals);
    		}
    		if(m_include_f0){
			//partial_fsum ///////////
    			partial_fsum=1.0;
    			for(int j=0;j<(NFIBRES);j++)
				partial_fsum-=fs[j];
			//////////////////////////
			double temp_f0=beta2f_gpu(params[nparams-1])*partial_fsum;

    			//derivative with respect to f0
    			J[nparams-1]= params[0]*(1-Iso_term)*sin(double(2*params[nparams-1]))*partial_fsum; 
			sig=params[0]*((temp_f0+(1-sumf-temp_f0)*Iso_term)+sig);
    			J[1] += params[0]*(1-sumf-temp_f0)*isoterm_lambda_PVM_single_c(i,params[1],bvals);
    		}else{
			sig = params[0]*((1-sumf)*Iso_term+sig);
	    		J[1] += params[0]*(1-sumf)*isoterm_lambda_PVM_single_c(i,params[1],bvals);
    		}
    		diff = sig - mdata[i];
    		J[0] = sig/params[0]; 

		for (int p=0;p<nparams;p++) grad[p] += 2*J[p]*diff;
  	}
}


//hessian function PVM_single_c
__device__ void hess_PVM_single_c(	//INPUT
					const double*		params,
					const double*		bvecs, 
					const double*		bvals,
					const int 		nparams,
					const bool 		m_include_f0,
					//OUTPUT
					double*			hess)
{
  	double _d = lambda2d_gpu(params[1]);
  	double fs[NFIBRES];
  	double bs[NFIBRES];
  	double x[NFIBRES*3];	
  	double3 xx;
  	double sumf=0;
  	double partial_fsum;

  	for(int k=0;k<NFIBRES;k++){
    		int kk = 2+3*(k);
    		bs[k]=params[kk];
    		//partial_fsum ///////////
		partial_fsum=1.0;
		for(int j=0;j<k;j++)
			partial_fsum-=fs[j];
    		//////////////////////////
    		fs[k] = beta2f_gpu(params[kk])*partial_fsum;
    		sumf += fs[k];
    		x[k*3] = sin(params[kk+1])*cos(params[kk+2]);
    		x[k*3+1] = sin(params[kk+1])*sin(params[kk+2]);
    		x[k*3+2] = cos(params[kk+1]);
  	}

  	////////// fraction deriv //////////////
  	// f_deriv=fractions_deriv(nfib, fs, bs);  
  	//////////////////////////////////////
  	double f_deriv[NFIBRES*NFIBRES];
  	double fsum;
  	for (int j=0; j<NFIBRES; j++){
    		for (int k=0; k<NFIBRES; k++){
			f_deriv[j*NFIBRES+k]=0;
    		}
  	}

  	for (int j=0; j<NFIBRES; j++){
    		for (int k=0; k<NFIBRES; k++){
      			if (j==k){
				fsum=1; 
				for (int n=0; n<=(j-1); n++)
	  			fsum-=fs[n];
	  			f_deriv[j*NFIBRES+k] = sin(double(2*bs[k]))*fsum;
      			}
      			else if (j>k){
				fsum=0;
				for (int n=0; n<=(j-1); n++)
	  				fsum += f_deriv[n*NFIBRES+k];
				f_deriv[j*NFIBRES+k] += -sin(bs[j])*sin(bs[j])*fsum;
      			}
    		} 	   
  	}
  	///////////////////////////////
  	double J[NPARAMS];
  	double sig,Iso_term;
  	double Aniso_terms[NFIBRES];

	for (int p=0;p<nparams;p++){
		for (int p2=0;p2<nparams;p2++){ 
			hess[p*nparams+p2] = 0;
		}
	}

  	for(int i=0;i<NDIRECTIONS;i++){
    		Iso_term=isoterm_PVM_single_c(i,_d,bvals);  //Precompute some terms for this datapoint
    		for(int k=0;k<NFIBRES;k++){
      			xx.x=x[k*3];
      			xx.y=x[k*3+1];
      			xx.z=x[k*3+2];	
      			Aniso_terms[k]=anisoterm_PVM_single_c(i,_d,xx,bvecs,bvals);
    		}
    		sig = 0;
    		for(int a=0;a<nparams;a++) J[a]=0;
    		for(int k=0;k<NFIBRES;k++){
      			int kk = 2+3*(k);
      			xx.x=x[k*3];
      			xx.y=x[k*3+1];
      			xx.z=x[k*3+2];		 
      			sig += fs[k]*Aniso_terms[k];
      			J[1] += params[0]*fs[k]*anisoterm_lambda_PVM_single_c(i,params[1],xx,bvecs,bvals);	 
      			J[kk] = 0;
      			for (int j=0; j<NFIBRES; j++){
				if (f_deriv[j*NFIBRES+k]!=0)
	  			J[kk] += params[0]*(Aniso_terms[j]-Iso_term)*f_deriv[j*NFIBRES+k]; 
      			}
      			J[kk+1] = params[0]*fs[k]*anisoterm_th_PVM_single_c(i,_d,xx,params[kk+1],params[kk+2],bvecs,bvals);
      			J[kk+2] = params[0]*fs[k]*anisoterm_ph_PVM_single_c(i,_d,xx,params[kk+1],params[kk+2],bvecs,bvals);
    		}
    		if(m_include_f0){
			//partial_fsum ///////////
	    		partial_fsum=1.0;
	    		for(int j=0;j<(NFIBRES);j++)
				partial_fsum-=fs[j];
	    		//////////////////////////
    			double temp_f0=beta2f_gpu(params[nparams-1])*partial_fsum;
    			//derivative with respect to f0
    			J[nparams-1]= params[0]*(1-Iso_term)*sin(double(2*params[nparams-1]))*partial_fsum; 
			sig= params[0]*((temp_f0+(1-sumf-temp_f0)*Iso_term)+sig);
    			J[1] += params[0]*(1-sumf-temp_f0)*isoterm_lambda_PVM_single_c(i,params[1],bvals);
    		}else{
	    		sig = params[0]*((1-sumf)*Iso_term+sig);
	    		J[1] += params[0]*(1-sumf)*isoterm_lambda_PVM_single_c(i,params[1],bvals);
    		}
    		J[0] = sig/params[0]; 

		for (int p=0;p<nparams;p++){
			for (int p2=p;p2<nparams;p2++){ 
				hess[p*nparams+p2] += 2*(J[p]*J[p2]);
			}
		}
  	}

  	for (int j=0; j<nparams; j++) {
    		for (int i=j+1; i<nparams; i++) {
     			hess[i*nparams+j]=hess[j*nparams+i];	
    		}
  	}
}

//in diffmodel.cc
extern "C" __global__ void fit_PVM_single_c_kernel(	//INPUT
							const double* 		data, 
							const double* 		bvecs, 
							const double* 		bvals, 
							const int 		nvox, 
							const int 		nfib, 
							const bool		m_eval_BIC,
							const bool 		m_include_f0,
							const bool	 	m_return_fanning,
							//INPUT - OUTPUT
							double* 		params)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;	
   	if (id >=nvox) { return; }	

	int nparams;
	if (m_include_f0)
      		nparams = nfib*3 + 3; 
    	else
      		nparams = nfib*3 + 2;

   	double myparams[NPARAMS];
   	double mydata[NDIRECTIONS];

	for(int i=0;i<nparams;i++){
		myparams[i]=params[(id*nparams)+i];
   	}

   	for(int i=0;i<NDIRECTIONS;i++){
		mydata[i]=data[(id*NDIRECTIONS)+i];
   	}

	//if(id==1) for(int i=0;i<nparams;i++)printf("START[%i]: %.20f\n",i,myparams[i]);
	//do the fit
	levenberg_marquardt_PVM_single_c_gpu(mydata, &bvecs[id*3*NDIRECTIONS], &bvals[id*NDIRECTIONS], nparams, m_include_f0, myparams);

	//double m_BIC;
	//if (m_eval_BIC){  
    	//	double RSS= cf_PVM_single_c(myparams,mydata,&bvecs[id*3*NDIRECTIONS], &bvals[id*NDIRECTIONS], nparams,m_include_f0); // get the sum of squared residuals
    	//	m_BIC=NDIRECTIONS*log(double(RSS/NDIRECTIONS))+log(double(NDIRECTIONS))*nparams; // evaluate BIC
  	//}   NOT USED at the moment

	// finalise parameters
	// m_s0-myparams[0] 	m_d-myparams[1] 	m_f-m_th-m_ph-myparams[2,3,4,5, etc..]   	m_f0-myparams[nparams-1]
	
	double m_f[NFIBRES]; 					// for partial_fsum

  	myparams[1] = lambda2d_gpu(myparams[1]); 
  	for(int k=0;k<nfib;k++){
    		int kk = 2 + 3*(k);
    		myparams[kk]  = beta2f_gpu(myparams[kk])*partial_fsum_PVM_single_c(m_f,k);
		m_f[k]=myparams[kk];
  	}
  
  	//if (m_return_fanning)
    		//Fanning_angles_from_Hessian(); NOT USED at the moment
  
  	if (m_include_f0)
    		myparams[nparams-1]= beta2f_gpu(myparams[nparams-1])*partial_fsum_PVM_single_c(m_f,nfib);

	sort_PVM_single_c(nfib,nparams,myparams);

	for(int i=0;i<nparams;i++){
		params[(id*nparams)+i] = myparams[i];
   	}
}

//in diffmodel.cc
extern "C" __global__ void get_residuals_PVM_single_c_kernel(	//INPUT
								const double* 		data, 
								const double* 		params,
								const double* 		bvecs, 
								const double* 		bvals, 
								const int 		nvox, 
								const int 		nfib, 
								const bool 		m_include_f0,
								const bool* 		includes_f0,
								//OUTPUT
								double*			residuals)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;	
   	if (id >=nvox) { return; }	

	int nparams;
	if (m_include_f0)
      		nparams = nfib*3 + 3; 
    	else
      		nparams = nfib*3 + 2;

	bool my_include_f0 = includes_f0[id];

   	double myparams[NPARAMS];
   	double mydata[NDIRECTIONS];

	for(int i=0;i<nparams;i++){
		myparams[i]=params[(id*nparams)+i];
   	}

   	for(int i=0;i<NDIRECTIONS;i++){
		mydata[i]=data[(id*NDIRECTIONS)+i];
   	}

	double predicted_signal[NDIRECTIONS];

	get_prediction_PVM_single_c(myparams, &bvecs[id*3*NDIRECTIONS], &bvals[id*NDIRECTIONS], nfib, nparams, my_include_f0, predicted_signal);

	for(int i=0;i<NDIRECTIONS;i++){		//residuals=m_data-predicted_signal;
		residuals[id*NDIRECTIONS+i]= mydata[i] - predicted_signal[i];
	}
}

