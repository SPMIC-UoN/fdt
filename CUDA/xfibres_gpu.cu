#include "hip/hip_runtime.h"
/*  xfibres_gpu.cu

    Tim Behrens, Saad Jbabdi, Stam Sotiropoulos, Moises Hernandez  - FMRIB Image Analysis Group

    Copyright (C) 2005 University of Oxford  */

/*  CCOPYRIGHT  */

#include "newmat.h"
#include "newimage/newimageall.h"
#include "xfibresoptions.h"

#include "xfibres_gpu.cuh"
#include "diffmodels.cuh"
#include "runmcmc.h"
#include "samples.h"
#include "options.h"

#include <host_vector.h>
#include <device_vector.h> 

#include <time.h>
#include <sys/time.h>
#include "init_gpu.h"
#include <fstream>

using namespace Xfibres;

void xfibres_gpu(	//INPUT
			const Matrix			datam,
			const Matrix			bvecs,
			const Matrix			bvals,
			const Matrix	 		gradm, 
			int				idpart)
{

	xfibresOptions& opts = xfibresOptions::getInstance();

	int nvox = datam.Ncols();
	int ndirections = datam.Nrows();
	int nfib= opts.nfibres.value(); 

	if(nvox>0){
		thrust::host_vector<double> datam_host, bvecs_host, bvals_host, alpha_host, beta_host, params_host;
		thrust::host_vector<float> tau_host;
		vector<ColumnVector> datam_vec;
		vector<Matrix> bvecs_vec, bvals_vec;

		///// FIT /////
		prepare_data_gpu_FIT(datam,bvecs,bvals,gradm,datam_vec, bvecs_vec, bvals_vec, datam_host, bvecs_host,  bvals_host, alpha_host, beta_host, params_host, tau_host);	

		thrust::device_vector<double> datam_gpu=datam_host;
		thrust::device_vector<double> bvecs_gpu=bvecs_host;
		thrust::device_vector<double> bvals_gpu=bvals_host;	
		thrust::device_vector<double> params_gpu=params_host;
		thrust::host_vector<int> vox_repeat;	//contains the id's of voxels repeated
		vox_repeat.resize(nvox);
		int nrepeat=0;

		fit(datam_vec,bvecs_vec,bvals_vec,datam_host,bvecs_host,bvals_host,datam_gpu,bvecs_gpu,bvals_gpu,ndirections,params_gpu,vox_repeat,nrepeat);

		if(opts.rician.value()){
			calculate_tau(datam_gpu,params_gpu,bvecs_gpu,bvals_gpu,vox_repeat,nrepeat, ndirections, tau_host);
		}

		bvecs_gpu.clear();		//free bvecs_gpu
		bvecs_gpu.shrink_to_fit();
	
		//////   RUN MCMC  //////
		thrust::host_vector<double> signals_host,isosignals_host;
		thrust::host_vector<FibreGPU> fibres_host;
		thrust::host_vector<MultifibreGPU> multifibres_host;
	
		prepare_data_gpu_MCMC(nvox, ndirections, nfib, signals_host, isosignals_host, fibres_host, multifibres_host);

		thrust::device_vector<double> signals_gpu=signals_host;
		thrust::device_vector<double> isosignals_gpu=isosignals_host;
		thrust::device_vector<FibreGPU> fibres_gpu=fibres_host;
		thrust::device_vector<MultifibreGPU> multifibres_gpu=multifibres_host;
		thrust::device_vector<float> tau_gpu = tau_host;
		thrust::device_vector<double> alpha_gpu=alpha_host;
		thrust::device_vector<double> beta_gpu=beta_host;

		init_Fibres_Multifibres(datam_gpu, params_gpu, tau_gpu, bvals_gpu, alpha_gpu, beta_gpu, ndirections, fibres_gpu, multifibres_gpu, signals_gpu, isosignals_gpu);

		srand(opts.seed.value());  //randoms seed

		runmcmc_burnin(datam_gpu, bvals_gpu, alpha_gpu, beta_gpu, ndirections, rand(), fibres_gpu,multifibres_gpu, signals_gpu, isosignals_gpu);

		thrust::device_vector<int> multirecords_gpu;
		thrust::device_vector<float> rf0_gpu, rtau_gpu, rs0_gpu, rd_gpu, rdstd_gpu, rth_gpu, rph_gpu, rf_gpu, rlikelihood_energy_gpu;

		prepare_data_gpu_MCMC_record(nvox, multirecords_gpu, rf0_gpu, rtau_gpu, rs0_gpu, rd_gpu, rdstd_gpu, rth_gpu, rph_gpu, rf_gpu, rlikelihood_energy_gpu);

		runmcmc_record(datam_gpu, bvals_gpu, alpha_gpu,beta_gpu, fibres_gpu, multifibres_gpu, signals_gpu, isosignals_gpu, ndirections, rand(), multirecords_gpu, rf0_gpu, rtau_gpu, rs0_gpu, rd_gpu, rdstd_gpu, rth_gpu, rph_gpu, rf_gpu, rlikelihood_energy_gpu);

		/////// FINISH ALL VOXELS  ///////
		record_finish_voxels(multirecords_gpu, rf0_gpu, rtau_gpu, rs0_gpu, rd_gpu, rdstd_gpu, rth_gpu, rph_gpu, rf_gpu, rlikelihood_energy_gpu, nvox, ndirections, idpart);
	}else{
		/////// FINISH EMPTY SLICE  ///////	
		Samples samples(nvox,ndirections);
		samples.save(idpart);
	}
}


// Correct bvals/bvecs accounting for Gradient Nonlinearities
// ColumnVector grad_nonlin has 9 entries, corresponding to the 3 components of each of the x,y and z gradient deviation
void correct_bvals_bvecs(const Matrix& bvals,const Matrix& bvecs, const ColumnVector& grad_nonlin, Matrix& bvals_c, Matrix& bvecs_c){
  	bvals_c=bvals; bvecs_c=bvecs;
  	Matrix L(3,3);  //gradient coil tensor
  	float mag;
  	L(1,1)=grad_nonlin(1);  L(1,2)=grad_nonlin(4);  L(1,3)=grad_nonlin(7);
  	L(2,1)=grad_nonlin(2);  L(2,2)=grad_nonlin(5);  L(2,3)=grad_nonlin(8);
  	L(3,1)=grad_nonlin(3);  L(3,2)=grad_nonlin(6);  L(3,3)=grad_nonlin(9);

  	IdentityMatrix Id(3);
  
  	for (int l=1; l<=bvals.Ncols(); l++){
    		if (bvals(1,l)>0){ //do not correct b0s
     		 	bvecs_c.Column(l)=(Id+L)*bvecs.Column(l);
      			mag=sqrt(bvecs_c(1,l)*bvecs_c(1,l)+bvecs_c(2,l)*bvecs_c(2,l)+bvecs_c(3,l)*bvecs_c(3,l));
      			if (mag!=0)
				bvecs_c.Column(l)=bvecs_c.Column(l)/mag;
      			bvals_c(1,l)=mag*mag*bvals(1,l);//mag^2 as b propto |G|^2
    		}
  	}
}

//////   FIT  //////
void fit(	//INPUT
		const vector<ColumnVector> 	datam_vec, 
		const vector<Matrix> 		bvecs_vec,
		const vector<Matrix> 		bvals_vec,
		thrust::host_vector<double> 	datam_host,
		thrust::host_vector<double>	bvecs_host, 
		thrust::host_vector<double>	bvals_host,
		thrust::device_vector<double> 	datam_gpu, 
		thrust::device_vector<double>	bvecs_gpu, 
		thrust::device_vector<double>	bvals_gpu,
		int 				ndirections,
		//OUTPUT
		thrust::device_vector<double>&	params_gpu,
		thrust::host_vector<int>&	vox_repeat,	//for get residuals with or withot f0
		int&				nrepeat)
{
	cout << "----------------------------------------------------- " << "\n"; 
   	cout << "------------------- FIT IN GPU ---------------------- " << "\n"; 
   	cout << "----------------------------------------------------- " << "\n"; 

	struct timeval t1,t2;
   	double time;
   	gettimeofday(&t1,NULL);

	xfibresOptions& opts = xfibresOptions::getInstance();
	int nvox = datam_vec.size();
	int nfib= opts.nfibres.value();
	int nparams_fit = 2+3*opts.nfibres.value();
	if(opts.modelnum.value()==2) nparams_fit++;
	if(opts.f0.value()) nparams_fit++;

	if(opts.modelnum.value()==1){
		if(opts.nonlin.value()){ 
			fit_PVM_single(datam_vec,bvecs_vec,bvals_vec,datam_gpu,bvecs_gpu,bvals_gpu,ndirections,opts.f0.value(),params_gpu);

			if (opts.f0.value()){
				float md,mf,f0;	
				thrust::host_vector<double> params_host;
				params_host.resize(nvox*nparams_fit);
				thrust::copy(params_gpu.begin(), params_gpu.end(), params_host.begin());	
				for(int vox=0;vox<nvox;vox++){			
					md = params_host[vox*nparams_fit+(1)];
					mf = params_host[vox*nparams_fit+(2)];
					f0 = params_host[vox*nparams_fit+(nparams_fit-1)];
					if ((opts.nfibres.value()>0 && mf<0.05) || md>0.007 || f0>0.4){		//if true we need to repeat this voxel
						vox_repeat[nrepeat]=vox;
						nrepeat++;
					}
				}
				if(nrepeat>0){
					//prepare structures for the voxels that need to be reprocessed
					vector<ColumnVector> 	datam_repeat_vec; 
					vector<Matrix> 		bvecs_repeat_vec;
					vector<Matrix> 		bvals_repeat_vec;
					thrust::host_vector<double> 	datam_repeat_host;
					thrust::host_vector<double> 	bvecs_repeat_host;	
					thrust::host_vector<double> 	bvals_repeat_host;	
					thrust::host_vector<double> 	params_repeat_host;	
								
					prepare_data_gpu_FIT_repeat(datam_host, bvecs_host, bvals_host, vox_repeat, nrepeat, ndirections, datam_repeat_vec, bvecs_repeat_vec, bvals_repeat_vec, datam_repeat_host, bvecs_repeat_host, bvals_repeat_host, params_repeat_host);

					thrust::device_vector<double> datam_repeat_gpu=datam_repeat_host;
					thrust::device_vector<double> bvecs_repeat_gpu=bvecs_repeat_host;
					thrust::device_vector<double> bvals_repeat_gpu=bvals_repeat_host;	
					thrust::device_vector<double> params_repeat_gpu=params_repeat_host;
				
		 			fit_PVM_single(datam_repeat_vec,bvecs_repeat_vec,bvals_repeat_vec,datam_repeat_gpu,bvecs_repeat_gpu,bvals_repeat_gpu,ndirections,false,params_repeat_gpu);
					thrust::copy(params_repeat_gpu.begin(), params_repeat_gpu.end(), params_repeat_host.begin());	
					//mix all the parameteres: repeated and not repeated
					mix_params(params_repeat_host,vox_repeat, nrepeat, nvox, params_gpu);
				}
	  		}
		}else{
			fit_PVM_single_c(datam_vec,bvecs_vec,bvals_vec,datam_gpu,bvecs_gpu,bvals_gpu,ndirections,opts.f0.value(),params_gpu);

			if (opts.f0.value()){
				float md,mf,f0;	
				thrust::host_vector<double> params_host;
				params_host.resize(nvox*nparams_fit);
				thrust::copy(params_gpu.begin(), params_gpu.end(), params_host.begin());	
				for(int vox=0;vox<nvox;vox++){		
					md = params_host[vox*nparams_fit+(1)];
					mf = params_host[vox*nparams_fit+(2)];
					f0 = params_host[vox*nparams_fit+(nparams_fit-1)];
					if ((opts.nfibres.value()>0 && mf<0.05) || md>0.007 || f0>0.4){		//if true we need to repeat this voxel
						vox_repeat[nrepeat]=vox;
						nrepeat++;
					}
				}
				if(nrepeat>0){
					//prepare structures for the voxels that need to be reprocessed
					vector<ColumnVector> 	datam_repeat_vec; 
					vector<Matrix> 		bvecs_repeat_vec;
					vector<Matrix> 		bvals_repeat_vec;
					thrust::host_vector<double> 	datam_repeat_host;
					thrust::host_vector<double> 	bvecs_repeat_host;	
					thrust::host_vector<double> 	bvals_repeat_host;	
					thrust::host_vector<double> 	params_repeat_host;	
								
					prepare_data_gpu_FIT_repeat(datam_host, bvecs_host, bvals_host, vox_repeat, nrepeat, ndirections, datam_repeat_vec, bvecs_repeat_vec, bvals_repeat_vec, datam_repeat_host, bvecs_repeat_host, bvals_repeat_host, params_repeat_host);

					thrust::device_vector<double> datam_repeat_gpu=datam_repeat_host;
					thrust::device_vector<double> bvecs_repeat_gpu=bvecs_repeat_host;
					thrust::device_vector<double> bvals_repeat_gpu=bvals_repeat_host;	
					thrust::device_vector<double> params_repeat_gpu=params_repeat_host;
				
		 			fit_PVM_single_c(datam_repeat_vec,bvecs_repeat_vec,bvals_repeat_vec,datam_repeat_gpu,bvecs_repeat_gpu,bvals_repeat_gpu,ndirections,false,params_repeat_gpu);
					thrust::copy(params_repeat_gpu.begin(), params_repeat_gpu.end(), params_repeat_host.begin());	

					//mix all the parameteres: repeated and not repeated
					mix_params(params_repeat_host ,vox_repeat, nrepeat, nvox, params_gpu);
				}
	  		}
		}
	}else{
      		//model 2 : non-mono-exponential
		fit_PVM_single_c(datam_vec,bvecs_vec,bvals_vec,datam_gpu,bvecs_gpu,bvals_gpu,ndirections,opts.f0.value(),params_gpu);
	
		fit_PVM_multi(datam_gpu,bvecs_gpu,bvals_gpu,nvox,ndirections,opts.f0.value(),params_gpu);	

		if (opts.f0.value()){
				float md,mf,f0;	
				thrust::host_vector<double> params_host;
				params_host.resize(nvox*nparams_fit);
				thrust::copy(params_gpu.begin(), params_gpu.end(), params_host.begin());	
				for(int vox=0;vox<nvox;vox++){			
					md = params_host[vox*nparams_fit+(1)];
					mf = params_host[vox*nparams_fit+(3)];
					f0 = params_host[vox*nparams_fit+(nparams_fit-1)];
					if ((opts.nfibres.value()>0 && mf<0.05) || md>0.007 || f0>0.4){		//if true we need to repeat this voxel
						vox_repeat[nrepeat]=vox;
						nrepeat++;
					}
				}
				if(nrepeat>0){
					//prepare structures for the voxels that need to be reprocessed
					vector<ColumnVector> 	datam_repeat_vec; 
					vector<Matrix> 		bvecs_repeat_vec;
					vector<Matrix> 		bvals_repeat_vec;
					thrust::host_vector<double> 	datam_repeat_host;
					thrust::host_vector<double> 	bvecs_repeat_host;	
					thrust::host_vector<double> 	bvals_repeat_host;	
					thrust::host_vector<double> 	params_repeat_host;		
								
					prepare_data_gpu_FIT_repeat(datam_host, bvecs_host, bvals_host, vox_repeat, nrepeat, ndirections, datam_repeat_vec, bvecs_repeat_vec, bvals_repeat_vec, datam_repeat_host, bvecs_repeat_host,  bvals_repeat_host, params_repeat_host);

					thrust::device_vector<double> datam_repeat_gpu=datam_repeat_host;
					thrust::device_vector<double> bvecs_repeat_gpu=bvecs_repeat_host;
					thrust::device_vector<double> bvals_repeat_gpu=bvals_repeat_host;	
					thrust::device_vector<double> params_repeat_gpu=params_repeat_host;
				
		 			fit_PVM_single_c(datam_repeat_vec,bvecs_repeat_vec,bvals_repeat_vec,datam_repeat_gpu,bvecs_repeat_gpu,bvals_repeat_gpu,ndirections,false,params_repeat_gpu);

					fit_PVM_multi(datam_repeat_gpu,bvecs_repeat_gpu,bvals_repeat_gpu,nrepeat,ndirections,false,params_repeat_gpu);	
					thrust::copy(params_repeat_gpu.begin(), params_repeat_gpu.end(), params_repeat_host.begin());	
		
					//mix all the parameteres: repeated and not repeated
					mix_params(params_repeat_host ,vox_repeat, nrepeat,  nvox, params_gpu);
				}
	  		}	
	}

	gettimeofday(&t2,NULL);
    	time=timeval_diff(&t2,&t1);
   	cout << "TIME TOTAL: " << time << " seconds\n"; 
	cout << "--------------------------------------------" << "\n\n" ; 
}

//prepare the structures for copy all neccesary data to FIT in GPU
void prepare_data_gpu_FIT(	//INPUT
				const Matrix				datam,
				const Matrix				bvecs,
				const Matrix				bvals,
				const Matrix	 			gradm, 
				//OUTPUT
				vector<ColumnVector>&			datam_vec,
				vector<Matrix>&				bvecs_vec,
				vector<Matrix>&				bvals_vec,
				thrust::host_vector<double>&   		datam_host,	//data prepared for copy to GPU
				thrust::host_vector<double>&		bvecs_host,				
				thrust::host_vector<double>&		bvals_host,
				thrust::host_vector<double>&		alpha_host,
				thrust::host_vector<double>&		beta_host,
				thrust::host_vector<double>&		params_host,
				thrust::host_vector<float>&		tau_host)
{
	xfibresOptions& opts = xfibresOptions::getInstance();
	int nvox = datam.Ncols(); 
	int ndirections = datam.Nrows(); 

	datam_vec.resize(nvox);
	datam_host.resize(nvox*ndirections); 
	for(int vox=0;vox<nvox;vox++){
		datam_vec[vox]=datam.Column(vox+1);
		for(int j=0;j<ndirections;j++){
			datam_host[vox*ndirections+j]=datam(j+1,vox+1);
		}
	}

	bvecs_vec.resize(nvox);
	bvals_vec.resize(nvox);
	bvecs_host.resize(nvox*bvecs.Nrows()*bvecs.Ncols());
	bvals_host.resize(nvox*bvals.Ncols());

	alpha_host.resize(nvox*bvecs.Ncols());
	beta_host.resize(nvox*bvecs.Ncols());
	
	ColumnVector alpha,beta;

	if (opts.grad_file.set()){
		for(int vox=0;vox<nvox;vox++){
			correct_bvals_bvecs(bvals,bvecs, gradm.Column(vox+1),bvals_vec[vox],bvecs_vec[vox]); //correct for gradient nonlinearities
 			MISCMATHS::cart2sph(bvecs_vec[vox],alpha,beta);
			
			for(int dir=0;dir<ndirections;dir++){
				bvecs_host[vox*ndirections*3+dir] = bvecs_vec[vox](1,dir+1);
				bvecs_host[vox*ndirections*3+ndirections+dir] = bvecs_vec[vox](2,dir+1);
				bvecs_host[vox*ndirections*3+ndirections*2+dir] = bvecs_vec[vox](3,dir+1);
				bvals_host[vox*ndirections+dir] = bvals_vec[vox](1,dir+1);

				alpha_host[vox*ndirections+dir] = alpha(dir+1);
        			beta_host[vox*ndirections+dir] = beta(dir+1);
			}
		}
		
	}else{
 		MISCMATHS::cart2sph(bvecs,alpha,beta);

		for(int vox=0;vox<nvox;vox++){
			bvecs_vec[vox]=bvecs;
			bvals_vec[vox]=bvals;
			for(int dir=0;dir<ndirections;dir++){
				bvecs_host[vox*ndirections*3+dir] = bvecs(1,dir+1);
				bvecs_host[vox*ndirections*3+ndirections+dir] = bvecs(2,dir+1);
				bvecs_host[vox*ndirections*3+ndirections*2+dir] = bvecs(3,dir+1);
        			bvals_host[vox*ndirections+dir] = bvals(1,dir+1);
			
				alpha_host[vox*ndirections+dir] = alpha(dir+1);
        			beta_host[vox*ndirections+dir] = beta(dir+1);
			}
		}
	}
	
	int nfib= opts.nfibres.value();
	int nparams;

	if(opts.f0.value()) nparams=3+nfib*3;
	else nparams=2+nfib*3;	
	if(opts.modelnum.value()==2) nparams++;

	params_host.resize(nvox*nparams);
	tau_host.resize(nvox);
}

//prepare the structures for copy all neccesary data to FIT in GPU when is repeated because f0. Only some voxels
void prepare_data_gpu_FIT_repeat(	//INPUT
					thrust::host_vector<double>   		datam_host,	
					thrust::host_vector<double>		bvecs_host,				
					thrust::host_vector<double>		bvals_host,
					thrust::host_vector<int>		vox_repeat,
					int					nrepeat,
					int					ndirections,
					//OUTPUT
					vector<ColumnVector>&			datam_repeat_vec,
					vector<Matrix>&				bvecs_repeat_vec,
					vector<Matrix>&				bvals_repeat_vec,
					thrust::host_vector<double>&   		datam_repeat_host,	//data prepared for copy to GPU
					thrust::host_vector<double>&		bvecs_repeat_host,				
					thrust::host_vector<double>&		bvals_repeat_host,
					thrust::host_vector<double>&		params_repeat_host)
{
	xfibresOptions& opts = xfibresOptions::getInstance();

	ColumnVector datam(ndirections);
	Matrix	bvecs(3,ndirections);
	Matrix	bvals(1,ndirections);

	datam_repeat_vec.resize(nrepeat);
	datam_repeat_host.resize(nrepeat*ndirections); 
	bvecs_repeat_vec.resize(nrepeat);
	bvals_repeat_vec.resize(nrepeat);
	bvecs_repeat_host.resize(nrepeat*3*ndirections);
	bvals_repeat_host.resize(nrepeat*ndirections);

	for(int vox=0;vox<nrepeat;vox++){
		for(int dir=0;dir<ndirections;dir++){
			datam(dir+1)= datam_host[vox_repeat[vox]*ndirections+dir]; 
			datam_repeat_host[vox*ndirections+dir]=datam_host[vox_repeat[vox]*ndirections+dir];

			bvecs_repeat_host[vox*ndirections*3+dir] = bvecs_host[vox_repeat[vox]*ndirections*3+dir];
			bvecs_repeat_host[vox*ndirections*3+ndirections+dir] = bvecs_host[vox_repeat[vox]*ndirections*3+ndirections+dir];
			bvecs_repeat_host[vox*ndirections*3+ndirections*2+dir] = bvecs_host[vox_repeat[vox]*ndirections*3+ndirections*2+dir];
        		bvals_repeat_host[vox*ndirections+dir] = bvals_host[vox_repeat[vox]*ndirections+dir];
			
			bvecs(1,dir+1)= bvecs_host[vox_repeat[vox]*ndirections*3+dir];
			bvecs(2,dir+1)= bvecs_host[vox_repeat[vox]*ndirections*3+ndirections+dir];
			bvecs(3,dir+1)= bvecs_host[vox_repeat[vox]*ndirections*3+ndirections*2+dir];
			bvals(1,dir+1)= bvals_host[vox_repeat[vox]*ndirections+dir];
		}
		datam_repeat_vec[vox]=datam;	
		bvecs_repeat_vec[vox]=bvecs;
		bvals_repeat_vec[vox]=bvals;
	}
	
	int nfib= opts.nfibres.value();
	int nparams;

	nparams=2+nfib*3;	
	if(opts.modelnum.value()==2) nparams++;

	params_repeat_host.resize(nrepeat*nparams);
}


void mix_params(	//INPUT
			thrust::host_vector<double>   		params_repeat_host,
			thrust::host_vector<int>		vox_repeat,
			int					nrepeat,
			int					nvox,
			//INPUT-OUTPUT
			thrust::device_vector<double>&   	params_gpu)
{
	xfibresOptions& opts = xfibresOptions::getInstance();
	int nfib= opts.nfibres.value();
	int nparams = 2+3*opts.nfibres.value();
	if(opts.modelnum.value()==2) nparams++;

	thrust::host_vector<double> params_host;
	params_host.resize(nvox*(nparams+1));
	thrust::copy(params_gpu.begin(), params_gpu.end(), params_host.begin());	

	for(int vox=0;vox<nrepeat;vox++){
		for(int par=0;par<nparams;par++){
			params_host[vox_repeat[vox]*(nparams+1)+par] = params_repeat_host[vox*nparams+par]; //(nparams+1) to count f0
		}
		params_host[vox_repeat[vox]*(nparams+1)+nparams] = 0.001;	//pvmf0=0.001
	}
	thrust::copy(params_host.begin(), params_host.end(), params_gpu.begin());	
}

void prepare_data_gpu_MCMC(	//INPUT
				int 					nvox,
				int					ndirections,
				int 					nfib,
				//OUTPUT
				thrust::host_vector<double>&		signals_host,
				thrust::host_vector<double>&		isosignals_host,
				thrust::host_vector<FibreGPU>& 		fibres_host,
				thrust::host_vector<MultifibreGPU>& 	multifibres_host)
{ 	
	signals_host.resize(nvox*nfib*ndirections);
	isosignals_host.resize(nvox*ndirections);	
	fibres_host.resize(nvox*nfib);	
	multifibres_host.resize(nvox);
}

void prepare_data_gpu_MCMC_record(	//INPUT
					int 						nvox,
					//OUTPUT
					thrust::device_vector<int>&			multirecords_gpu,
					thrust::device_vector<float>&			rf0_gpu,
					thrust::device_vector<float>&			rtau_gpu,
					thrust::device_vector<float>&			rs0_gpu,
					thrust::device_vector<float>&			rd_gpu,
					thrust::device_vector<float>&			rdstd_gpu,
					thrust::device_vector<float>&			rth_gpu,
					thrust::device_vector<float>&			rph_gpu,
					thrust::device_vector<float>&			rf_gpu,
					thrust::device_vector<float>&			rlikelihood_energy_gpu)
{ 	
	xfibresOptions& opts = xfibresOptions::getInstance();

	int nfib = opts.nfibres.value();	
	int nrecords = (opts.njumps.value()/opts.sampleevery.value());   
	
	multirecords_gpu.resize(nvox*nrecords); 
	if(opts.f0.value()) rf0_gpu.resize(nvox*nrecords); 
	if(opts.rician.value()) rtau_gpu.resize(nvox*nrecords);  
	rs0_gpu.resize(nvox*nrecords);  
	rd_gpu.resize(nvox*nrecords);
	if(opts.modelnum.value()==2) rdstd_gpu.resize(nvox*nrecords);  
	rth_gpu.resize(nvox*nrecords*nfib);  
	rph_gpu.resize(nvox*nrecords*nfib);  
	rf_gpu.resize(nvox*nrecords*nfib);  
	rlikelihood_energy_gpu.resize(nvox*nrecords); 
}

void record_finish_voxels(	//INPUT
				thrust::device_vector<int>&			multirecords_gpu,
				thrust::device_vector<float>&			rf0_gpu,
				thrust::device_vector<float>&			rtau_gpu,
				thrust::device_vector<float>&			rs0_gpu,
				thrust::device_vector<float>&			rd_gpu,
				thrust::device_vector<float>&			rdstd_gpu,
				thrust::device_vector<float>&			rth_gpu,
				thrust::device_vector<float>&			rph_gpu,
				thrust::device_vector<float>&			rf_gpu,
				thrust::device_vector<float>&			rlikelihood_energy_gpu,
				int 						nvox,
				int						ndirections,
				int						idpart)
{
	xfibresOptions& opts = xfibresOptions::getInstance();

	int nfib = opts.nfibres.value();	
	int nrecords = (opts.njumps.value()/opts.sampleevery.value());   

	thrust::host_vector<int> multirecords_host;
	thrust::host_vector<float> rf0_host,rtau_host,rs0_host,rd_host,rdstd_host,rth_host,rph_host,rf_host,rlikelihood_energy_host;

	multirecords_host.resize(nvox*nrecords);
	rf0_host.resize(nvox*nrecords);
	rtau_host.resize(nvox*nrecords);
	rs0_host.resize(nvox*nrecords);
	rd_host.resize(nvox*nrecords);
	rdstd_host.resize(nvox*nrecords);
	rth_host.resize(nvox*nfib*nrecords);
	rph_host.resize(nvox*nfib*nrecords);
	rf_host.resize(nvox*nfib*nrecords);
	rlikelihood_energy_host.resize(nvox*nrecords);

	thrust::copy(multirecords_gpu.begin(), multirecords_gpu.end(), multirecords_host.begin());
	if(opts.f0.value()) thrust::copy(rf0_gpu.begin(), rf0_gpu.end(), rf0_host.begin());
	if(opts.rician.value()) thrust::copy(rtau_gpu.begin(), rtau_gpu.end(), rtau_host.begin());
	thrust::copy(rs0_gpu.begin(), rs0_gpu.end(), rs0_host.begin());
	thrust::copy(rd_gpu.begin(), rd_gpu.end(), rd_host.begin());
	if(opts.modelnum.value()==2) thrust::copy(rdstd_gpu.begin(), rdstd_gpu.end(), rdstd_host.begin());
	thrust::copy(rth_gpu.begin(), rth_gpu.end(), rth_host.begin());
	thrust::copy(rph_gpu.begin(), rph_gpu.end(), rph_host.begin());
	thrust::copy(rf_gpu.begin(), rf_gpu.end(), rf_host.begin());	
	thrust::copy(rlikelihood_energy_gpu.begin(), rlikelihood_energy_gpu.end(), rlikelihood_energy_host.begin());	

	Samples samples(nvox,ndirections);

	float ard,arf0,artau,ardstd,ars0,arlikelihood_energy;	
	float *arth = new float[nfib];
    	float *arph = new float[nfib]; 
    	float *arf = new float[nfib];
	int samp;

	for(int vox=0;vox<nvox;vox++){
		for(int rec=0;rec<nrecords;rec++){	
			ard=rd_host[(vox*nrecords)+rec];
			if(opts.f0.value()){	
				arf0=rf0_host[(vox*nrecords)+rec];
			}

			if(opts.rician.value()){	
				artau=rtau_host[(vox*nrecords)+rec];
			}

			if(opts.modelnum.value()==2){	
				ardstd=rdstd_host[(vox*nrecords)+rec];
			}
		
			ars0=rs0_host[(vox*nrecords)+rec];
		
			arlikelihood_energy=rlikelihood_energy_host[(vox*nrecords)+rec];	

			for(int j=0;j<nfib;j++){
				arth[j]=rth_host[(vox*nfib*nrecords)+(j*nrecords)+rec];
				arph[j]=rph_host[(vox*nfib*nrecords)+(j*nrecords)+rec];
				arf[j]=rf_host[(vox*nfib*nrecords)+(j*nrecords)+rec];

			}

			samp=multirecords_host[(vox*nrecords)+rec];	
			samples.record(ard,arf0,artau,ardstd,ars0,arlikelihood_energy,arth,arph,arf,vox+1,samp);
		}	
		samples.finish_voxel(vox+1);
   	}

	samples.save(idpart);
}

