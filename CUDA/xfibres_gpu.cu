#include "hip/hip_runtime.h"
#include "newmat.h"
#include "newimage/newimageall.h"
#include "xfibresoptions.h"

#include "xfibres_gpu.cuh"
#include "diffmodels.cuh"
#include "runmcmc.h"
#include "samples.h"
#include "options.h"

#include <host_vector.h>
#include <device_vector.h> 

#include <time.h>
#include <sys/time.h>
#include "init_gpu.h"
#include <fstream>

using namespace Xfibres;

void xfibres_gpu(	//INPUT
			const Matrix			datam,
			const Matrix			bvecs,
			const Matrix			bvals,
			const Matrix	 		gradm, 
			const Matrix 			Qform, 
			const Matrix 			Qform_inv,
			const NEWIMAGE::volume<int> 	vol2matrixkey,
			const NEWMAT::Matrix		matrix2volkey,
			const NEWIMAGE::volume<float>	mask,
			const int 			slice,
			const char*			subjdir)
{
	//write num of slice in a string for log file
	char slice_str[8];
	char aux[8];
	sprintf(slice_str,"%d",slice);
	while(strlen(slice_str)<4){
		strcpy(aux,"0");
		strcat(aux,slice_str);
		strcpy(slice_str,aux);
	}
	string gpu_log(subjdir);		//logfile
	gpu_log += ".bedpostX/logs/times_gpu_";
	gpu_log += slice_str;

	xfibresOptions& opts = xfibresOptions::getInstance();

	///// FIT /////
	thrust::host_vector<double> datam_host, bvecs_host, bvals_host, alpha_host, beta_host, params_host;
	thrust::host_vector<float> tau_host;
	vector<ColumnVector> datam_vec;
	vector<Matrix> bvecs_vec, bvals_vec;

	prepare_data_gpu_FIT(datam,bvecs,bvals,gradm,Qform,Qform_inv,datam_vec, bvecs_vec, bvals_vec, datam_host, bvecs_host,  bvals_host, alpha_host, beta_host, params_host, tau_host);	

	int nvox = datam.Ncols();
	int nfib= opts.nfibres.value(); 

	thrust::device_vector<double> datam_gpu=datam_host;
	thrust::device_vector<double> bvecs_gpu=bvecs_host;
	thrust::device_vector<double> bvals_gpu=bvals_host;	
	thrust::device_vector<double> params_gpu=params_host;
	thrust::host_vector<int> vox_repeat;	//contains the id's of voxels repeated
	vox_repeat.resize(nvox);
	int nrepeat=0;

	fit(datam_vec,bvecs_vec,bvals_vec,datam_host,bvecs_host,bvals_host,datam_gpu,bvecs_gpu,bvals_gpu,gpu_log,params_gpu,vox_repeat,nrepeat);

	if(opts.rician.value()){
		calculate_tau(datam_gpu,params_gpu,bvecs_gpu,bvals_gpu,vox_repeat,nrepeat,gpu_log, tau_host);
	}

	bvecs_gpu.clear();		//free bvecs_gpu
	bvecs_gpu.shrink_to_fit();
	
	//////   RUN MCMC  //////
	thrust::host_vector<double> signals_host,isosignals_host;
	thrust::host_vector<FibreGPU> fibres_host;
	thrust::host_vector<MultifibreGPU> multifibres_host;
	
	prepare_data_gpu_MCMC(nvox, nfib, signals_host, isosignals_host, fibres_host, multifibres_host);

	thrust::device_vector<double> signals_gpu=signals_host;
	thrust::device_vector<double> isosignals_gpu=isosignals_host;
	thrust::device_vector<FibreGPU> fibres_gpu=fibres_host;
	thrust::device_vector<MultifibreGPU> multifibres_gpu=multifibres_host;
	thrust::device_vector<float> tau_gpu = tau_host;
	thrust::device_vector<double> alpha_gpu=alpha_host;
	thrust::device_vector<double> beta_gpu=beta_host;

	init_Fibres_Multifibres(datam_gpu, params_gpu, tau_gpu, bvals_gpu, alpha_gpu, beta_gpu, gpu_log, fibres_gpu, multifibres_gpu, signals_gpu, isosignals_gpu);

	srand(opts.seed.value());
	//double seed1 = rand(); 
	//double seed2 = rand(); 

	runmcmc_burnin(datam_gpu, bvals_gpu, alpha_gpu, beta_gpu, rand(), gpu_log, fibres_gpu,multifibres_gpu, signals_gpu, isosignals_gpu);

	thrust::device_vector<int> multirecords_gpu;
	thrust::device_vector<float> rf0_gpu, rtau_gpu, rs0_gpu, rd_gpu, rdstd_gpu, rth_gpu, rph_gpu, rf_gpu, rlikelihood_energy_gpu;

	prepare_data_gpu_MCMC_record(nvox, multirecords_gpu, rf0_gpu, rtau_gpu, rs0_gpu, rd_gpu, rdstd_gpu, rth_gpu, rph_gpu, rf_gpu, rlikelihood_energy_gpu);

	runmcmc_record(datam_gpu, bvals_gpu, alpha_gpu,beta_gpu, fibres_gpu, multifibres_gpu, signals_gpu, isosignals_gpu, rand(), gpu_log, multirecords_gpu, rf0_gpu, rtau_gpu, rs0_gpu, rd_gpu, rdstd_gpu, rth_gpu, rph_gpu, rf_gpu, rlikelihood_energy_gpu);

	/////// FINISH ALL VOXELS  ///////
	record_finish_voxels(vol2matrixkey, matrix2volkey, mask, multirecords_gpu, rf0_gpu, rtau_gpu, rs0_gpu, rd_gpu, rdstd_gpu, rth_gpu, rph_gpu, rf_gpu, rlikelihood_energy_gpu, nvox);
}


// Correct bvals/bvecs accounting for Gradient Nonlinearities
// ColumnVector grad_nonlin has 9 entries, corresponding to the 3 components of each of the x,y and z gradient deviation
void correct_bvals_bvecs(const Matrix& bvals,const Matrix& bvecs, const ColumnVector& grad_nonlin, const Matrix& Qform, const Matrix& Qform_inv, Matrix& bvals_c, Matrix& bvecs_c){
  	bvals_c=bvals; bvecs_c=bvecs;
  	Matrix L(3,3);  //gradient coil tensor
  	float mag;
  	L(1,1)=grad_nonlin(1);  L(1,2)=grad_nonlin(4);  L(1,3)=grad_nonlin(7);
  	L(2,1)=grad_nonlin(2);  L(2,2)=grad_nonlin(5);  L(2,3)=grad_nonlin(8);
  	L(3,1)=grad_nonlin(3);  L(3,2)=grad_nonlin(6);  L(3,3)=grad_nonlin(9);

  	IdentityMatrix Id(3);
  
  	for (int l=1; l<=bvals.Ncols(); l++){
    		if (bvals(1,l)>0){ //do not correct b0s
      			//Rotate bvecs to scanner's coordinate system
      			ColumnVector bvec_tmp(3);
      			bvec_tmp=Qform*bvecs.Column(l);
      			bvec_tmp(1)=-bvec_tmp(1); //Sign-flip X coordinate

      			//Correct for grad-nonlin in scanner's coordinate system
     		 	bvecs_c.Column(l)=(Id+L)*bvec_tmp;//bvecs.Column(l);
      			mag=sqrt(bvecs_c(1,l)*bvecs_c(1,l)+bvecs_c(2,l)*bvecs_c(2,l)+bvecs_c(3,l)*bvecs_c(3,l));
      			if (mag!=0)
				bvecs_c.Column(l)=bvecs_c.Column(l)/mag;
      			bvals_c(1,l)=mag*mag*bvals(1,l);
      			bvec_tmp=bvecs_c.Column(l);

      			//Rotate corrected bvecs back to voxel coordinate system
      			bvec_tmp(1)=-bvec_tmp(1); //Sign-flip X coordinate
      			bvecs_c.Column(l)=Qform_inv*bvec_tmp;
    		}
  	}
}

//////   FIT  //////
void fit(	//INPUT
		const vector<ColumnVector> 	datam_vec, 
		const vector<Matrix> 		bvecs_vec,
		const vector<Matrix> 		bvals_vec,
		thrust::host_vector<double> 	datam_host,
		thrust::host_vector<double>	bvecs_host, 
		thrust::host_vector<double>	bvals_host,
		thrust::device_vector<double> 	datam_gpu, 
		thrust::device_vector<double>	bvecs_gpu, 
		thrust::device_vector<double>	bvals_gpu,
		string 				output_file,
		//OUTPUT
		thrust::device_vector<double>&	params_gpu,
		thrust::host_vector<int>&	vox_repeat,	//for get residuals with or withot f0
		int&				nrepeat)
{
	std::ofstream myfile;
	myfile.open (output_file.data(), ios::out | ios::app );
	myfile << "----------------------------------------------------- " << "\n"; 
   	myfile << "------------------- FIT IN GPU ---------------------- " << "\n"; 
   	myfile << "----------------------------------------------------- " << "\n"; 

	struct timeval t1,t2;
   	double time;
   	gettimeofday(&t1,NULL);

	xfibresOptions& opts = xfibresOptions::getInstance();
	int nvox = datam_vec.size();
	int nfib= opts.nfibres.value();
	int nparams_fit = 2+3*opts.nfibres.value();
	if(opts.modelnum.value()==2) nparams_fit++;
	if(opts.f0.value()) nparams_fit++;

	if(opts.modelnum.value()==1){
		if(opts.nonlin.value()){ 
			fit_PVM_single(datam_vec,bvecs_vec,bvals_vec,datam_gpu,bvecs_gpu,bvals_gpu,opts.f0.value(),params_gpu);

			if (opts.f0.value()){
				float md,mf,f0;	

				for(int vox=0;vox<nvox;vox++){			
					md = params_gpu[vox*nparams_fit+(1)];
					mf = params_gpu[vox*nparams_fit+(2)];
					f0 = params_gpu[vox*nparams_fit+(nparams_fit-1)];
					if ((opts.nfibres.value()>0 && mf<0.05) || md>0.007 || f0>0.4){		//if true we need to repeat this voxel
						vox_repeat[nrepeat]=vox;
						nrepeat++;
					}
				}
				if(nrepeat>0){
					//prepare structures for the voxels that need to be reprocessed
					vector<ColumnVector> 	datam_repeat_vec; 
					vector<Matrix> 		bvecs_repeat_vec;
					vector<Matrix> 		bvals_repeat_vec;
					thrust::host_vector<double> 	datam_repeat_host;
					thrust::host_vector<double> 	bvecs_repeat_host;	
					thrust::host_vector<double> 	bvals_repeat_host;	
					thrust::host_vector<double> 	params_repeat_host;	
								
					prepare_data_gpu_FIT_repeat(datam_host, bvecs_host, bvals_host, vox_repeat, nrepeat, datam_repeat_vec, bvecs_repeat_vec, bvals_repeat_vec, datam_repeat_host, bvecs_repeat_host, bvals_repeat_host, params_repeat_host);

					thrust::device_vector<double> datam_repeat_gpu=datam_repeat_host;
					thrust::device_vector<double> bvecs_repeat_gpu=bvecs_repeat_host;
					thrust::device_vector<double> bvals_repeat_gpu=bvals_repeat_host;	
					thrust::device_vector<double> params_repeat_gpu=params_repeat_host;
				
		 			fit_PVM_single(datam_repeat_vec, bvecs_repeat_vec, bvals_repeat_vec, datam_repeat_gpu, bvecs_repeat_gpu, bvals_repeat_gpu, false, params_repeat_gpu);

					//mix all the parameteres: repeated and not repeated
					mix_params(params_repeat_gpu,vox_repeat, nrepeat, params_gpu);
				}
	  		}
		}else{
			fit_PVM_single_c(datam_vec,bvecs_vec,bvals_vec,datam_gpu,bvecs_gpu,bvals_gpu,opts.f0.value(),params_gpu);

			if (opts.f0.value()){
				float md,mf,f0;	
	
				for(int vox=0;vox<nvox;vox++){		
					md = params_gpu[vox*nparams_fit+(1)];
					mf = params_gpu[vox*nparams_fit+(2)];
					f0 = params_gpu[vox*nparams_fit+(nparams_fit-1)];
					if ((opts.nfibres.value()>0 && mf<0.05) || md>0.007 || f0>0.4){		//if true we need to repeat this voxel
						vox_repeat[nrepeat]=vox;
						nrepeat++;
					}
				}
				if(nrepeat>0){
					//prepare structures for the voxels that need to be reprocessed
					vector<ColumnVector> 	datam_repeat_vec; 
					vector<Matrix> 		bvecs_repeat_vec;
					vector<Matrix> 		bvals_repeat_vec;
					thrust::host_vector<double> 	datam_repeat_host;
					thrust::host_vector<double> 	bvecs_repeat_host;	
					thrust::host_vector<double> 	bvals_repeat_host;	
					thrust::host_vector<double> 	params_repeat_host;	
								
					prepare_data_gpu_FIT_repeat(datam_host, bvecs_host, bvals_host, vox_repeat, nrepeat, datam_repeat_vec, bvecs_repeat_vec, bvals_repeat_vec, datam_repeat_host, bvecs_repeat_host, bvals_repeat_host, params_repeat_host);

					thrust::device_vector<double> datam_repeat_gpu=datam_repeat_host;
					thrust::device_vector<double> bvecs_repeat_gpu=bvecs_repeat_host;
					thrust::device_vector<double> bvals_repeat_gpu=bvals_repeat_host;	
					thrust::device_vector<double> params_repeat_gpu=params_repeat_host;
				
		 			fit_PVM_single_c(datam_repeat_vec, bvecs_repeat_vec, bvals_repeat_vec, datam_repeat_gpu, bvecs_repeat_gpu, bvals_repeat_gpu, false, params_repeat_gpu);

					//mix all the parameteres: repeated and not repeated
					mix_params(params_repeat_gpu ,vox_repeat, nrepeat, params_gpu);
				}
	  		}
		}
	}else{
      		//model 2 : non-mono-exponential
		fit_PVM_single_c(datam_vec,bvecs_vec,bvals_vec,datam_gpu,bvecs_gpu,bvals_gpu,opts.f0.value(),params_gpu);
	
		fit_PVM_multi(datam_gpu,bvecs_gpu,bvals_gpu,nvox,opts.f0.value(),params_gpu);	

		if (opts.f0.value()){
				float md,mf,f0;	
	
				for(int vox=0;vox<nvox;vox++){			
					md = params_gpu[vox*nparams_fit+(1)];
					mf = params_gpu[vox*nparams_fit+(3)];
					f0 = params_gpu[vox*nparams_fit+(nparams_fit-1)];
					if ((opts.nfibres.value()>0 && mf<0.05) || md>0.007 || f0>0.4){		//if true we need to repeat this voxel
						vox_repeat[nrepeat]=vox;
						nrepeat++;
					}
				}
				if(nrepeat>0){
					//prepare structures for the voxels that need to be reprocessed
					vector<ColumnVector> 	datam_repeat_vec; 
					vector<Matrix> 		bvecs_repeat_vec;
					vector<Matrix> 		bvals_repeat_vec;
					thrust::host_vector<double> 	datam_repeat_host;
					thrust::host_vector<double> 	bvecs_repeat_host;	
					thrust::host_vector<double> 	bvals_repeat_host;	
					thrust::host_vector<double> 	params_repeat_host;		
								
					prepare_data_gpu_FIT_repeat(datam_host, bvecs_host, bvals_host, vox_repeat, nrepeat, datam_repeat_vec, bvecs_repeat_vec, bvals_repeat_vec, datam_repeat_host, bvecs_repeat_host,  bvals_repeat_host, params_repeat_host);

					thrust::device_vector<double> datam_repeat_gpu=datam_repeat_host;
					thrust::device_vector<double> bvecs_repeat_gpu=bvecs_repeat_host;
					thrust::device_vector<double> bvals_repeat_gpu=bvals_repeat_host;	
					thrust::device_vector<double> params_repeat_gpu=params_repeat_host;
				
		 			fit_PVM_single_c(datam_repeat_vec, bvecs_repeat_vec, bvals_repeat_vec, datam_repeat_gpu, bvecs_repeat_gpu, bvals_repeat_gpu, false, params_repeat_gpu);

					fit_PVM_multi(datam_repeat_gpu, bvecs_repeat_gpu, bvals_repeat_gpu, nrepeat, false, params_repeat_gpu);
		
					//mix all the parameteres: repeated and not repeated
					mix_params(params_repeat_gpu ,vox_repeat, nrepeat,params_gpu);
				}
	  		}	
	}

	gettimeofday(&t2,NULL);
    	time=timeval_diff(&t2,&t1);
   	myfile << "TIME TOTAL: " << time << " seconds\n"; 
	myfile << "--------------------------------------------" << "\n\n" ; 
	myfile.close();

}

//prepare the structures for copy all neccesary data to FIT in GPU
void prepare_data_gpu_FIT(	//INPUT
				const Matrix				datam,
				const Matrix				bvecs,
				const Matrix				bvals,
				const Matrix	 			gradm, 
				const Matrix 				Qform, 
				const Matrix 				Qform_inv,
				//OUTPUT
				vector<ColumnVector>&			datam_vec,
				vector<Matrix>&				bvecs_vec,
				vector<Matrix>&				bvals_vec,
				thrust::host_vector<double>&   		datam_host,	//data prepared for copy to GPU
				thrust::host_vector<double>&		bvecs_host,				
				thrust::host_vector<double>&		bvals_host,
				thrust::host_vector<double>&		alpha_host,
				thrust::host_vector<double>&		beta_host,
				thrust::host_vector<double>&		params_host,
				thrust::host_vector<float>&		tau_host)
{
	xfibresOptions& opts = xfibresOptions::getInstance();
	int nvox = datam.Ncols(); 

	datam_vec.resize(nvox);
	datam_host.resize(nvox*NDIRECTIONS); 
	for(int vox=0;vox<nvox;vox++){
		datam_vec[vox]=datam.Column(vox+1);
		for(int j=0;j<NDIRECTIONS;j++){
			datam_host[vox*NDIRECTIONS+j]=datam(j+1,vox+1);
		}
	}

	bvecs_vec.resize(nvox);
	bvals_vec.resize(nvox);
	bvecs_host.resize(nvox*bvecs.Nrows()*bvecs.Ncols());
	bvals_host.resize(nvox*bvals.Ncols());

	alpha_host.resize(nvox*bvecs.Ncols());
	beta_host.resize(nvox*bvecs.Ncols());
	
	ColumnVector alpha,beta;

	if (opts.grad_file.set()){
		for(int vox=0;vox<nvox;vox++){
			correct_bvals_bvecs(bvals,bvecs, gradm.Column(vox+1),Qform,Qform_inv,bvals_vec[vox],bvecs_vec[vox]); //correct for gradient nonlinearities
 			MISCMATHS::cart2sph(bvecs_vec[vox],alpha,beta);
			
			for(int dir=0;dir<NDIRECTIONS;dir++){
				bvecs_host[vox*NDIRECTIONS*3+dir] = bvecs_vec[vox](1,dir+1);
				bvecs_host[vox*NDIRECTIONS*3+NDIRECTIONS+dir] = bvecs_vec[vox](2,dir+1);
				bvecs_host[vox*NDIRECTIONS*3+NDIRECTIONS*2+dir] = bvecs_vec[vox](3,dir+1);
				bvals_host[vox*NDIRECTIONS+dir] = bvals_vec[vox](1,dir+1);

				alpha_host[vox*NDIRECTIONS+dir] = alpha(dir+1);
        			beta_host[vox*NDIRECTIONS+dir] = beta(dir+1);
			}
		}
		
	}else{
 		MISCMATHS::cart2sph(bvecs,alpha,beta);

		for(int vox=0;vox<nvox;vox++){
			bvecs_vec[vox]=bvecs;
			bvals_vec[vox]=bvals;
			for(int dir=0;dir<NDIRECTIONS;dir++){
				bvecs_host[vox*NDIRECTIONS*3+dir] = bvecs(1,dir+1);
				bvecs_host[vox*NDIRECTIONS*3+NDIRECTIONS+dir] = bvecs(2,dir+1);
				bvecs_host[vox*NDIRECTIONS*3+NDIRECTIONS*2+dir] = bvecs(3,dir+1);
        			bvals_host[vox*NDIRECTIONS+dir] = bvals(1,dir+1);
			
				alpha_host[vox*NDIRECTIONS+dir] = alpha(dir+1);
        			beta_host[vox*NDIRECTIONS+dir] = beta(dir+1);
			}
		}
	}
	
	int nfib= opts.nfibres.value();
	int nparams;

	if(opts.f0.value()) nparams=3+nfib*3;
	else nparams=2+nfib*3;	
	if(opts.modelnum.value()==2) nparams++;

	params_host.resize(nvox*nparams);
	tau_host.resize(nvox);
}

//prepare the structures for copy all neccesary data to FIT in GPU when is repeated because f0. Only some voxels
void prepare_data_gpu_FIT_repeat(	//INPUT
					thrust::host_vector<double>   		datam_host,	
					thrust::host_vector<double>		bvecs_host,				
					thrust::host_vector<double>		bvals_host,
					thrust::host_vector<int>		vox_repeat,
					int					nrepeat,
					//OUTPUT
					vector<ColumnVector>&			datam_repeat_vec,
					vector<Matrix>&				bvecs_repeat_vec,
					vector<Matrix>&				bvals_repeat_vec,
					thrust::host_vector<double>&   		datam_repeat_host,	//data prepared for copy to GPU
					thrust::host_vector<double>&		bvecs_repeat_host,				
					thrust::host_vector<double>&		bvals_repeat_host,
					thrust::host_vector<double>&		params_repeat_host)
{
	xfibresOptions& opts = xfibresOptions::getInstance();

	ColumnVector datam(NDIRECTIONS);
	Matrix	bvecs(3,NDIRECTIONS);
	Matrix	bvals(1,NDIRECTIONS);

	datam_repeat_vec.resize(nrepeat);
	datam_repeat_host.resize(nrepeat*NDIRECTIONS); 
	bvecs_repeat_vec.resize(nrepeat);
	bvals_repeat_vec.resize(nrepeat);
	bvecs_repeat_host.resize(nrepeat*3*NDIRECTIONS);
	bvals_repeat_host.resize(nrepeat*NDIRECTIONS);

	for(int vox=0;vox<nrepeat;vox++){
		for(int dir=0;dir<NDIRECTIONS;dir++){
			datam(dir+1)= datam_host[vox_repeat[vox]*NDIRECTIONS+dir]; 
			datam_repeat_host[vox*NDIRECTIONS+dir]=datam_host[vox_repeat[vox]*NDIRECTIONS+dir];

			bvecs_repeat_host[vox*NDIRECTIONS*3+dir] = bvecs_host[vox_repeat[vox]*NDIRECTIONS*3+dir];
			bvecs_repeat_host[vox*NDIRECTIONS*3+NDIRECTIONS+dir] = bvecs_host[vox_repeat[vox]*NDIRECTIONS*3+NDIRECTIONS+dir];
			bvecs_repeat_host[vox*NDIRECTIONS*3+NDIRECTIONS*2+dir] = bvecs_host[vox_repeat[vox]*NDIRECTIONS*3+NDIRECTIONS*2+dir];
        		bvals_repeat_host[vox*NDIRECTIONS+dir] = bvals_host[vox_repeat[vox]*NDIRECTIONS+dir];
			
			bvecs(1,dir+1)= bvecs_host[vox_repeat[vox]*NDIRECTIONS*3+dir];
			bvecs(2,dir+1)= bvecs_host[vox_repeat[vox]*NDIRECTIONS*3+NDIRECTIONS+dir];
			bvecs(3,dir+1)= bvecs_host[vox_repeat[vox]*NDIRECTIONS*3+NDIRECTIONS*2+dir];
			bvals(1,dir+1)= bvals_host[vox_repeat[vox]*NDIRECTIONS+dir];
		}
		datam_repeat_vec[vox]=datam;	
		bvecs_repeat_vec[vox]=bvecs;
		bvals_repeat_vec[vox]=bvals;
	}
	
	int nfib= opts.nfibres.value();
	int nparams;

	nparams=2+nfib*3;	
	if(opts.modelnum.value()==2) nparams++;

	params_repeat_host.resize(nrepeat*nparams);
}


void mix_params(	//INPUT
			thrust::device_vector<double>   		params_repeat_gpu,
			thrust::host_vector<int>			vox_repeat,
			int						nrepeat,
			//INPUT-OUTPUT
			thrust::device_vector<double>&   		params_gpu)
{
	xfibresOptions& opts = xfibresOptions::getInstance();
	int nfib= opts.nfibres.value();
	int nparams;
	nparams=2+nfib*3;	
	if(opts.modelnum.value()==2) nparams++;

	for(int vox=0;vox<nrepeat;vox++){
		for(int par=0;par<nparams;par++){
			params_gpu[vox_repeat[vox]*(nparams+1)+par] = params_repeat_gpu[vox*nparams+par]; //(nparams+1) to count f0
		}
		params_gpu[vox_repeat[vox]*(nparams+1)+nparams] = 0.001;	//pvmf0=0.001
	}
}

void prepare_data_gpu_MCMC(	//INPUT
				int 					nvox,
				int 					nfib,
				//OUTPUT
				thrust::host_vector<double>&		signals_host,
				thrust::host_vector<double>&		isosignals_host,
				thrust::host_vector<FibreGPU>& 		fibres_host,
				thrust::host_vector<MultifibreGPU>& 	multifibres_host)
{ 	
	signals_host.resize(nvox*nfib*NDIRECTIONS);
	isosignals_host.resize(nvox*NDIRECTIONS);	
	fibres_host.resize(nvox*nfib);	
	multifibres_host.resize(nvox);
}

void prepare_data_gpu_MCMC_record(	//INPUT
					int 						nvox,
					//OUTPUT
					thrust::device_vector<int>&			multirecords_gpu,
					thrust::device_vector<float>&			rf0_gpu,
					thrust::device_vector<float>&			rtau_gpu,
					thrust::device_vector<float>&			rs0_gpu,
					thrust::device_vector<float>&			rd_gpu,
					thrust::device_vector<float>&			rdstd_gpu,
					thrust::device_vector<float>&			rth_gpu,
					thrust::device_vector<float>&			rph_gpu,
					thrust::device_vector<float>&			rf_gpu,
					thrust::device_vector<float>&			rlikelihood_energy_gpu)
{ 	
	xfibresOptions& opts = xfibresOptions::getInstance();

	int nfib = opts.nfibres.value();	
	int nrecords = (opts.njumps.value()/opts.sampleevery.value());   
	
	multirecords_gpu.resize(nvox*nrecords); 
	if(opts.f0.value()) rf0_gpu.resize(nvox*nrecords); 
	if(opts.rician.value()) rtau_gpu.resize(nvox*nrecords);  
	rs0_gpu.resize(nvox*nrecords);  
	rd_gpu.resize(nvox*nrecords);
	if(opts.modelnum.value()==2) rdstd_gpu.resize(nvox*nrecords);  
	rth_gpu.resize(nvox*nrecords*nfib);  
	rph_gpu.resize(nvox*nrecords*nfib);  
	rf_gpu.resize(nvox*nrecords*nfib);  
	rlikelihood_energy_gpu.resize(nvox*nrecords); 
}

void record_finish_voxels(	//INPUT
				const NEWIMAGE::volume<int> 			vol2matrixkey,
				const NEWMAT::Matrix				matrix2volkey,
				const NEWIMAGE::volume<float>			mask,
				thrust::device_vector<int>&			multirecords_gpu,
				thrust::device_vector<float>&			rf0_gpu,
				thrust::device_vector<float>&			rtau_gpu,
				thrust::device_vector<float>&			rs0_gpu,
				thrust::device_vector<float>&			rd_gpu,
				thrust::device_vector<float>&			rdstd_gpu,
				thrust::device_vector<float>&			rth_gpu,
				thrust::device_vector<float>&			rph_gpu,
				thrust::device_vector<float>&			rf_gpu,
				thrust::device_vector<float>&			rlikelihood_energy_gpu,
				int 						nvox)
{
	xfibresOptions& opts = xfibresOptions::getInstance();

	int nfib = opts.nfibres.value();	
	int nrecords = (opts.njumps.value()/opts.sampleevery.value());   

	thrust::host_vector<int> multirecords_host;
	thrust::host_vector<float> rf0_host,rtau_host,rs0_host,rd_host,rdstd_host,rth_host,rph_host,rf_host,rlikelihood_energy_host;

	multirecords_host.resize(nvox*nrecords);
	rf0_host.resize(nvox*nrecords);
	rtau_host.resize(nvox*nrecords);
	rs0_host.resize(nvox*nrecords);
	rd_host.resize(nvox*nrecords);
	rdstd_host.resize(nvox*nrecords);
	rth_host.resize(nvox*nfib*nrecords);
	rph_host.resize(nvox*nfib*nrecords);
	rf_host.resize(nvox*nfib*nrecords);
	rlikelihood_energy_host.resize(nvox*nrecords);

	thrust::copy(multirecords_gpu.begin(), multirecords_gpu.end(), multirecords_host.begin());
	if(opts.f0.value()) thrust::copy(rf0_gpu.begin(), rf0_gpu.end(), rf0_host.begin());
	if(opts.rician.value()) thrust::copy(rtau_gpu.begin(), rtau_gpu.end(), rtau_host.begin());
	thrust::copy(rs0_gpu.begin(), rs0_gpu.end(), rs0_host.begin());
	thrust::copy(rd_gpu.begin(), rd_gpu.end(), rd_host.begin());
	if(opts.modelnum.value()==2) thrust::copy(rdstd_gpu.begin(), rdstd_gpu.end(), rdstd_host.begin());
	thrust::copy(rth_gpu.begin(), rth_gpu.end(), rth_host.begin());
	thrust::copy(rph_gpu.begin(), rph_gpu.end(), rph_host.begin());
	thrust::copy(rf_gpu.begin(), rf_gpu.end(), rf_host.begin());	
	thrust::copy(rlikelihood_energy_gpu.begin(), rlikelihood_energy_gpu.end(), rlikelihood_energy_host.begin());	

	Samples samples(vol2matrixkey,matrix2volkey,nvox,NDIRECTIONS);

	float ard,arf0,artau,ardstd,ars0,arlikelihood_energy;	
	float *arth = new float[nfib];
    	float *arph = new float[nfib]; 
    	float *arf = new float[nfib];
	int samp;

	for(int vox=0;vox<nvox;vox++){
		for(int rec=0;rec<nrecords;rec++){	
			ard=rd_host[(vox*nrecords)+rec];
			if(opts.f0.value()){	
				arf0=rf0_host[(vox*nrecords)+rec];
			}

			if(opts.rician.value()){	
				artau=rtau_host[(vox*nrecords)+rec];
			}

			if(opts.modelnum.value()==2){	
				ardstd=rdstd_host[(vox*nrecords)+rec];
			}
		
			ars0=rs0_host[(vox*nrecords)+rec];
		
			arlikelihood_energy=rlikelihood_energy_host[(vox*nrecords)+rec];	

			for(int j=0;j<nfib;j++){
				arth[j]=rth_host[(vox*nfib*nrecords)+(j*nrecords)+rec];
				arph[j]=rph_host[(vox*nfib*nrecords)+(j*nrecords)+rec];
				arf[j]=rf_host[(vox*nfib*nrecords)+(j*nrecords)+rec];

			}

			samp=multirecords_host[(vox*nrecords)+rec];	
			samples.record(ard,arf0,artau,ardstd,ars0,arlikelihood_energy,arth,arph,arf,vox+1,samp);
		}	
		samples.finish_voxel(vox+1);
   	}

	samples.save(mask);
}

