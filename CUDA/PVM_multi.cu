#include "hip/hip_runtime.h"
/*  PVM_multi.cu

    Tim Behrens, Saad Jbabdi, Stam Sotiropoulos, Moises Hernandez  - FMRIB Image Analysis Group

    Copyright (C) 2005 University of Oxford  */

/*  CCOPYRIGHT  */

#include "diffmodels_utils.h"
#include "levenberg_marquardt.cu"
#include "options.h"

/////////////////////////////////////
/////////////////////////////////////
/// 	    PVM_multi	 	  /// 
/////////////////////////////////////
/////////////////////////////////////

__device__ inline double isoterm_PVM_multi(const int pt,const double _a,const double _b, const double *bvals){
	return exp(double(-_a*log(double(1+bvals[pt]*_b))));
}

__device__ inline double isoterm_a_PVM_multi(const int pt,const double _a,const double _b, const double *bvals){
    	return  -log(double(1+bvals[pt]*_b))*exp(double(-_a*log(double(1+bvals[pt]*_b))));
}

__device__ inline double isoterm_b_PVM_multi(const int pt,const double _a,const double _b, const double *bvals){
      	return -_a*bvals[pt]/(1+bvals[pt]*_b)*exp(double(-_a*log(double(1+bvals[pt]*_b))));
}

__device__ inline double anisoterm_PVM_multi(const int pt,const double _a,const double _b,const double3 x,const double *bvecs, const double *bvals){
	double dp = bvecs[pt]*x.x+bvecs[NDIRECTIONS+pt]*x.y+bvecs[(2*NDIRECTIONS)+pt]*x.z;
  	return exp(double(-_a*log(double(1+bvals[pt]*_b*(dp*dp)))));
}
 
__device__ inline double anisoterm_a_PVM_multi(const int pt,const double _a,const double _b,const double3 x,const double *bvecs, const double *bvals){
	double dp = bvecs[pt]*x.x+bvecs[NDIRECTIONS+pt]*x.y+bvecs[(2*NDIRECTIONS)+pt]*x.z;
  	return -log(double(1+bvals[pt]*(dp*dp)*_b))* exp(double(-_a*log(double(1+bvals[pt]*(dp*dp)*_b))));
}

__device__ inline double anisoterm_b_PVM_multi(const int pt,const double _a,const double _b,const double3 x,const double *bvecs, const double *bvals){
  	double dp = bvecs[pt]*x.x+bvecs[NDIRECTIONS+pt]*x.y+bvecs[(2*NDIRECTIONS)+pt]*x.z;
  	return (-_a*bvals[pt]*(dp*dp)/ (1+bvals[pt]*(dp*dp)*_b)*exp(double(-_a*log(double(1+bvals[pt]*(dp*dp)*_b)))));
}

__device__ inline double anisoterm_th_PVM_multi(const int pt,const double _a,const double _b,const double3 x,const double _th,const double _ph,const double *bvecs, const double *bvals){
  	double dp = bvecs[pt]*x.x+bvecs[NDIRECTIONS+pt]*x.y+bvecs[(2*NDIRECTIONS)+pt]*x.z;
  	double dp1 = cos(double(_th))* (bvecs[pt]*cos(double(_ph)) + bvecs[NDIRECTIONS+pt]*sin(double(_ph))) - bvecs[(2*NDIRECTIONS)+pt]*sin(double(_th));
	return  (-_a*_b*bvals[pt]/(1+bvals[pt]*(dp*dp)*_b)*exp(double(-_a*log(double(1+bvals[pt]*(dp*dp)*_b))))*2*dp*dp1);	
}

__device__ inline double anisoterm_ph_PVM_multi(const int pt,const double _a,const double _b,const double3 x,const double _th,const double _ph,const double *bvecs, const double *bvals){
  	double dp = bvecs[pt]*x.x+bvecs[NDIRECTIONS+pt]*x.y+bvecs[(2*NDIRECTIONS)+pt]*x.z;
  	double dp1 = sin(double(_th))* (-bvecs[pt]*sin(double(_ph)) + bvecs[NDIRECTIONS+pt]*cos(double(_ph)));
  	return  (-_a*_b*bvals[pt]/(1+bvals[pt]*(dp*dp)*_b)*exp(double(-_a*log(double(1+bvals[pt]*(dp*dp)*_b))))*2*dp*dp1);
}

//in diffmodel.cc
__device__ void fix_fsum_PVM_multi(	//INPUT 
					bool m_include_f0, 
					int nfib,
					int nparams,
					//INPUT - OUTPUT){
					double *params)
{
  	double sumf=0;
  	if (m_include_f0) 
    		sumf=params[nparams-1];
  	for(int i=0;i<nfib;i++){
    		if (params[3+(i*3)]==0) 
			params[3+(i*3)]=FSMALL_gpu;
    		sumf+=params[3+(i*3)];
    		if(sumf>=1){
			for(int j=i;j<nfib;j++)
				params[3+(j*3)]=FSMALL_gpu;
			break;
		}
	}
}

//in diffmodel.cc
__device__ void sort_PVM_multi(int nfib,int nparams,double* params)
{
	double temp_f, temp_th, temp_ph;
	// Order vector descending using f parameters as index
	for(int i=1; i<(nfib); i++){ 
    		for(int j=0; j<(nfib-i); j++){ 
      			if (params[3+j*3] < params[3+i*3]){ 
        			temp_f = params[3+j*3];
				temp_th = params[3+j*3+1];
				temp_ph = params[3+j*3+2];
        			params[3+j*3] = params[3+i*3]; 
				params[3+j*3+1] = params[3+i*3+1]; 
				params[3+j*3+2] = params[3+i*3+2]; 
        			params[3+i*3] = temp_f; 
				params[3+i*3+1] = temp_th; 
				params[3+i*3+2] = temp_ph; 
      			} 
    		} 
  	} 
}

//in diffmodels.cc -- for calculate residuals
__device__ void  forwardModel_PVM_multi(	//INPUT
						const double* 		p,
						const double*		bvecs, 
						const double*		bvals,
						const int		nfib,
						const int 		nparams,
						const bool 		m_include_f0,
						//OUTPUT
						double*		 	predicted_signal)
{
  	for(int i=0;i<NDIRECTIONS;i++){
		predicted_signal[i]=0;		//pred = 0;
	}
  	double val;
  	double _a = abs(p[1]);
  	double _b = abs(p[2]);
  	////////////////////////////////////
  	double fs[NFIBRES];
  	double x[NFIBRES*3];	
  	double sumf=0;
	double3 x2;
  	for(int k=0;k<nfib;k++){
    		int kk = 3+3*k;
	    	fs[k] = x2f_gpu(p[kk]);
	    	sumf += fs[k];
		x[k*3] = sin(p[kk+1])*cos(p[kk+2]);
    		x[k*3+1] = sin(p[kk+1])*sin(p[kk+2]);
    		x[k*3+2] = cos(p[kk+1]);
  	}
  	////////////////////////////////////
  	for(int i=0;i<NDIRECTIONS;i++){
    		val = 0.0;
    		for(int k=0;k<nfib;k++){
			x2.x=x[k*3];
			x2.y=x[k*3+1];
			x2.z=x[k*3+2];	 
      			val += fs[k]*anisoterm_PVM_multi(i,_a,_b,x2,bvecs,bvals);
    		}	
    		if (m_include_f0){
      			double temp_f0=x2f_gpu(p[nparams-1]);
      			predicted_signal[i] = abs(p[0])*(temp_f0+(1-sumf-temp_f0)*isoterm_PVM_multi(i,_a,_b,bvals)+val);
    		} 
    		else
      			predicted_signal[i] = abs(p[0])*((1-sumf)*isoterm_PVM_multi(i,_a,_b,bvals)+val); 
  		}   
}

//in diffmodels.cc -- for calculate residuals
__device__ void get_prediction_PVM_multi(	//INPUT
						const double*	params,
						const double*	bvecs, 
						const double*	bvals,
						const int 	nfib,
						const int 	nparams,
						const bool 	m_include_f0,
						//OUTPUT
						double* 	predicted_signal)
{
	//m_s0-myparams[0] 	m_d-myparams[1] 	m_d_std-myparams[2]		m_f-m_th-m_ph-myparams[3,4,5,6 etc..]   	m_f0-myparams[nparams-1]
  	double p[NPARAMS];
  	p[0] = params[0];
  	p[1] = params[1]*params[1]/params[2]/params[2];		//m_d*m_d/m_d_std/m_d_std;
  	p[2] = params[2]*params[2]/params[1];			//m_d_std*m_d_std/m_d; // =1/beta
  	for(int k=0;k<nfib;k++){
    		int kk = 3+3*k;
    		p[kk]   = f2x_gpu(params[kk]);
    		p[kk+1] = params[kk+1];
    		p[kk+2] = params[kk+2];
  	}
  	if (m_include_f0)
    		p[nparams-1]=f2x_gpu(params[nparams-1]);
  	forwardModel_PVM_multi(p,bvecs,bvals,nfib,nparams,m_include_f0,predicted_signal);
}

//cost function PVM_multi
__device__ double cf_PVM_multi(		//INPUT
					const double*		params,
					const double*		mdata,
					const double*		bvecs, 
					const double*		bvals,
					const int 		nparams,
					const bool 		m_include_f0)
{
	double cfv = 0.0;
  	double err;	
	double _a= abs(params[1]);
	double _b= abs(params[2]); 
	double fs[NFIBRES];    
	double x[NFIBRES*3];	
	double sumf=0;
	double3 x2;

	for(int k=0;k<NFIBRES;k++){
		int kk = 3+3*(k);
		fs[k] = x2f_gpu(params[kk]);
    		sumf += fs[k];
    		x[k*3] = sin(params[kk+1])*cos(params[kk+2]);
    		x[k*3+1] = sin(params[kk+1])*sin(params[kk+2]);
    		x[k*3+2] = cos(params[kk+1]);
  	}
	for(int i=0;i<NDIRECTIONS;i++){
    		err = 0.0;
    		for(int k=0;k<NFIBRES;k++){
			x2.x=x[k*3];
			x2.y=x[k*3+1];
			x2.z=x[k*3+2];	 
			err += fs[k]*anisoterm_PVM_multi(i,_a,_b,x2,bvecs,bvals); 
    		}
		if(m_include_f0){
			double temp_f0=x2f_gpu(params[nparams-1]);
			err = (abs(params[0])*(temp_f0+((1-sumf-temp_f0)*isoterm_PVM_multi(i,_a,_b,bvals)+err)))-mdata[i];
		}else{
			err = abs(params[0])*((1-sumf)*isoterm_PVM_multi(i,_a,_b,bvals)+err)-mdata[i];
		}
		cfv += err*err;
  	}  
	return(cfv);
}


//gradient function PVM_multi
__device__ void grad_PVM_multi(		//INPUT
					const double*		params,
					const double*		mdata,
					const double*		bvecs, 
					const double*		bvals,
					const int 		nparams,
					const bool 		m_include_f0,
					//OUTPUT
					double*			grad)
{
  	double _a= abs(params[1]);
  	double _b= abs(params[2]);
  	double fs[NFIBRES];
  	double x[NFIBRES*3];	
  	double3 xx;		
  	double sumf=0;

  	for(int k=0;k<NFIBRES;k++){
    		int kk = 3+3*(k);	
    		fs[k] = x2f_gpu(params[kk]);
    		sumf += fs[k];
    		x[k*3] = sin(params[kk+1])*cos(params[kk+2]);
    		x[k*3+1] = sin(params[kk+1])*sin(params[kk+2]);
    		x[k*3+2] = cos(params[kk+1]);
  	}

  	double J[NPARAMS];
  	double diff;
  	double sig;
  
	for (int p=0;p<nparams;p++) grad[p]=0;

  	for(int i=0;i<NDIRECTIONS;i++){
    		sig = 0;
    		for(int a=0;a<nparams;a++) J[a]=0;
    		for(int k=0;k<NFIBRES;k++){
      			int kk = 3+3*(k);
      			xx.x=x[k*3];
      			xx.y=x[k*3+1];
      			xx.z=x[k*3+2];		
      			sig += fs[k]*anisoterm_PVM_multi(i,_a,_b,xx,bvecs,bvals);
      			J[1] += (params[1]>0?1.0:-1.0)*abs(params[0])*fs[k]*anisoterm_a_PVM_multi(i,_a,_b,xx,bvecs,bvals); 
			J[2] += (params[2]>0?1.0:-1.0)*abs(params[0])*fs[k]*anisoterm_b_PVM_multi(i,_a,_b,xx,bvecs,bvals);
			J[kk] = abs(params[0])*(anisoterm_PVM_multi(i,_a,_b,xx,bvecs,bvals)-isoterm_PVM_multi(i,_a,_b,bvals))*two_pi_gpu*sign_gpu(params[kk])*1/(1+params[kk]*params[kk]); 
      			J[kk+1] = abs(params[0])*fs[k]*anisoterm_th_PVM_multi(i,_a,_b,xx,params[kk+1],params[kk+2],bvecs,bvals);  
      			J[kk+2] = abs(params[0])*fs[k]*anisoterm_ph_PVM_multi(i,_a,_b,xx,params[kk+1],params[kk+2],bvecs,bvals);
    		}
    		if(m_include_f0){
			double temp_f0=x2f_gpu(params[nparams-1]);
			J[nparams-1]= abs(params[0])*(1-isoterm_PVM_multi(i,_a,_b,bvals))*two_pi_gpu*sign_gpu(params[nparams-1])*1/(1+params[nparams-1]*params[nparams-1]);
			sig=abs(params[0])*((temp_f0+(1-sumf-temp_f0)*isoterm_PVM_multi(i,_a,_b,bvals))+sig);
    			J[1] += (params[1]>0?1.0:-1.0)*abs(params[0])*(1-sumf-temp_f0)*isoterm_a_PVM_multi(i,_a,_b,bvals);
			J[2] += (params[2]>0?1.0:-1.0)*abs(params[0])*(1-sumf-temp_f0)*isoterm_b_PVM_multi(i,_a,_b,bvals);
    		}else{
	    		sig = abs(params[0]) * ((1-sumf)*isoterm_PVM_multi(i,_a,_b,bvals)+sig);
	    		J[1] += (params[1]>0?1.0:-1.0)*abs(params[0])*(1-sumf)*isoterm_a_PVM_multi(i,_a,_b,bvals);
	    		J[2] += (params[2]>0?1.0:-1.0)*abs(params[0])*(1-sumf)*isoterm_b_PVM_multi(i,_a,_b,bvals);	
    		}
    
    		diff = sig - mdata[i];
    		J[0] = (params[0]>0?1.0:-1.0)*sig/params[0]; 

		for (int p=0;p<nparams;p++) grad[p] += 2*J[p]*diff;
  	}
}


//hessian function PVM_multi 
__device__ void hess_PVM_multi(		//INPUT
					const double*		params,
					const double*		bvecs, 
					const double*		bvals,
					const int 		nparams,
					const bool 		m_include_f0,
					double*			hess)
{
  	double _a= abs(params[1]);
  	double _b= abs(params[2]);
  	double fs[NFIBRES];
  	double x[NFIBRES*3];	
  	double3 xx;
  	double sumf=0;
 
  	for(int k=0;k<NFIBRES;k++){
    		int kk = 3+3*(k);	
    		fs[k] = x2f_gpu(params[kk]);
    		sumf += fs[k];
    		x[k*3] = sin(params[kk+1])*cos(params[kk+2]);
    		x[k*3+1] = sin(params[kk+1])*sin(params[kk+2]);
    		x[k*3+2] = cos(params[kk+1]);
  	}

  	double J[NPARAMS];
  	double sig;

	for (int p=0;p<nparams;p++){
		for (int p2=0;p2<nparams;p2++){ 
			hess[p*nparams+p2] = 0;
		}
	}
  
  	for(int i=0;i<NDIRECTIONS;i++){
    		sig = 0;
    		for(int a=0;a<nparams;a++) J[a]=0;
    		for(int k=0;k<NFIBRES;k++){
      			int kk = 3+3*(k);
      			xx.x=x[k*3];
      			xx.y=x[k*3+1];
      			xx.z=x[k*3+2];		
      			sig += fs[k]*anisoterm_PVM_multi(i,_a,_b,xx,bvecs,bvals);
      			double cov = two_pi_gpu*sign_gpu(params[kk])*1/(1+params[kk]*params[kk]);	
      			J[1] += (params[1]>0?1.0:-1.0)*abs(params[0])*fs[k]*anisoterm_a_PVM_multi(i,_a,_b,xx,bvecs,bvals);
			J[2] += (params[2]>0?1.0:-1.0)*abs(params[0])*fs[k]*anisoterm_b_PVM_multi(i,_a,_b,xx,bvecs,bvals);
			J[kk] = abs(params[0])*(anisoterm_PVM_multi(i,_a,_b,xx,bvecs,bvals)-isoterm_PVM_multi(i,_a,_b,bvals))*cov;
      			J[kk+1] = abs(params[0])*fs[k]*anisoterm_th_PVM_multi(i,_a,_b,xx,params[kk+1],params[kk+2],bvecs,bvals);
      			J[kk+2] = abs(params[0])*fs[k]*anisoterm_ph_PVM_multi(i,_a,_b,xx,params[kk+1],params[kk+2],bvecs,bvals);
    		}
    		if(m_include_f0){
			double temp_f0=x2f_gpu(params[nparams-1]);
			J[nparams-1]= abs(params[0])*(1-isoterm_PVM_multi(i,_a,_b,bvals))*two_pi_gpu*sign_gpu(params[nparams-1])*1/(1+params[nparams-1]*params[nparams-1]);
	    		sig = abs(params[0])* (temp_f0+(1-sumf-temp_f0)*isoterm_PVM_multi(i,_a,_b,bvals)+sig);
    			J[1] += (params[1]>0?1.0:-1.0)*abs(params[0])*(1-sumf-temp_f0)*isoterm_a_PVM_multi(i,_a,_b,bvals);
			J[2] += (params[2]>0?1.0:-1.0)*abs(params[0])*(1-sumf-temp_f0)*isoterm_b_PVM_multi(i,_a,_b,bvals);
    		}else{
			sig = abs(params[0])*((1-sumf)*isoterm_PVM_multi(i,_a,_b,bvals)+sig);
	    		J[1] += (params[1]>0?1.0:-1.0)*abs(params[0])*(1-sumf)*isoterm_a_PVM_multi(i,_a,_b,bvals);
	    		J[2] += (params[2]>0?1.0:-1.0)*abs(params[0])*(1-sumf)*isoterm_b_PVM_multi(i,_a,_b,bvals);	
    		}
	
    		J[0] = sig/params[0]; 

		for (int p=0;p<nparams;p++){
			for (int p2=p;p2<nparams;p2++){ 
				hess[p*nparams+p2] += 2*(J[p]*J[p2]);
			}
		}
  	}

  	for (int j=0; j<nparams; j++) {
    		for (int i=j+1; i<nparams; i++) {
     			hess[i*nparams+j]=hess[j*nparams+i];	
    		}
  	}
}

//in diffmodel.cc
extern "C" __global__ void fit_PVM_multi_kernel(	//INPUT
							const double* 		data, 
							const double* 		params_PVM_single_c,
							const double* 		bvecs, 
							const double* 		bvals, 
							const int 		nvox, 
							const int 		nfib, 				
							const bool 		m_include_f0,
							//OUTPUT
							double* 		params)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;	
  	if (id >=nvox) { return; }	
	
	int nparams;
	if (m_include_f0)
      		nparams = nfib*3 + 4; 
    	else
      		nparams = nfib*3 + 3;

	int nparams_single_c = nparams-1;

   	double myparams[NPARAMS];
	double myparams_aux[NPARAMS];
   	double mydata[NDIRECTIONS];

   	for(int i=0;i<NDIRECTIONS;i++){
		mydata[i]=data[(id*NDIRECTIONS)+i];
   	}

	for(int i=0;i<nparams_single_c;i++){
		myparams_aux[i]=params_PVM_single_c[(id*nparams_single_c)+i];
   	}

  	myparams[0] = myparams_aux[0];			//pvm1.get_s0();
  	myparams[1] = 1.0;				//start with d=d_std
  	for(int i=0,ii=3;i<nfib;i++,ii+=3){
    		myparams[ii] = f2x_gpu(myparams_aux[ii-1]);
    		myparams[ii+1] = myparams_aux[ii];
    		myparams[ii+2] = myparams_aux[ii+1];
  	}
	myparams[2] = myparams_aux[1];   		//pvm1.get_d();
  	if (m_include_f0)
		myparams[nparams-1]=f2x_gpu(myparams_aux[nparams_single_c-1]);

  	//do the fit
	levenberg_marquardt_PVM_multi_gpu(mydata, &bvecs[id*3*NDIRECTIONS], &bvals[id*NDIRECTIONS], nparams, m_include_f0, myparams);

  	// finalise parameters
	//m_s0-myparams[0] 	m_d-myparams[1] 	m_d_std-myparams[2]		m_f-m_th-m_ph-myparams[3,4,5,6 etc..]   	m_f0-myparams[nparams-1]

	myparams_aux[1] = myparams[1];

  	myparams[1] = abs(myparams_aux[1]*myparams[2]);
  	myparams[2] = sqrt(double(abs(myparams_aux[1]*myparams[2]*myparams[2])));
  	for(int i=3,k=0;k<nfib;i+=3,k++){
    		myparams[i]  = x2f_gpu(myparams[i]);
  	}
  	if (m_include_f0)
    		myparams[nparams-1]=x2f_gpu(myparams[nparams-1]);

	sort_PVM_multi(nfib,nparams,myparams);
  	fix_fsum_PVM_multi(m_include_f0,nfib,nparams,myparams);

	for(int i=0;i<nparams;i++){
		params[(id*nparams)+i] = myparams[i];
		//printf("PARAM[%i]: %.20f\n",i,myparams[i]);
   	}
}

//in diffmodel.cc
extern "C" __global__ void get_residuals_PVM_multi_kernel(	//INPUT
								const double* 		data, 
								const double* 		params,
								const double* 		bvecs, 
								const double* 		bvals, 
								const int 		nvox, 
								const int 		nfib, 
								const bool 		m_include_f0,
								const bool* 		includes_f0,
								//OUTPUT
								double*			residuals)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;	
  	if (id >=nvox) { return; }	
	
	int nparams;
	if (m_include_f0)
      		nparams = nfib*3 + 4; 
    	else
      		nparams = nfib*3 + 3;

	bool my_include_f0 = includes_f0[id];

   	double myparams[NPARAMS];
   	double mydata[NDIRECTIONS];

	for(int i=0;i<nparams;i++){
		myparams[i]=params[(id*nparams)+i];
   	}

   	for(int i=0;i<NDIRECTIONS;i++){
		mydata[i]=data[(id*NDIRECTIONS)+i];
   	}

	double predicted_signal[NDIRECTIONS];

	get_prediction_PVM_multi(myparams, &bvecs[id*3*NDIRECTIONS], &bvals[id*NDIRECTIONS], nfib, nparams, my_include_f0, predicted_signal);

	for(int i=0;i<NDIRECTIONS;i++){		//residuals=m_data-predicted_signal;
		residuals[id*NDIRECTIONS+i]= mydata[i] - predicted_signal[i];
	}
}

