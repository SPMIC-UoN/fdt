/*  runmcmc.cu

    Tim Behrens, Saad Jbabdi, Stam Sotiropoulos, Moises Hernandez  - FMRIB Image Analysis Group

    Copyright (C) 2005 University of Oxford  */

/*  CCOPYRIGHT  */


#include <time.h>
#include <sys/time.h>
#include <string>
#include <fstream>
#include <hiprand.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include "xfibresoptions.h"
#include "runmcmc_kernels.cu"
#include "sync_check.h"

#include "init_gpu.h"

using namespace Xfibres;

//////////////////////////////////////////////////////
//   MCMC ON GPU
//////////////////////////////////////////////////////

void init_Fibres_Multifibres(	//INPUT
				thrust::device_vector<float>& 			datam_gpu,
				thrust::device_vector<float>& 			params_gpu,
				thrust::device_vector<float>& 			tau_gpu,
				thrust::device_vector<float>& 			bvals_gpu,
				thrust::device_vector<double>& 			alpha_gpu,
				thrust::device_vector<double>& 			beta_gpu,
				const int 					ndirections,
				std::string 						output_file,
				double 						seed,
				//OUTPUT
				thrust::device_vector<FibreGPU>& 		fibres_gpu,
				thrust::device_vector<MultifibreGPU>& 		multifibres_gpu,
				thrust::device_vector<double>&			signals_gpu,
				thrust::device_vector<double>&			isosignals_gpu,
				thrust::device_vector<hiprandState>&		randStates_gpu)
{
	std::ofstream myfile;
	myfile.open (output_file.data(), std::ios::out | std::ios::app );
   	myfile << "----- MCMC ALGORITHM PART INITIALITATION ON GPU ----- " << "\n";

   	struct timeval t1,t2;
   	double time;
   	gettimeofday(&t1,NULL);

	int nvox = multifibres_gpu.size();

	xfibresOptions& opts = xfibresOptions::getInstance();
	int nfib= opts.nfibres.value();
	int nparams_fit = 2+3*opts.nfibres.value();
	if(opts.modelnum.value()>=2) nparams_fit++;
	if(opts.f0.value()) nparams_fit++;

	thrust::device_vector<double> angtmp_gpu;
	angtmp_gpu.resize(nvox*ndirections*nfib);


	bool gradnonlin = opts.grad_file.set();

	int blocks = nvox/VOXELS_BLOCK_MCMC;
	if(nvox%VOXELS_BLOCK_MCMC) blocks++;
	int nthreads_block = THREADS_VOXEL_MCMC*VOXELS_BLOCK_MCMC;
  	dim3 Dim_Grid_MCMC(blocks, 1);
  	dim3 Dim_Block_MCMC(nthreads_block ,1);	///dimensions for MCMC

	float *datam_ptr = thrust::raw_pointer_cast(datam_gpu.data());
	float *params_ptr = thrust::raw_pointer_cast(params_gpu.data());
	float *tau_ptr = thrust::raw_pointer_cast(tau_gpu.data());
	float *bvals_ptr = thrust::raw_pointer_cast(bvals_gpu.data());
	double *alpha_ptr = thrust::raw_pointer_cast(alpha_gpu.data());
	double *beta_ptr = thrust::raw_pointer_cast(beta_gpu.data());
	FibreGPU *fibres_ptr =  thrust::raw_pointer_cast(fibres_gpu.data());
	MultifibreGPU *multifibres_ptr = thrust::raw_pointer_cast(multifibres_gpu.data());
	double *signals_ptr = thrust::raw_pointer_cast(signals_gpu.data());
	double *isosignals_ptr = thrust::raw_pointer_cast(isosignals_gpu.data());
	double *angtmp_ptr = thrust::raw_pointer_cast(angtmp_gpu.data());
	hiprandState *randStates_ptr = thrust::raw_pointer_cast(randStates_gpu.data());

	int amount_shared = VOXELS_BLOCK_MCMC*((THREADS_VOXEL_MCMC)*sizeof(double) + (3*nfib + 9)*sizeof(float) + sizeof(int));

	myfile << "Shared Memory Used in init_Fibres_Multifibres: " << amount_shared << "\n";

	init_Fibres_Multifibres_kernel<<< Dim_Grid_MCMC, Dim_Block_MCMC, amount_shared>>>(datam_ptr, params_ptr, tau_ptr, bvals_ptr, alpha_ptr, beta_ptr, opts.R_prior_mean.value(), opts.R_prior_std.value(),opts.R_prior_fudge.value(), ndirections, nfib, nparams_fit, opts.modelnum.value(), opts.fudge.value(), opts.f0.value(), opts.rician.value(), opts.ardf0.value(), opts.all_ard.value(), opts.no_ard.value(), gradnonlin, angtmp_ptr, fibres_ptr, multifibres_ptr, signals_ptr, isosignals_ptr);
	sync_check("init_Fibres_Multifibres_kernel");

	// Initialise Randoms
	int total_threads= nvox;
	int blocks_Rand = total_threads/THREADS_BLOCK_RAND;
	if(total_threads%THREADS_BLOCK_RAND) blocks_Rand++;
	dim3 Dim_Grid_Rand(blocks_Rand,1);
	dim3 Dim_Block_Rand(THREADS_BLOCK_RAND,1);
	setup_randoms_kernel <<<Dim_Grid_Rand,Dim_Block_Rand>>>(randStates_ptr,seed,nvox);
	sync_check("Setup_Randoms_kernel");

	gettimeofday(&t2,NULL);
    	time=timeval_diff(&t2,&t1);
   	myfile << "TIME: " << time << " seconds\n";
	myfile << "-----------------------------------------------------" << "\n\n" ;
	myfile.close();
}

void runmcmc_burnin(	//INPUT
			thrust::device_vector<float>& 			datam_gpu,
			thrust::device_vector<float>& 			bvals_gpu,
			thrust::device_vector<double>& 			alpha_gpu,
			thrust::device_vector<double>& 			beta_gpu,
			const int 					ndirections,
			std::string 						output_file,
			//INPUT-OUTPUT
			thrust::device_vector<FibreGPU>& 		fibres_gpu,
			thrust::device_vector<MultifibreGPU>& 		multifibres_gpu,
			thrust::device_vector<double>&			signals_gpu,
			thrust::device_vector<double>&			isosignals_gpu,
			thrust::device_vector<hiprandState>&		randStates_gpu)
{
	xfibresOptions& opts = xfibresOptions::getInstance();

	std::ofstream myfile;
	myfile.open (output_file.data(), std::ios::out | std::ios::app );
   	myfile << "--------- MCMC ALGORITHM PART BURNIN ON GPU --------- " << "\n";

   	struct timeval t_tot1,t_tot2;
   	double time;
   	time=0;

   	gettimeofday(&t_tot1,NULL);

	int nvox = multifibres_gpu.size();
   	int nfib= opts.nfibres.value();
	int nparams;

	bool gradnonlin=opts.grad_file.set();

	if(opts.f0.value()) nparams=3+nfib*3;
	else nparams=2+nfib*3;
	if(opts.modelnum.value()>=2) nparams++;
	if(opts.modelnum.value()==3) nparams++;
	if(opts.rician.value()) nparams++;

	thrust::device_vector<float> recors_null_gpu;
	recors_null_gpu.resize(1);

	thrust::device_vector<double> angtmp_gpu;
	thrust::device_vector<double> oldangtmp_gpu;
	thrust::device_vector<double> oldsignals_gpu;
	thrust::device_vector<double> oldisosignals_gpu;

	angtmp_gpu.resize(nvox*ndirections*nfib);
	oldangtmp_gpu.resize(nvox*ndirections);
	oldsignals_gpu.resize(nvox*ndirections*nfib);
	oldisosignals_gpu.resize(nvox*ndirections);

	myfile << "Processing " << nvox << " voxels \n";

  	int blocks = nvox/VOXELS_BLOCK_MCMC;
	if(nvox%VOXELS_BLOCK_MCMC) blocks++;
	int nthreads_block = THREADS_VOXEL_MCMC*VOXELS_BLOCK_MCMC;
  	dim3 Dim_Grid(blocks, 1);
  	dim3 Dim_Block(nthreads_block,1);	//dimensions for MCMC

   	myfile << "NUM BLOCKS: " << blocks << "\n";
   	myfile << "THREADS PER BLOCK : " << nthreads_block << "\n";


	//get pointers
	float *datam_ptr = thrust::raw_pointer_cast(datam_gpu.data());
	float *bvals_ptr = thrust::raw_pointer_cast(bvals_gpu.data());
	double *alpha_ptr = thrust::raw_pointer_cast(alpha_gpu.data());
	double *beta_ptr = thrust::raw_pointer_cast(beta_gpu.data());
	FibreGPU *fibres_ptr =  thrust::raw_pointer_cast(fibres_gpu.data());
	MultifibreGPU *multifibres_ptr = thrust::raw_pointer_cast(multifibres_gpu.data());
	double *signals_ptr = thrust::raw_pointer_cast(signals_gpu.data());
	double *isosignals_ptr = thrust::raw_pointer_cast(isosignals_gpu.data());
	hiprandState *randStates_ptr = thrust::raw_pointer_cast(randStates_gpu.data());

	double *angtmp_ptr = thrust::raw_pointer_cast(angtmp_gpu.data());
	double *oldangtmp_ptr = thrust::raw_pointer_cast(oldangtmp_gpu.data());
	double *oldsignals_ptr = thrust::raw_pointer_cast(oldsignals_gpu.data());
	double *oldisosignals_ptr = thrust::raw_pointer_cast(oldisosignals_gpu.data());

	float *records_null = thrust::raw_pointer_cast(recors_null_gpu.data());

	int amount_shared = VOXELS_BLOCK_MCMC*((THREADS_VOXEL_MCMC)*sizeof(double) + (10*nfib + 27)*sizeof(float) + (7*nfib + 20)*sizeof(int)+ sizeof(hiprandState));

	myfile << "Shared Memory Used in runmcmc_burnin: " << amount_shared << "\n";

   	if(nvox!=0){
		runmcmc_kernel<<< Dim_Grid, Dim_Block, amount_shared >>>(datam_ptr, bvals_ptr, alpha_ptr, beta_ptr, randStates_ptr, opts.R_prior_mean.value(), opts.R_prior_std.value(),opts.R_prior_fudge.value(), ndirections, nfib, nparams, opts.modelnum.value(), opts.fudge.value(), opts.f0.value(), opts.ardf0.value(), !opts.no_ard.value(), opts.rician.value(), gradnonlin, opts.updateproposalevery.value(), opts.nburn.value(), 0, 0, 0, oldsignals_ptr, oldisosignals_ptr, angtmp_ptr, oldangtmp_ptr, fibres_ptr, multifibres_ptr, signals_ptr, isosignals_ptr,records_null,records_null,records_null,records_null,records_null,records_null,records_null, records_null,records_null);
   		sync_check("runmcmc_burnin_kernel");
   	}

	gettimeofday(&t_tot2,NULL);
    	time=timeval_diff(&t_tot2,&t_tot1);
   	myfile << "TIME: " << time << " seconds\n";
	myfile << "-----------------------------------------------------" << "\n\n" ;
	myfile.close();
}


void runmcmc_record(	//INPUT
			thrust::device_vector<float>& 			datam_gpu,
			thrust::device_vector<float>& 			bvals_gpu,
			thrust::device_vector<double>& 			alpha_gpu,
			thrust::device_vector<double>& 			beta_gpu,
			thrust::device_vector<FibreGPU>& 		fibres_gpu,
			thrust::device_vector<MultifibreGPU>& 		multifibres_gpu,
			thrust::device_vector<double>&			signals_gpu,
			thrust::device_vector<double>&			isosignals_gpu,
			const int 					ndirections,
			thrust::device_vector<hiprandState>&		randStates_gpu,
			std::string 						output_file,
			//OUTPUT
			thrust::device_vector<float>&			rf0_gpu,
			thrust::device_vector<float>&			rtau_gpu,
			thrust::device_vector<float>&			rs0_gpu,
			thrust::device_vector<float>&			rd_gpu,
			thrust::device_vector<float>&			rdstd_gpu,
			thrust::device_vector<float>&			rR_gpu,
			thrust::device_vector<float>&			rth_gpu,
			thrust::device_vector<float>&			rph_gpu,
			thrust::device_vector<float>&			rf_gpu)
{
	xfibresOptions& opts = xfibresOptions::getInstance();

	std::ofstream myfile;
	myfile.open (output_file.data(), std::ios::out | std::ios::app );
   	myfile << "--------- MCMC ALGORITHM PART RECORD ON GPU --------- " << "\n";

   	struct timeval t_tot1,t_tot2;
   	double time;
   	time=0;

   	gettimeofday(&t_tot1,NULL);

	int totalrecords = (opts.njumps.value()/opts.sampleevery.value());

	int nvox = multifibres_gpu.size();
   	int nfib= opts.nfibres.value();
	int nparams;

	bool gradnonlin=opts.grad_file.set();

	if(opts.f0.value()) nparams=3+nfib*3;
	else nparams=2+nfib*3;
	if(opts.modelnum.value()>=2) nparams++;
	if(opts.modelnum.value()==3) nparams++;
	if(opts.rician.value()) nparams++;

	thrust::device_vector<double> angtmp_gpu;
	thrust::device_vector<double> oldangtmp_gpu;
	thrust::device_vector<double> oldsignals_gpu;
	thrust::device_vector<double> oldisosignals_gpu;

	angtmp_gpu.resize(nvox*ndirections*nfib);
	oldangtmp_gpu.resize(nvox*ndirections);
	oldsignals_gpu.resize(nvox*ndirections*nfib);
	oldisosignals_gpu.resize(nvox*ndirections);

	myfile << "Processing " << nvox << " voxels \n";

  	int blocks = nvox/VOXELS_BLOCK_MCMC;
	int nthreads_block = THREADS_VOXEL_MCMC*VOXELS_BLOCK_MCMC;
	if(nvox%VOXELS_BLOCK_MCMC) blocks++;
  	dim3 Dim_Grid(blocks, 1);
  	dim3 Dim_Block(nthreads_block,1);	//dimensions for MCMC

   	myfile << "NUM BLOCKS: " << blocks << "\n";
   	myfile << "THREADS PER BLOCK : " << nthreads_block << "\n";

	//get pointers
	float *datam_ptr = thrust::raw_pointer_cast(datam_gpu.data());
	float *bvals_ptr = thrust::raw_pointer_cast(bvals_gpu.data());
	double *alpha_ptr = thrust::raw_pointer_cast(alpha_gpu.data());
	double *beta_ptr = thrust::raw_pointer_cast(beta_gpu.data());
	FibreGPU *fibres_ptr =  thrust::raw_pointer_cast(fibres_gpu.data());
	MultifibreGPU *multifibres_ptr = thrust::raw_pointer_cast(multifibres_gpu.data());
	double *signals_ptr = thrust::raw_pointer_cast(signals_gpu.data());
	double *isosignals_ptr = thrust::raw_pointer_cast(isosignals_gpu.data());
	hiprandState *randStates_ptr = thrust::raw_pointer_cast(randStates_gpu.data());

	double *angtmp_ptr = thrust::raw_pointer_cast(angtmp_gpu.data());
	double *oldangtmp_ptr = thrust::raw_pointer_cast(oldangtmp_gpu.data());
	double *oldsignals_ptr = thrust::raw_pointer_cast(oldsignals_gpu.data());
	double *oldisosignals_ptr = thrust::raw_pointer_cast(oldisosignals_gpu.data());

	float *rf0_ptr = thrust::raw_pointer_cast(rf0_gpu.data());
	float *rtau_ptr = thrust::raw_pointer_cast(rtau_gpu.data());
	float *rs0_ptr = thrust::raw_pointer_cast(rs0_gpu.data());
	float *rd_ptr = thrust::raw_pointer_cast(rd_gpu.data());
	float *rdstd_ptr = thrust::raw_pointer_cast(rdstd_gpu.data());
	float *rR_ptr = thrust::raw_pointer_cast(rR_gpu.data());
	float *rth_ptr = thrust::raw_pointer_cast(rth_gpu.data());
	float *rph_ptr = thrust::raw_pointer_cast(rph_gpu.data());
	float *rf_ptr = thrust::raw_pointer_cast(rf_gpu.data());

	int amount_shared = VOXELS_BLOCK_MCMC*((THREADS_VOXEL_MCMC)*sizeof(double) + (10*nfib + 27)*sizeof(float) + (7*nfib + 20)*sizeof(int)+ sizeof(hiprandState));

	myfile << "Shared Memory Used in runmcmc_record: " << amount_shared << "\n";

   	if(nvox!=0){
		runmcmc_kernel<<< Dim_Grid, Dim_Block, amount_shared >>>(datam_ptr, bvals_ptr, alpha_ptr, beta_ptr, randStates_ptr, opts.R_prior_mean.value(), opts.R_prior_std.value(),opts.R_prior_fudge.value(), ndirections, nfib, nparams, opts.modelnum.value(), opts.fudge.value(), opts.f0.value(), opts.ardf0.value(), !opts.no_ard.value(), opts.rician.value(), gradnonlin, opts.updateproposalevery.value(), opts.njumps.value(), opts.nburn.value(), opts.sampleevery.value(), totalrecords, oldsignals_ptr, oldisosignals_ptr, angtmp_ptr, oldangtmp_ptr, fibres_ptr, multifibres_ptr, signals_ptr, isosignals_ptr, rf0_ptr, rtau_ptr, rs0_ptr, rd_ptr, rdstd_ptr, rR_ptr, rth_ptr, rph_ptr, rf_ptr);
   		sync_check("runmcmc_record_kernel");
   	}

   	gettimeofday(&t_tot2,NULL);
    	time=timeval_diff(&t_tot2,&t_tot1);
   	myfile << "TIME: " << time << " seconds\n";
	myfile << "-----------------------------------------------------" << "\n" ;
	myfile.close();
}
